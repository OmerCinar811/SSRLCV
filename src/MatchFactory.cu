#include "hip/hip_runtime.h"
#include "MatchFactory.cuh"

template class ssrlcv::MatchFactory<ssrlcv::SIFT_Descriptor>;


template<typename T>
ssrlcv::MatchFactory<T>::MatchFactory(){

}

template<typename T>
void ssrlcv::MatchFactory<T>::refineMatches(ssrlcv::Unity<ssrlcv::DMatch>* matches, float cutoffRatio){
  if(cutoffRatio == 0.0f){
    std::cout<<"ERROR illegal value used for cutoff ratio: 0.0"<<std::endl;
    exit(-1);
  }
  MemoryState origin = matches->state;
  if(origin != both){
    matches->transferMemoryTo(both);
  }

  float max = 0.0f;
  float min = FLT_MAX;
  for(int i = 0; i < matches->numElements; ++i){
    if(matches->host[i].distance < min) min = matches->host[i].distance;
    if(matches->host[i].distance > max) max = matches->host[i].distance;
  }

  if(origin == gpu) matches->clear(cpu);

  thrust::device_ptr<DMatch> needsCompacting(matches->device);
  thrust::device_ptr<DMatch> end = thrust::remove_if(needsCompacting, needsCompacting + matches->numElements, match_dist_thresholder((max-min)*cutoffRatio + min));
  unsigned int numElementsBelowThreshold = end - needsCompacting;

  printf("%d matches have been refined to %d matches using a cutoff of %f (%f percentile)\n",matches->numElements,numElementsBelowThreshold,(max-min)*cutoffRatio + min,cutoffRatio);

  FeatureMatch<T>* compactedMatches_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&compactedMatches_device,numElementsBelowThreshold*sizeof(DMatch)));
  CudaSafeCall(hipMemcpy(compactedMatches_device,matches->device,numElementsBelowThreshold*sizeof(DMatch),hipMemcpyDeviceToDevice));

  matches->setData(compactedMatches_device,numElementsBelowThreshold,gpu);

  if(origin == cpu) matches->setMemoryState(cpu);
}
template<typename T>
void ssrlcv::MatchFactory<T>::refineMatches(ssrlcv::Unity<ssrlcv::FeatureMatch<T>>* matches, float cutoffRatio){
  if(cutoffRatio == 0.0f){
    std::cout<<"ERROR illegal value used for cutoff ratio: 0.0"<<std::endl;
    exit(-1);
  }
  MemoryState origin = matches->state;
  if(origin != both){
    matches->transferMemoryTo(both);
  }

  float max = 0.0f;
  float min = FLT_MAX;
  for(int i = 0; i < matches->numElements; ++i){
    if(matches->host[i].distance < min) min = matches->host[i].distance;
    if(matches->host[i].distance > max) max = matches->host[i].distance;
  }
  if(origin == gpu) matches->clear(cpu);

  thrust::device_ptr<FeatureMatch<T>> needsCompacting(matches->device);
  thrust::device_ptr<FeatureMatch<T>> end = thrust::remove_if(needsCompacting, needsCompacting + matches->numElements, match_dist_thresholder((max-min)*cutoffRatio + min));
  unsigned int numElementsBelowThreshold = end - needsCompacting;

  printf("%d matches have been refined to %d matches using a cutoff of %f (%f percentile)\n",matches->numElements,numElementsBelowThreshold,(max-min)*cutoffRatio + min,cutoffRatio);

  FeatureMatch<T>* compactedMatches_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&compactedMatches_device,numElementsBelowThreshold*sizeof(FeatureMatch<T>)));
  CudaSafeCall(hipMemcpy(compactedMatches_device,matches->device,numElementsBelowThreshold*sizeof(FeatureMatch<T>),hipMemcpyDeviceToDevice));

  matches->setData(compactedMatches_device,numElementsBelowThreshold,gpu);

  if(origin == cpu) matches->setMemoryState(cpu);
}

template<typename T>
void ssrlcv::MatchFactory<T>::sortMatches(Unity<DMatch>* matches){
  if(matches->state == gpu || matches->fore == gpu){
    thrust::device_ptr<DMatch> toSort(matches->device);
    thrust::sort(toSort, toSort + matches->numElements,match_dist_comparator());
    matches->fore = gpu;
    if(matches->state == both) matches->transferMemoryTo(cpu);
  }
  else{
    unsigned long len = matches->numElements;
    // insertion sort
    // each match element is accessed with allMatches->host[]
    unsigned long i = 0;
    unsigned long j = 0;
    ssrlcv::DMatch temp;
    while (i < len){
      j = i;
      while (j > 0 && matches->host[j-1].distance > matches->host[j].distance){
        temp = matches->host[j];
        matches->host[j] = matches->host[j-1];
        matches->host[j-1] = temp;
        j--;
      }
      i++;
    }
    matches->fore = cpu;
    if(matches->state == both) matches->transferMemoryTo(gpu);
  }
}
template<typename T>
void ssrlcv::MatchFactory<T>::sortMatches(Unity<FeatureMatch<T>>* matches){
  if(matches->state == gpu || matches->fore == gpu){
    thrust::device_ptr<FeatureMatch<T>> toSort(matches->device);
    thrust::sort(toSort, toSort + matches->numElements,match_dist_comparator());
    matches->fore = gpu;
    if(matches->state == both) matches->transferMemoryTo(cpu);
  }
  else{
    unsigned long len = matches->numElements;
    // insertion sort
    // each match element is accessed with allMatches->host[]
    unsigned long i = 0;
    unsigned long j = 0;
    ssrlcv::FeatureMatch<T> temp;
    while (i < len){
      j = i;
      while (j > 0 && matches->host[j-1].distance > matches->host[j].distance){
        temp = matches->host[j];
        matches->host[j] = matches->host[j-1];
        matches->host[j-1] = temp;
        j--;
      }
      i++;
    }
    matches->fore = cpu;
    if(matches->state == both) matches->transferMemoryTo(gpu);
  }
}

template<typename T>
ssrlcv::Unity<ssrlcv::Match>* ssrlcv::MatchFactory<T>::getRawMatches(Unity<DMatch>* matches){
  if(matches->state == gpu || matches->fore == gpu){
    Match* rawMatches_device = nullptr;
    CudaSafeCall(hipMalloc((void**)&rawMatches_device, matches->numElements*sizeof(DMatch)));
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    getFlatGridBlock(matches->numElements,grid,block);
    convertMatchToRaw<<<grid,block>>>(matches->numElements,rawMatches_device,matches->device);
    hipDeviceSynchronize();
    CudaCheckError();
    return new Unity<Match>(rawMatches_device,matches->numElements,gpu);
  }
  else{
    Match* rawMatches_host = new Match[matches->numElements];
    for(int i = 0; i < matches->numElements; ++i){
      for(int f = 0; f < 2; ++f){
        rawMatches_host[i] = Match(matches->host[i]);
      }
    }
    return new Unity<Match>(rawMatches_host, matches->numElements, cpu);
  }
}
template<typename T>
ssrlcv::Unity<ssrlcv::Match>* ssrlcv::MatchFactory<T>::getRawMatches(Unity<FeatureMatch<T>>* matches){
  if(matches->state == gpu || matches->fore == gpu){
    Match* rawMatches_device = nullptr;
    CudaSafeCall(hipMalloc((void**)&rawMatches_device, matches->numElements*sizeof(Match)));
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    getFlatGridBlock(matches->numElements,grid,block);
    convertMatchToRaw<<<grid,block>>>(matches->numElements,rawMatches_device,matches->device);
    hipDeviceSynchronize();
    CudaCheckError();
    return new Unity<Match>(rawMatches_device,matches->numElements,gpu);
  }
  else{
    Match* rawMatches_host = new Match[matches->numElements];
    for(int i = 0; i < matches->numElements; ++i){
      for(int f = 0; f < 2; ++f){
        rawMatches_host[i] = Match(matches->host[i]);
      }
    }
    return new Unity<Match>(rawMatches_host, matches->numElements, cpu);
  }
}

template<typename T>
ssrlcv::Unity<ssrlcv::Match>* ssrlcv::MatchFactory<T>::generateMatches(Image* query, Unity<Feature<T>>* queryFeatures, Image* target, Unity<Feature<T>>* targetFeatures){
  MemoryState origin[2] = {queryFeatures->state, targetFeatures->state};

  if(queryFeatures->fore == cpu) queryFeatures->setMemoryState(gpu);
  if(targetFeatures->fore == cpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->numElements;

  Match* matches_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&matches_device, numPossibleMatches*sizeof(Match)));

  Unity<Match>* matches = new Unity<Match>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {1024,1,1};
  getGrid(matches->numElements,grid);

  clock_t timer = clock();

  matchFeaturesBruteForce<<<grid, block>>>(query->id, queryFeatures->numElements, queryFeatures->device,
    target->id, targetFeatures->numElements, targetFeatures->device, matches->device);

  hipDeviceSynchronize();
  CudaCheckError();

  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != queryFeatures->state) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != targetFeatures->state) targetFeatures->setMemoryState(origin[1]);

  return matches;
}
template<typename T>
ssrlcv::Unity<ssrlcv::Match>* ssrlcv::MatchFactory<T>::generateMatchesConstrained(Image* query, Unity<Feature<T>>* queryFeatures, Image* target, Unity<Feature<T>>* targetFeatures, float epsilon){
  MemoryState origin[2] = {queryFeatures->state, targetFeatures->state};

  if(queryFeatures->fore == cpu) queryFeatures->setMemoryState(gpu);
  if(targetFeatures->fore == cpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->numElements;

  Match* matches_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&matches_device, numPossibleMatches*sizeof(Match)));

  Unity<Match>* matches = new Unity<Match>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {1024,1,1};
  getGrid(matches->numElements,grid);

  clock_t timer = clock();
  float3* fundamental = new float3[3];
  calcFundamentalMatrix_2View(query, target, fundamental);

  float3* fundamental_device;
  CudaSafeCall(hipMalloc((void**)&fundamental_device, 3*sizeof(float3)));
  CudaSafeCall(hipMemcpy(fundamental_device, fundamental, 3*sizeof(float3), hipMemcpyHostToDevice));

  matchFeaturesConstrained<<<grid, block>>>(query->id, queryFeatures->numElements, queryFeatures->device,
    target->id, targetFeatures->numElements, targetFeatures->device, matches->device, epsilon, fundamental_device);
  hipDeviceSynchronize();
  CudaCheckError();

  CudaSafeCall(hipFree(fundamental_device));

  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != queryFeatures->state) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != targetFeatures->state) targetFeatures->setMemoryState(origin[1]);

  return matches;
}
template<typename T>
ssrlcv::Unity<ssrlcv::DMatch>*ssrlcv::MatchFactory<T>:: generateDistanceMatches(Image* query, Unity<Feature<T>>* queryFeatures, Image* target, Unity<Feature<T>>* targetFeatures){
  MemoryState origin[2] = {queryFeatures->state, targetFeatures->state};

  if(queryFeatures->fore == cpu) queryFeatures->setMemoryState(gpu);
  if(targetFeatures->fore == cpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->numElements;

  DMatch* matches_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&matches_device, numPossibleMatches*sizeof(DMatch)));

  Unity<DMatch>* matches = new Unity<DMatch>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {1024,1,1};
  getGrid(matches->numElements,grid);

  clock_t timer = clock();

  matchFeaturesBruteForce<<<grid, block>>>(query->id, queryFeatures->numElements, queryFeatures->device,
    target->id, targetFeatures->numElements, targetFeatures->device, matches->device);

  hipDeviceSynchronize();
  CudaCheckError();

  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != queryFeatures->state) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != targetFeatures->state) targetFeatures->setMemoryState(origin[1]);

  return matches;
}
template<typename T>
ssrlcv::Unity<ssrlcv::DMatch>*ssrlcv::MatchFactory<T>:: generateDistanceMatchesConstrained(Image* query, Unity<Feature<T>>* queryFeatures, Image* target, Unity<Feature<T>>* targetFeatures, float epsilon){
  MemoryState origin[2] = {queryFeatures->state, targetFeatures->state};

  if(queryFeatures->fore == cpu) queryFeatures->setMemoryState(gpu);
  if(targetFeatures->fore == cpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->numElements;

  DMatch* matches_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&matches_device, numPossibleMatches*sizeof(DMatch)));

  Unity<DMatch>* matches = new Unity<DMatch>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {1024,1,1};
  getGrid(matches->numElements,grid);

  clock_t timer = clock();
  float3* fundamental = new float3[3];
  calcFundamentalMatrix_2View(query, target, fundamental);

  float3* fundamental_device;
  CudaSafeCall(hipMalloc((void**)&fundamental_device, 3*sizeof(float3)));
  CudaSafeCall(hipMemcpy(fundamental_device, fundamental, 3*sizeof(float3), hipMemcpyHostToDevice));

  matchFeaturesConstrained<<<grid, block>>>(query->id, queryFeatures->numElements, queryFeatures->device,
    target->id, targetFeatures->numElements, targetFeatures->device, matches->device, epsilon, fundamental_device);
  hipDeviceSynchronize();
  CudaCheckError();

  CudaSafeCall(hipFree(fundamental_device));

  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != queryFeatures->state) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != targetFeatures->state) targetFeatures->setMemoryState(origin[1]);

  return matches;
}
template<typename T>
ssrlcv::Unity<ssrlcv::FeatureMatch<T>>* ssrlcv::MatchFactory<T>::generateFeatureMatches(ssrlcv::Image* query, ssrlcv::Unity<ssrlcv::Feature<T>>* queryFeatures,
ssrlcv::Image* target, ssrlcv::Unity<ssrlcv::Feature<T>>* targetFeatures){

  MemoryState origin[2] = {queryFeatures->state, targetFeatures->state};

  if(queryFeatures->fore == cpu) queryFeatures->setMemoryState(gpu);
  if(targetFeatures->fore == cpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->numElements;

  FeatureMatch<T>* matches_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&matches_device, numPossibleMatches*sizeof(FeatureMatch<T>)));

  Unity<FeatureMatch<T>>* matches = new Unity<FeatureMatch<T>>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {1024,1,1};
  getGrid(matches->numElements,grid);

  clock_t timer = clock();

  matchFeaturesBruteForce<T><<<grid, block>>>(query->id, queryFeatures->numElements, queryFeatures->device,
    target->id, targetFeatures->numElements, targetFeatures->device, matches->device);

  hipDeviceSynchronize();
  CudaCheckError();

  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != queryFeatures->state) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != targetFeatures->state) targetFeatures->setMemoryState(origin[1]);

  return matches;
}
template<typename T>
ssrlcv::Unity<ssrlcv::FeatureMatch<T>>* ssrlcv::MatchFactory<T>::generateFeatureMatchesConstrained(ssrlcv::Image* query, ssrlcv::Unity<ssrlcv::Feature<T>>* queryFeatures,
ssrlcv::Image* target, ssrlcv::Unity<ssrlcv::Feature<T>>* targetFeatures, float epsilon){

  MemoryState origin[2] = {queryFeatures->state, targetFeatures->state};

  if(queryFeatures->fore == cpu) queryFeatures->setMemoryState(gpu);
  if(targetFeatures->fore == cpu) targetFeatures->setMemoryState(gpu);

  unsigned int numPossibleMatches = queryFeatures->numElements;

  FeatureMatch<T>* matches_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&matches_device, numPossibleMatches*sizeof(FeatureMatch<T>)));

  Unity<FeatureMatch<T>>* matches = new Unity<FeatureMatch<T>>(matches_device, numPossibleMatches, gpu);

  dim3 grid = {1,1,1};
  dim3 block = {1024,1,1};
  getGrid(matches->numElements,grid);

  clock_t timer = clock();
  float3* fundamental = new float3[3];
  calcFundamentalMatrix_2View(query, target, fundamental);

  float3* fundamental_device;
  CudaSafeCall(hipMalloc((void**)&fundamental_device, 3*sizeof(float3)));
  CudaSafeCall(hipMemcpy(fundamental_device, fundamental, 3*sizeof(float3), hipMemcpyHostToDevice));

  matchFeaturesConstrained<T><<<grid, block>>>(query->id, queryFeatures->numElements, queryFeatures->device,
    target->id, targetFeatures->numElements, targetFeatures->device, matches->device, epsilon, fundamental_device);
  hipDeviceSynchronize();
  CudaCheckError();

  CudaSafeCall(hipFree(fundamental_device));

  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);

  if(origin[0] != queryFeatures->state) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != targetFeatures->state) targetFeatures->setMemoryState(origin[1]);

  return matches;

}

template<typename T>
ssrlcv::Unity<ssrlcv::FeatureMatch<T>>* ssrlcv::MatchFactory<T>::generateSubPixelMatches(ssrlcv::Image* query, ssrlcv::Unity<ssrlcv::Feature<T>>* queryFeatures,
ssrlcv::Image* target, ssrlcv::Unity<ssrlcv::Feature<T>>* targetFeatures){

  MemoryState origin[2] = {queryFeatures->state, targetFeatures->state};

  if(queryFeatures->fore == cpu) queryFeatures->setMemoryState(gpu);
  if(targetFeatures->fore == cpu) targetFeatures->setMemoryState(gpu);

  Unity<FeatureMatch<T>>* matches = this->generateFeatureMatches(query, queryFeatures, target, targetFeatures);
  matches->transferMemoryTo(gpu);

  SubpixelM7x7* subDescriptors_device;
  CudaSafeCall(hipMalloc((void**)&subDescriptors_device, matches->numElements*sizeof(SubpixelM7x7)));

  dim3 grid = {1,1,1};
  dim3 block = {9,9,1};
  getGrid(matches->numElements, grid);
  std::cout<<"initializing subPixelMatches..."<<std::endl;
  clock_t timer = clock();
  initializeSubPixels<T><<<grid, block>>>(matches->numElements, matches->device, subDescriptors_device,
    query->size, queryFeatures->numElements, queryFeatures->device,
    target->size, targetFeatures->numElements, targetFeatures->device);

  hipDeviceSynchronize();
  CudaCheckError();
  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);

  Spline* splines_device;
  CudaSafeCall(hipMalloc((void**)&splines_device, matches->numElements*2*sizeof(Spline)));

  grid = {1,1,1};
  block = {6,6,4};
  getGrid(matches->numElements*2, grid);

  std::cout<<"filling bicubic splines..."<<std::endl;
  timer = clock();
  fillSplines<<<grid,block>>>(matches->numElements, subDescriptors_device, splines_device);
  hipDeviceSynchronize();
  CudaCheckError();
  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);
  CudaSafeCall(hipFree(subDescriptors_device));

  std::cout<<"determining subpixel locations..."<<std::endl;
  timer = clock();
  determineSubPixelLocationsBruteForce<T><<<grid,block>>>(0.1, matches->numElements, matches->device, splines_device);
  hipDeviceSynchronize();
  CudaCheckError();
  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);
  CudaSafeCall(hipFree(splines_device));

  if(origin[0] != queryFeatures->state) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != targetFeatures->state) targetFeatures->setMemoryState(origin[1]);

  return matches;
}
template<typename T>
ssrlcv::Unity<ssrlcv::FeatureMatch<T>>* ssrlcv::MatchFactory<T>::generateSubPixelMatchesConstrained(ssrlcv::Image* query, ssrlcv::Unity<ssrlcv::Feature<T>>* queryFeatures,
ssrlcv::Image* target, ssrlcv::Unity<ssrlcv::Feature<T>>* targetFeatures, float epsilon){
  MemoryState origin[2] = {queryFeatures->state, targetFeatures->state};

  if(queryFeatures->fore == cpu) queryFeatures->setMemoryState(gpu);
  if(targetFeatures->fore == cpu) targetFeatures->setMemoryState(gpu);

  Unity<FeatureMatch<T>>* matches = this->generateFeatureMatchesConstrained(query, queryFeatures, target, targetFeatures, epsilon);
  matches->transferMemoryTo(gpu);

  SubpixelM7x7* subDescriptors_device;
  CudaSafeCall(hipMalloc((void**)&subDescriptors_device, matches->numElements*sizeof(SubpixelM7x7)));

  dim3 grid = {1,1,1};
  dim3 block = {9,9,1};
  getGrid(matches->numElements, grid);
  std::cout<<"initializing subPixelMatches..."<<std::endl;
  clock_t timer = clock();
  initializeSubPixels<T><<<grid, block>>>(matches->numElements, matches->device, subDescriptors_device,
    query->size, queryFeatures->numElements, queryFeatures->device,
    target->size, targetFeatures->numElements, targetFeatures->device);

  hipDeviceSynchronize();
  CudaCheckError();
  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);

  Spline* splines_device;
  CudaSafeCall(hipMalloc((void**)&splines_device, matches->numElements*2*sizeof(Spline)));

  grid = {1,1,1};
  block = {6,6,4};
  getGrid(matches->numElements*2, grid);

  std::cout<<"filling bicubic splines..."<<std::endl;
  timer = clock();
  fillSplines<<<grid,block>>>(matches->numElements, subDescriptors_device, splines_device);
  hipDeviceSynchronize();
  CudaCheckError();
  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);
  CudaSafeCall(hipFree(subDescriptors_device));

  std::cout<<"determining subpixel locations..."<<std::endl;
  timer = clock();
  determineSubPixelLocationsBruteForce<T><<<grid,block>>>(0.1, matches->numElements, matches->device, splines_device);
  hipDeviceSynchronize();
  CudaCheckError();
  printf("done in %f seconds.\n\n",((float) clock() -  timer)/CLOCKS_PER_SEC);
  CudaSafeCall(hipFree(splines_device));

  if(origin[0] != queryFeatures->state) queryFeatures->setMemoryState(origin[0]);
  if(origin[1] != targetFeatures->state) targetFeatures->setMemoryState(origin[1]);

  return matches;
}

/*
CUDA implementations
*/

__constant__ int ssrlcv::splineHelper[4][4] = {
  {1,0,0,0},
  {0,0,1,0},
  {-3,3,-2,-1},
  {2,-2,1,1}
};
__constant__ int ssrlcv::splineHelperInv[4][4] = {
  {1,0,-3,2},
  {0,0,3,-2},
  {0,1,-2,1},
  {0,0,-1,1}
};

__device__ __host__ __forceinline__ float ssrlcv::sum(const float3 &a){
  return a.x + a.y + a.z;
}
__device__ __forceinline__ float ssrlcv::square(const float &a){
  return a*a;
}
__device__ __forceinline__ float ssrlcv::atomicMinFloat (float * addr, float value) {
  float old;
  old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
    __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));
  return old;
}
__device__ __forceinline__ float ssrlcv::findSubPixelContributer(const float2 &loc, const int &width){
  return ((loc.y - 12)*(width - 24)) + (loc.x - 12);
}

__device__ __forceinline__ float ssrlcv::calcElucidSq(const Feature<SIFT_Descriptor>& a, const Feature<SIFT_Descriptor>& b){
  float dist = 0.0f;
  for(int i = 0; i < 128; ++i){
    dist += square(((float)a.descriptor.values[i]-b.descriptor.values[i]));
  }
  dist += a.descriptor.theta - b.descriptor.theta;
  dist += a.descriptor.sigma - b.descriptor.sigma;
  dist += dotProduct(a.loc - b.loc,a.loc - b.loc);
  return dist;
}
__device__ __forceinline__ float ssrlcv::calcElucidSq(const Feature<SIFT_Descriptor>& a, const Feature<SIFT_Descriptor>& b, const float &bestMatch){
  float dist = 0.0f;
  for(int i = 0; i < 128 && dist < bestMatch; ++i){
    dist += square(((float)a.descriptor.values[i]-b.descriptor.values[i]));
  }
  if(dist < bestMatch) dist += a.descriptor.theta - b.descriptor.theta;
  else return dist;
  if(dist < bestMatch) dist += a.descriptor.sigma - b.descriptor.sigma;
  else return dist;
  if(dist < bestMatch) dist += dotProduct(a.loc - b.loc,a.loc - b.loc);
  return dist;
}
__device__ __forceinline__ float ssrlcv::calcElucidSq(const SIFT_Descriptor& a, const SIFT_Descriptor& b){
  float dist = 0.0f;
  for(int i = 0; i < 128; ++i){
    dist += square(((float)a.values[i]-b.values[i]));
  }
  dist += a.theta - b.theta;
  dist += a.sigma - b.sigma;
  return dist;
}
__device__ __forceinline__ float ssrlcv::calcElucidSq(const SIFT_Descriptor& a, const SIFT_Descriptor& b, const float &bestMatch){
  float dist = 0.0f;
  for(int i = 0; i < 128 && dist < bestMatch; ++i){
    dist += square(((float)a.values[i]-b.values[i]));
  }
  if(dist < bestMatch) dist += a.theta - b.theta;
  else return dist;
  if(dist < bestMatch) dist += a.sigma - b.sigma;
  else return dist;
  return dist;
}


/*
matching
*/
//base matching kernels
template<typename T>
__global__ void ssrlcv::matchFeaturesBruteForce(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, Match* matches){
  unsigned long blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[1024];
    __shared__ float localDist[1024];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = FLT_MAX;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesQuery;
    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 1024){
      currentDist = calcElucidSq(feature,featuresTarget[f],localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = FLT_MAX;
    int matchIndex = -1;
    for(int i = 0; i < 1024; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    Match match;
    match.keyPoints[0].loc = feature.loc;
    match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
    match.keyPoints[0].parentId = queryImageID;
    match.keyPoints[1].parentId = targetImageID;
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesBruteForce(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, DMatch* matches){
  unsigned long blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[1024];
    __shared__ float localDist[1024];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = FLT_MAX;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesQuery;
    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 1024){
      currentDist = calcElucidSq(feature,featuresTarget[f],localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = FLT_MAX;
    int matchIndex = -1;
    for(int i = 0; i < 1024; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    DMatch match;
    match.keyPoints[0].loc = feature.loc;
    match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
    match.keyPoints[0].parentId = queryImageID;
    match.keyPoints[1].parentId = targetImageID;
    match.distance = currentDist;
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesBruteForce(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, ssrlcv::FeatureMatch<T>* matches){
  unsigned long blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[1024];
    __shared__ float localDist[1024];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = FLT_MAX;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesQuery;
    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 1024){
      currentDist = calcElucidSq(feature,featuresTarget[f],localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = FLT_MAX;
    int matchIndex = -1;
    for(int i = 0; i < 1024; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    FeatureMatch<T> match;
    match.descriptors[0] = feature.descriptor;
    match.descriptors[1] = featuresTarget[matchIndex].descriptor;
    match.keyPoints[0].loc = feature.loc;
    match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
    match.keyPoints[0].parentId = queryImageID;
    match.keyPoints[1].parentId = targetImageID;
    match.distance = currentDist;
    matches[blockId] = match;
  }
}

template<typename T>
__global__ void ssrlcv::matchFeaturesConstrained(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, Match* matches, float epsilon, float3 fundamental[3]){
  unsigned long blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[1024];
    __shared__ float localDist[1024];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = FLT_MAX;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesQuery;

    float3 epipolar = {0.0f,0.0f,0.0f};
    epipolar.x = (fundamental[0].x*feature.loc.x) + (fundamental[0].y*feature.loc.y) + fundamental[0].z;
    epipolar.y = (fundamental[1].x*feature.loc.x) + (fundamental[1].y*feature.loc.y) + fundamental[1].z;
    epipolar.z = (fundamental[2].x*feature.loc.x) + (fundamental[2].y*feature.loc.y) + fundamental[2].z;

    float p = 0.0f;

    Feature<T> currentFeature;
    float regEpsilon = epsilon;

    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 1024){

      currentFeature = featuresTarget[f];
      //ax + by + c = 0
      p = -1*((epipolar.x*currentFeature.loc.x) + epipolar.z)/epipolar.y;
      if(abs(currentFeature.loc.y - p) >= regEpsilon) continue;
      currentDist = calcElucidSq(feature,currentFeature,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = FLT_MAX;
    int matchIndex = -1;
    for(int i = 0; i < 1024; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    Match match;
    match.keyPoints[0].loc = feature.loc;
    match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
    match.keyPoints[0].parentId = queryImageID;
    match.keyPoints[1].parentId = targetImageID;
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesConstrained(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, DMatch* matches, float epsilon, float3 fundamental[3]){
  unsigned long blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[1024];
    __shared__ float localDist[1024];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = FLT_MAX;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesQuery;

    float3 epipolar = {0.0f,0.0f,0.0f};
    epipolar.x = (fundamental[0].x*feature.loc.x) + (fundamental[0].y*feature.loc.y) + fundamental[0].z;
    epipolar.y = (fundamental[1].x*feature.loc.x) + (fundamental[1].y*feature.loc.y) + fundamental[1].z;
    epipolar.z = (fundamental[2].x*feature.loc.x) + (fundamental[2].y*feature.loc.y) + fundamental[2].z;

    float p = 0.0f;

    Feature<T> currentFeature;
    float regEpsilon = epsilon;

    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 1024){

      currentFeature = featuresTarget[f];
      //ax + by + c = 0
      p = -1*((epipolar.x*currentFeature.loc.x) + epipolar.z)/epipolar.y;
      if(abs(currentFeature.loc.y - p) >= regEpsilon) continue;
      currentDist = calcElucidSq(feature,currentFeature,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = FLT_MAX;
    int matchIndex = -1;
    for(int i = 0; i < 1024; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    DMatch match;
    match.keyPoints[0].loc = feature.loc;
    match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
    match.keyPoints[0].parentId = queryImageID;
    match.keyPoints[1].parentId = targetImageID;
    match.distance = currentDist;
    matches[blockId] = match;
  }
}
template<typename T>
__global__ void ssrlcv::matchFeaturesConstrained(unsigned int queryImageID, unsigned long numFeaturesQuery,
ssrlcv::Feature<T>* featuresQuery, unsigned int targetImageID, unsigned long numFeaturesTarget,
ssrlcv::Feature<T>* featuresTarget, ssrlcv::FeatureMatch<T>* matches, float epsilon, float3 fundamental[3]){
  unsigned long blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numFeaturesQuery){
    Feature<T> feature = featuresQuery[blockId];
    __shared__ int localMatch[1024];
    __shared__ float localDist[1024];
    localMatch[threadIdx.x] = -1;
    localDist[threadIdx.x] = FLT_MAX;
    __syncthreads();
    float currentDist = 0.0f;
    unsigned long numFeaturesTarget_register = numFeaturesQuery;

    float3 epipolar = {0.0f,0.0f,0.0f};
    epipolar.x = (fundamental[0].x*feature.loc.x) + (fundamental[0].y*feature.loc.y) + fundamental[0].z;
    epipolar.y = (fundamental[1].x*feature.loc.x) + (fundamental[1].y*feature.loc.y) + fundamental[1].z;
    epipolar.z = (fundamental[2].x*feature.loc.x) + (fundamental[2].y*feature.loc.y) + fundamental[2].z;

    float p = 0.0f;

    Feature<T> currentFeature;
    float regEpsilon = epsilon;

    for(int f = threadIdx.x; f < numFeaturesTarget_register; f += 1024){

      currentFeature = featuresTarget[f];
      //ax + by + c = 0
      p = -1*((epipolar.x*currentFeature.loc.x) + epipolar.z)/epipolar.y;
      if(abs(currentFeature.loc.y - p) >= regEpsilon) continue;
      currentDist = calcElucidSq(feature,currentFeature,localDist[threadIdx.x]);
      if(localDist[threadIdx.x] > currentDist){
        localDist[threadIdx.x] = currentDist;
        localMatch[threadIdx.x] = f;
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    currentDist = FLT_MAX;
    int matchIndex = -1;
    for(int i = 0; i < 1024; ++i){
      if(currentDist > localDist[i]){
        currentDist = localDist[i];
        matchIndex = localMatch[i];
      }
    }
    FeatureMatch<T> match;
    match.descriptors[0] = feature.descriptor;
    match.descriptors[1] = featuresTarget[matchIndex].descriptor;
    match.keyPoints[0].loc = feature.loc;
    match.keyPoints[1].loc = featuresTarget[matchIndex].loc;
    match.keyPoints[0].parentId = queryImageID;
    match.keyPoints[1].parentId = targetImageID;
    match.distance = currentDist;
    matches[blockId] = match;
  }
}


//subpixel kernels
template<typename T>
__global__ void ssrlcv::initializeSubPixels(unsigned long numMatches, ssrlcv::FeatureMatch<T>* matches, ssrlcv::SubpixelM7x7* subPixelDescriptors,
uint2 querySize, unsigned long numFeaturesQuery, ssrlcv::Feature<T>* featuresQuery,
uint2 targetSize, unsigned long numFeaturesTarget, ssrlcv::Feature<T>* featuresTarget){
  unsigned long blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numMatches){
    __shared__ SubpixelM7x7 subDescriptor;
    FeatureMatch<T> match = matches[blockId];

    //this now needs to be actual indices to contributers
    int2 contrib = {((int)threadIdx.x) - 4, ((int)threadIdx.y) - 4};
    int contribQuery = findSubPixelContributer(match.keyPoints[0].loc + contrib, querySize.x);
    int contribTarget = findSubPixelContributer(match.keyPoints[1].loc + contrib, targetSize.x);

    int pairedMatchIndex = findSubPixelContributer(match.keyPoints[1].loc, targetSize.x);

    bool foundM1 = false;
    bool foundM2 = false;

    if(contribTarget >= 0 && contribTarget < numFeaturesTarget){
      subDescriptor.M1[threadIdx.x][threadIdx.y] = calcElucidSq(featuresQuery[blockId], featuresTarget[contribTarget]);
      foundM1 = true;
    }
    if(contribQuery >= 0 && contribQuery < numFeaturesQuery){
      subDescriptor.M2[threadIdx.x][threadIdx.y] = calcElucidSq(featuresQuery[contribQuery], featuresTarget[pairedMatchIndex]);
      foundM2 = true;
    }
    __syncthreads();
    //COME up with better way to do this
    if(!foundM1){
      float val = 0.0f;
      for(int x = 0; x < 9; ++x){
        for(int y = 0; y < 9; ++y){
          val += subDescriptor.M1[x][y];
        }
      }
      subDescriptor.M1[threadIdx.x][threadIdx.y] = val/81;
    }
    if(!foundM2){
      float val = 0.0f;
      for(int x = 0; x < 9; ++x){
        for(int y = 0; y < 9; ++y){
          val += subDescriptor.M2[x][y];
        }
      }
      subDescriptor.M2[threadIdx.x][threadIdx.y] = val/81;
    }
    __syncthreads();
    if(threadIdx.x == 0 && threadIdx.y == 0){
      subPixelDescriptors[blockId] = subDescriptor;
    }
  }
}
__global__ void ssrlcv::fillSplines(unsigned long numMatches, SubpixelM7x7* subPixelDescriptors, ssrlcv::Spline* splines){
  unsigned long blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numMatches*2){
    float descriptor[9][9];
    for(int x = 0; x < 9; ++x){
      for(int y = 0; y < 9; ++y){
        descriptor[x][y] = (blockId%2 == 0) ? subPixelDescriptors[blockId/2].M1[x][y] : subPixelDescriptors[blockId/2].M2[x][y];
      }
    }

    __shared__ Spline spline;
    int2 corner = {
      ((int)threadIdx.z)%2,
      ((int)threadIdx.z)/2
    };
    int2 contributer = {
      ((int)threadIdx.x) + 2 + corner.x,
      ((int)threadIdx.y) + 2 + corner.y
    };
    float4 localCoeff;
    localCoeff.x = descriptor[contributer.x][contributer.y];
    localCoeff.y = descriptor[contributer.x + 1][contributer.y] - descriptor[contributer.x - 1][contributer.y];
    localCoeff.z = descriptor[contributer.x][contributer.y + 1] - descriptor[contributer.x][contributer.y - 1];
    localCoeff.w = descriptor[contributer.x + 1][contributer.y + 1] - descriptor[contributer.x - 1][contributer.y - 1];

    spline.coeff[threadIdx.x][threadIdx.y][corner.x][corner.y] = localCoeff.x;
    spline.coeff[threadIdx.x][threadIdx.y][corner.x][corner.y + 2] = localCoeff.y;
    spline.coeff[threadIdx.x][threadIdx.y][corner.x + 2][corner.y] = localCoeff.z;
    spline.coeff[threadIdx.x][threadIdx.y][corner.x + 2][corner.y + 2] = localCoeff.z;

    // Multiplying matrix a and b and storing in array mult.
    if(threadIdx.z != 0) return;
    float mult[4][4] = {0.0f};
    for(int i = 0; i < 4; ++i){
      for(int j = 0; j < 4; ++j){
        for(int c = 0; c < 4; ++c){
          mult[i][j] += splineHelper[i][c]*spline.coeff[threadIdx.x][threadIdx.y][c][j];
        }
      }
    }
    for(int i = 0; i < 4; ++i){
      for(int j = 0; j < 4; ++j){
        spline.coeff[threadIdx.x][threadIdx.y][i][j] = 0.0f;
      }
    }
    for(int i = 0; i < 4; ++i){
      for(int j = 0; j < 4; ++j){
        for(int c = 0; c < 4; ++c){
          spline.coeff[threadIdx.x][threadIdx.y][i][j] += mult[i][c]*splineHelperInv[c][j];
        }
      }
    }

    __syncthreads();
    splines[blockId] = spline;
  }
}
template<typename T>
__global__ void ssrlcv::determineSubPixelLocationsBruteForce(float increment, unsigned long numMatches, ssrlcv::FeatureMatch<T>* matches, ssrlcv::Spline* splines){
  unsigned long blockId = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockId < numMatches*2){
    __shared__ float minimum;
    minimum = FLT_MAX;
    __syncthreads();
    float localCoeff[4][4];
    for(int i = 0; i < 4; ++i){
      for(int j = 0; j < 4; ++j){
        localCoeff[i][j] = splines[blockId].coeff[threadIdx.x][threadIdx.y][i][j];
      }
    }
    float value = 0.0f;
    float localMin = FLT_MAX;
    float2 localSubLoc = {0.0f,0.0f};
    for(float x = -1.0f; x <= 1.0f; x+=increment){
      for(float y = -1.0f; y <= 1.0f; y+=increment){
        value = 0.0f;
        for(int i = 0; i < 4; ++i){
          for(int j = 0; j < 4; ++j){
            value += (localCoeff[i][j]*powf(x,i)*powf(y,j));
          }
        }
        if(value < localMin){
          localMin = value;
          localSubLoc = {x,y};
        }
      }
    }
    atomicMinFloat(&minimum, localMin);
    __syncthreads();
    if(localMin == minimum){
      if(blockId%2 == 0) matches[blockId/2].keyPoints[0].loc  = localSubLoc + matches[blockId/2].keyPoints[0].loc;
      else matches[blockId/2].keyPoints[1].loc = localSubLoc + matches[blockId/2].keyPoints[1].loc;
    }
    else return;
  }
}


//utility kernels
__global__ void ssrlcv::convertMatchToRaw(unsigned long numMatches, ssrlcv::Match* rawMatches, ssrlcv::Match* matches){
  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  if(globalID < numMatches){
    rawMatches[globalID] = Match(matches[globalID]);
  }
}
