#include "hip/hip_runtime.h"
#include "common_includes.h"
#include "Image.cuh"
#include "FeatureFactory.cuh"
#include "MatchFactory.cuh"

//TODO IO
//TODO determine image support
//TODO add versatility to image_io and use that to make Image constructors versatile

//WARNING pointer_states are as follows
//0 = NULL
//1 = __host__
//2 = __device__
//3 = both

//TODO look into use __restrict__

//TODO fix problem with feature stuff and inability to use different classes from parent feature array

int main(int argc, char *argv[]){
  try{
    //ARG PARSING
    if(argc < 2 || argc > 4){
      std::cout<<"USAGE ./bin/dsift_parallel </path/to/image/directory/> <optional:numorientations>"<<std::endl;
      exit(-1);
    }
    std::string path = argv[1];
    std::vector<std::string> imagePaths = findFiles(path);

    int numOrientations = (argc > 2) ? std::stoi(argv[2]) : 1;
    int numImages = (int) imagePaths.size();

    //CUDA INITIALIZATION
    hipInit(0);
    clock_t totalTimer = clock();

    //GET PIXEL ARRAYS & CREATE SIFT_FEATURES DENSLY
    SIFT_FeatureFactory featureFactory = SIFT_FeatureFactory(numOrientations);
    Image* images = new Image[numImages];
    MemoryState pixFeatureDescriptorMemoryState[3] = {gpu,gpu,gpu};
    for(int i = 0; i < numImages; ++i){
      images[i] = Image(imagePaths[i], i, pixFeatureDescriptorMemoryState);
      images[i].convertToBW();
      images[i].descriptor.foc = 160.0f;
      images[i].descriptor.fov = 11.0f;
      printf("%s size = %dx%d\n",imagePaths[i].c_str(), images[i].descriptor.size.x, images[i].descriptor.size.y);
      featureFactory.setImage(&(images[i]));
      featureFactory.generateFeaturesDensly();
    }
    std::cout<<"image features are set"<<std::endl;

    //camera parameters for everest254
    images[0].descriptor.cam_pos = {7.81417, 0.0f, 44.3630};
    images[0].descriptor.cam_vec = {-0.173648, 0.0f, -.984808};
    images[1].descriptor.cam_pos = {0.0f, 0.0f, 45.0f};
    images[1].descriptor.cam_vec = {0.0f, 0.0f, -1.0f};

    MatchFactory matchFactory = MatchFactory();
    SubPixelMatchSet* matchSet = NULL;
    matchFactory.generateSubPixelMatchesPairwiseConstrained(&(images[0]), &(images[1]), 10.0f, matchSet, cpu);
    delete matchSet;

    printf("\nParallel DSIFT took = %f seconds.\n\n",((float) clock() -  totalTimer)/CLOCKS_PER_SEC);

    return 0;
  }
  catch (const std::exception &e){
      std::cerr << "Caught exception: " << e.what() << '\n';
      std::exit(1);
  }
  catch (...){
      std::cerr << "Caught unknown exception\n";
      std::exit(1);
  }
}
