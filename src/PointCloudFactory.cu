#include "hip/hip_runtime.h"
#include "PointCloudFactory.cuh"

ssrlcv::PointCloudFactory::PointCloudFactory(){

}

ssrlcv::BundleSet ssrlcv::PointCloudFactory::generateBundles(MatchSet* matchSet, std::vector<ssrlcv::Image*> images){


  Unity<Bundle>* bundles = new Unity<Bundle>(nullptr,matchSet->matches->numElements,gpu);
  Unity<Bundle::Line>* lines = new Unity<Bundle::Line>(nullptr,matchSet->keyPoints->numElements,gpu);

  std::cout << "starting bundle generation ..." << std::endl;
  MemoryState origin[2] = {matchSet->matches->state,matchSet->keyPoints->state};
  if(origin[0] == cpu) matchSet->matches->transferMemoryTo(gpu);
  if(origin[1] == cpu) matchSet->keyPoints->transferMemoryTo(gpu);
  // the cameras
  size_t cam_bytes = images.size()*sizeof(ssrlcv::Image::Camera);
  // fill the cam boi
  ssrlcv::Image::Camera* h_cameras;
  h_cameras = (ssrlcv::Image::Camera*) malloc(cam_bytes);
  for(int i = 0; i < images.size(); i++){
    h_cameras[i] = images.at(i)->camera;
  }
  ssrlcv::Image::Camera* d_cameras;
  CudaSafeCall(hipMalloc(&d_cameras, cam_bytes));
  // copy the othe guy
  CudaSafeCall(hipMemcpy(d_cameras, h_cameras, cam_bytes, hipMemcpyHostToDevice));

  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(bundles->numElements,grid,block);

  //in this kernel fill lines and bundles from keyPoints and matches
  std::cout << "calling kernel ..." << std::endl;
  generateBundle<<<grid, block>>>(bundles->numElements,bundles->device, lines->device, matchSet->matches->device, matchSet->keyPoints->device, d_cameras);
  std::cout << "returned from kernel ..." << std::endl;

  hipDeviceSynchronize();
  CudaCheckError();


  // call the boi
  bundles->transferMemoryTo(cpu);
  bundles->clear(gpu);
  lines->transferMemoryTo(cpu);
  lines->clear(gpu);

  BundleSet bundleSet = {lines,bundles};

  if(origin[0] == cpu) matchSet->matches->setMemoryState(cpu);
  if(origin[1] == cpu) matchSet->keyPoints->setMemoryState(cpu);

  return bundleSet;
}

// TODO fillout
/**
* Preforms a Stereo Disparity with the correct scalar, calcualated form camera
* parameters
* @param matches0
* @param matches1
* @param points assumes this has been allocated prior to method call
* @param n the number of matches
* @param cameras a camera array of only 2 Image::Camera structs. This is used to
* dynamically calculate a scaling factor
*/
ssrlcv::Unity<float3>* ssrlcv::PointCloudFactory::stereo_disparity(Unity<Match>* matches, Image::Camera* cameras){

  float baseline = sqrtf( (cameras[0].cam_pos.x - cameras[1].cam_pos.x)*(cameras[0].cam_pos.x - cameras[1].cam_pos.x)
                        + (cameras[0].cam_pos.y - cameras[1].cam_pos.y)*(cameras[0].cam_pos.y - cameras[1].cam_pos.y)
                        + (cameras[0].cam_pos.z - cameras[1].cam_pos.z)*(cameras[0].cam_pos.z - cameras[1].cam_pos.z));
  float scale = (baseline * cameras[0].foc )/(cameras[0].dpix.x);

  std::cout << "Stereo Baseline: " << baseline << ", Stereo Scale Factor: " << scale <<  ", Inverted Stereo Scale Factor: " << (1.0/scale) << std::endl;

  MemoryState origin = matches->state;
  if(origin == cpu) matches->transferMemoryTo(gpu);

  // depth points
  float3 *points_device = nullptr;

  hipMalloc((void**) &points_device, matches->numElements*sizeof(float3));

  //
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(matches->numElements,grid,block);
  //
  // computeStereo<<<grid, block>>>(matches->numElements, matches->device, points_device, 8.0);
  computeStereo<<<grid, block>>>(matches->numElements, matches->device, points_device, scale);

  Unity<float3>* points = new Unity<float3>(points_device, matches->numElements,gpu);
  if(origin == cpu) matches->setMemoryState(cpu);

  return points;
}

// TODO fillout
/**
* Preforms a Stereo Disparity, this SHOULD NOT BE THE DEFAULT as the scale is not
* dyamically calculated
* @param matches0
* @param matches1
* @param points assumes this has been allocated prior to method call
* @param n the number of matches
* @param scale the scale factor that is multiplied
*/
ssrlcv::Unity<float3>* ssrlcv::PointCloudFactory::stereo_disparity(Unity<Match>* matches, float scale){

  MemoryState origin = matches->state;
  if(origin == cpu) matches->transferMemoryTo(gpu);

  // depth points
  float3 *points_device = nullptr;

  hipMalloc((void**) &points_device, matches->numElements*sizeof(float3));

  //
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(matches->numElements,grid,block);
  //
  computeStereo<<<grid, block>>>(matches->numElements, matches->device, points_device, scale);

  Unity<float3>* points = new Unity<float3>(points_device, matches->numElements,gpu);
  if(origin == cpu) matches->setMemoryState(cpu);

  return points;
}


// device methods


__global__ void ssrlcv::generateBundle(unsigned int numBundles, Bundle* bundles, Bundle::Line* lines, MultiMatch* matches, KeyPoint* keyPoints, Image::Camera* cameras){
  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  MultiMatch match = matches[globalID];
  float3* kp = new float3[match.numKeyPoints]();
  int end =  (int)match.numKeyPoints + match.index;
  KeyPoint currentKP = {-1,{0.0f,0.0f}};
  bundles[globalID] = {match.numKeyPoints,match.index};
  for (int i = match.index, k = 0; i < end; i++,k++){
    currentKP = keyPoints[i];
    printf("[%lu][%d] camera vec: <%f,%f,%f>\n", globalID,k, cameras[currentKP.parentId].cam_vec.x,cameras[currentKP.parentId].cam_vec.y,cameras[currentKP.parentId].cam_vec.z);
    normalize(cameras[currentKP.parentId].cam_vec);
    printf("[%lu][%d] norm camera vec: <%f,%f,%f>\n", globalID,k, cameras[currentKP.parentId].cam_vec.x,cameras[currentKP.parentId].cam_vec.y,cameras[currentKP.parentId].cam_vec.z);
    // set dpix values
    printf("[%lu][%d] dpix calc dump: (foc: %f) (fov: %f) (tanf: %f) (size: %d) \n", globalID,k, cameras[currentKP.parentId].foc, cameras[currentKP.parentId].fov, tanf(cameras[currentKP.parentId].fov / 2.0f), cameras[currentKP.parentId].size.x);
    cameras[currentKP.parentId].dpix.x = (cameras[currentKP.parentId].foc * tanf(cameras[currentKP.parentId].fov / 2.0f)) / (cameras[currentKP.parentId].size.x / 2.0f );
    cameras[currentKP.parentId].dpix.y = cameras[currentKP.parentId].dpix.x; // assume square pixel for now
    // temp
    printf("[%lu][%d] dpix calculated as: %f \n", globalID,k, cameras[currentKP.parentId].dpix.x);

    kp[k] = {
      cameras[currentKP.parentId].dpix.x * ((currentKP.loc.x) - (cameras[currentKP.parentId].size.x / 2.0f)),
      cameras[currentKP.parentId].dpix.y * ((-1.0f * currentKP.loc.y) - (cameras[currentKP.parentId].size.y / 2.0f)),
      0.0f
    }; // set the key point

    printf("[%lu][%d] kp, pre-rotation: (%f,%f,%f) \n", globalID,k, kp[k].x, kp[k].y, kp[k].z);
    kp[k] = rotatePoint(kp[k], getVectorAngles(cameras[currentKP.parentId].cam_vec));
    printf("[%lu][%d] kp, angles: (%f,%f,%f) \n", globalID,k, getVectorAngles(cameras[currentKP.parentId].cam_vec).x, getVectorAngles(cameras[currentKP.parentId].cam_vec).y, getVectorAngles(cameras[currentKP.parentId].cam_vec).z);
    printf("[%lu][%d] kp, post-rotation: (%f,%f,%f) \n", globalID,k, kp[k].x, kp[k].y, kp[k].z);
    // NOTE: will need to adjust foc with scale or x/y component here in the future
    kp[k].x = cameras[currentKP.parentId].cam_pos.x - (kp[k].x + (cameras[currentKP.parentId].cam_vec.x * cameras[currentKP.parentId].foc));
    kp[k].y = cameras[currentKP.parentId].cam_pos.y - (kp[k].y + (cameras[currentKP.parentId].cam_vec.y * cameras[currentKP.parentId].foc));
    kp[k].z = cameras[currentKP.parentId].cam_pos.z - (kp[k].z + (cameras[currentKP.parentId].cam_vec.z * cameras[currentKP.parentId].foc));
    printf("[%lu][%d] kp in R3: (%f,%f,%f)\n", globalID,k, kp[k].x, kp[k].y, kp[k].z);
    lines[i].vec = {
      cameras[currentKP.parentId].cam_pos.x - kp[k].x,
      cameras[currentKP.parentId].cam_pos.y - kp[k].y,
      cameras[currentKP.parentId].cam_pos.z - kp[k].z
    };
    normalize(lines[i].vec);
    printf("[%lu][%d] %f,%f,%f\n",globalID,k,lines[i].vec.x,lines[i].vec.y,lines[i].vec.z);
    lines[i].pnt = cameras[currentKP.parentId].cam_pos;
  }
}

__global__ void ssrlcv::computeStereo(unsigned int numMatches, Match* matches, float3* points, float scale){
  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  if (globalID < numMatches) {
    Match match = matches[globalID];
    float3 point = {match.keyPoints[0].loc.x,match.keyPoints[0].loc.y,0.0f};
    point.z = scale / sqrtf( dotProduct(match.keyPoints[0].loc-match.keyPoints[1].loc,match.keyPoints[0].loc-match.keyPoints[1].loc)) ;
    points[globalID] = point;
  }
}

__global__ void ssrlcv::two_view_reproject(int numMatches, float4* matches, float cam1C[3], float cam1V[3],float cam2C[3], float cam2V[3], float K_inv[9], float rotationTranspose1[9], float rotationTranspose2[9], float3* points){
   unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;

  if(!(globalID<numMatches))return;
	//check out globalID cheat sheet jackson gave you for this
	int matchIndex = globalID; //need to define once I calculate grid/block size
	float4 match = matches[globalID];


	float pix1[3] =
	{
		match.x, match.y, 1
	};
	float pix2[3] =
	{
		match.z, match.w, 1
  };
  float K_inv_reg[3][3];
  for(int r = 0; r < 3; ++r){
    for(int c = 0; c < 3; ++c){
      K_inv_reg[r][c] = K_inv[r*3 + c];
    }
  }
  float rotationTranspose1_reg[3][3];
   for(int r = 0; r < 3; ++r){
    for(int c = 0; c < 3; ++c){
      rotationTranspose1_reg[r][c] = rotationTranspose1[r*3 + c];
    }
  }
  float rotationTranspose2_reg[3][3];
   for(int r = 0; r < 3; ++r){
    for(int c = 0; c < 3; ++c){
      rotationTranspose2_reg[r][c] = rotationTranspose2[r*3 + c];
    }
  }

	float inter1[3];
	float inter2[3];

	float temp[3];
	multiply(K_inv_reg, pix1, temp);
	multiply(rotationTranspose1_reg, temp, inter1);
	multiply(K_inv_reg, pix2, temp);
	multiply(rotationTranspose2_reg, temp, inter2);

	float worldP1[3] =
	{
		inter1[0]+cam1C[0], inter1[1]+cam1C[1], inter1[2]+cam1C[2]
	};

	float worldP2[3] =
	{
		inter2[0]+cam2C[0], inter2[1]+cam2C[1], inter2[2]+cam2C[2]
	};

	float v1[3] =
	{
		worldP1[0] - cam1C[0], worldP1[1] - cam1C[1], worldP1[2] - cam1C[2]
	};

	float v2[3] =
	{
		worldP2[0] - cam2C[0], worldP2[1] - cam2C[1], worldP2[2] - cam2C[2]
	};

	normalize(v1);
	normalize(v2);



	//match1 and match2?
	float M1[3][3] =
	{
		{ 1-(v1[0]*v1[0]), 0-(v1[0]*v1[1]), 0-(v1[0]*v1[2]) },
		{ 0-(v1[0]*v1[1]), 1-(v1[1]*v1[1]), 0-(v1[1]*v1[2]) },
		{ 0-(v1[0]*v1[2]), 0-(v1[1]*v1[2]), 1-(v1[2]*v1[2]) }
	};

	float M2[3][3] =
	{
		{ 1-(v2[0]*v2[0]), 0-(v2[0]*v2[1]), 0-(v2[0]*v2[2]) },
		{ 0-(v2[0]*v2[1]), 1-(v2[1]*v2[1]), 0-(v2[1]*v2[2]) },
		{ 0-(v2[0]*v2[2]), 0-(v2[1]*v2[2]), 1-(v2[2]*v2[2]) }
	};

	float q1[3];
	float q2[3];
	float Q[3];

	multiply( M1, worldP1, q1);
	multiply( M2, worldP2, q2);

	float M[3][3];
	float M_inv[3][3];

	for(int r = 0; r < 3; ++r)
	{
		for(int c = 0; c < 3; ++c)
		{
			M[r][c] = M1[r][c] + M2[r][c];
		}
		Q[r] = q1[r] + q2[r];
	}

	float solution[3];
	inverse(M, M_inv);
	multiply(M_inv, Q, solution);



  	points[matchIndex].x = solution[0];
  	points[matchIndex].y = solution[1];
  	points[matchIndex].z = solution[2];

}


























































// yee
