#include "hip/hip_runtime.h"
#include "PointCloudFactory.cuh"

ssrlcv::PointCloudFactory::PointCloudFactory(){

}

/*
ssrlcv::Unity<float3>* ssrlcv::PointCloudFactory::reproject(Unity<Match>* matches, Image* target, Image* query){
  float3* pointCloud_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&pointCloud_device, matches->numElements*sizeof(float3)));
  Unity<float3>* pointCloud = new Unity<float3>(pointCloud_device,matches->numElements,gpu);

  // //initiliaze camera matrices
  // Camera cam1 = cData->cameras[0];
  // Camera cam2 = cData->cameras[1];
  // float cam1C[3] =
  // {
  //   cam1.val1, cam1.val2, cam1.val3
  // };
  // float cam1V[3] =
  // {
  //   -1*cam1.val4, -1*cam1.val5, -1*cam1.val6
  // };
  // float cam2C[3] =
  // {
  //   cam2.val1, cam2.val2, cam2.val3
  // };
  // float cam2V[3] =
  // {
  //   -1*cam2.val4, -1*cam2.val5, -1*cam2.val6
  // };
  //
  // //other matrix data needed by all threads
  // float K[3][3];  // intrinsic camera matrix
  // float K_inv[3][3];  // inverse of K
  //
  // K[0][0] = foc/dpix;
  // K[0][1] = 0;
  // K[0][2] = (float)(res/2.0);
  // K[1][0] = 0;
  // K[1][1] = foc/dpix;
  // K[1][2] = (float)(res/2.0);
  // K[2][0] = 0;
  // K[2][1] = 0;
  // K[2][2] = 1;
  // inverse3x3_cpu(K, K_inv);
  //
  // float x;
  // float y;
  // float z;
  // // Rotate cam1V about the x axis
  // x = cam1V[0];
  // y = cam1V[1];
  // z = cam1V[2];
  // float angle1;  // angle between cam1V and x axis
  // if (abs(z) < .00001)
  // {
  //   if (y > 0)
  //   {
  //     angle1 = PI/2;
  //   }
  //   else
  //   {
  //     angle1 = -1*PI/2;
  //   }
  // }
  // else
  // {
  //   angle1 = atan(y/z);
  //   if (z < 0 && y >= 0)
  //   {
  //     angle1 +=PI;
  //   }
  //   if (z < 0 && y < 0)
  //   {
  //     angle1 -= PI;
  //   }
  // }
  // float A1[3][3] =
  // {
  //   {1, 0, 0},
  //   {0, cos(angle1), -sin(angle1)},
  //   {0, sin(angle1), cos(angle1)}
  // };
  //
  // float temp[3];
  // //apply transform matrix we just got
  // multiply3x3x1_cpu(A1, cam1V, temp);
  //
  // //rotate around the y axis
  // x = temp[0];
  // y = temp[1];
  // z = temp[2];
  // float angle2;  // angle between temp and y axis
  // if (abs(z) < .00001)
  // {
  //   if (x <= 0)
  //   {
  //     angle2 = PI/2;
  //   }else
  //   {
  //     angle2 = -1*PI/2;
  //   }
  // }else
  // {
  //   angle2 = atan(-1*x / z);
  //   if(z < 0 && x < 0)
  //   {
  //     angle2 += PI;
  //   }
  //   if(z < 0 && x > 0)
  //   {
  //     angle2 -= PI;
  //   }
  // }
  //
  // float B1[3][3] =
  // {
  //   {cos(angle2), 0, sin(angle2)},
  //   {0, 1, 0},
  //   {-sin(angle2), 0, cos(angle2)}
  // };
  //
  // float rotCam1[3];
  // // apply transformation matrix B. store in rotcam1
  // multiply3x3x1_cpu(B1, temp, rotCam1);
  //
  // float rotationMatrix1[3][3];
  // float rotationTranspose1[3][3];
  //
  // //get rotation matrix as a single transform matrix
  // multiply3x3_cpu(B1, A1, rotationMatrix1);
  // transpose_cpu(rotationMatrix1, rotationTranspose1);
  // multiply3x3x1_cpu(rotationTranspose1, rotCam1, temp); // temp should be original cam1C now
  //
  // // Rotate cam2V about the x axis
  // x = cam2V[0];
  // y = cam2V[1];
  // z = cam2V[2];
  //
  // if(abs(z) < .00001)
  // {
  //   if(y > 0)
  //   {
  //     angle1 = PI/2;
  //   } else
  //   {
  //     angle1 = -1*PI/2;
  //   }
  // } else
  // {
  //   angle1 = atan(y / z);
  //   if(z<0 && y>=0)
  //   {
  //     angle1 += PI;
  //   }
  //   if(z<0 && y<0)
  //   {
  //     angle1 -= PI;
  //   }
  // }
  // float A2[3][3] =
  // {
  //   {1, 0, 0},
  //   {0, cos(angle1), -sin(angle1)},
  //   {0, sin(angle1), cos(angle1)}
  // };
  // // apply transformation matrix A
  // multiply3x3x1_cpu(A2, cam2V, temp);
  //
  // // Rotate about the y axis
  // x = temp[0];
  // y = temp[1];
  // z = temp[2];
  // if(abs(z) < .00001)
  // {
  //   if(x <= 0){
  //     angle2 = PI/2;
  //   }else
  //   {
  //     angle2 = -1*PI/2;
  //   }
  // } else
  // {
  //   angle2 = atan(-1*x / z);
  //   if(z<0 && x<0)
  //   {
  //     angle2 += PI;
  //   }
  //   if(z<0 && x>0)
  //   {
  //     angle2 -= PI;
  //   }
  // }
  // float B2[3][3] =
  // {
  //   {cos(angle2), 0, sin(angle2)},
  //   {0, 1, 0},
  //   {-sin(angle2), 0, cos(angle2)}
  // };
  // // apply transformation matrix B
  // float rotCam2[3];
  // multiply3x3x1_cpu(B2, temp, rotCam2);
  //
  // float rotationMatrix2[3][3];
  // float rotationTranspose2[3][3];
  //
  // // Get rotation matrix as a single transformation matrix
  // multiply3x3_cpu(B2, A2, rotationMatrix2);
  // transpose_cpu(rotationMatrix2, rotationTranspose2);
  // multiply3x3x1_cpu(rotationTranspose2, rotCam2, temp); // temp should be original cam2C now
  //
  // //linearize matrices
  // //position in linear matrix = 3*x +y, [x][y]
  // float K_inv_lin[9];
  // K_inv_lin[0] = K_inv[0][0];
  // K_inv_lin[1] = K_inv[0][1];
  // K_inv_lin[2] = K_inv[0][2];
  // K_inv_lin[3] = K_inv[1][0];
  // K_inv_lin[4] = K_inv[1][1];
  // K_inv_lin[5] = K_inv[1][2];
  // K_inv_lin[6] = K_inv[2][0];
  // K_inv_lin[7] = K_inv[2][1];
  // K_inv_lin[8] = K_inv[2][2];
  //
  // float rotTran1_lin[9];
  // rotTran1_lin[0] = rotationTranspose1[0][0];
  // rotTran1_lin[1] = rotationTranspose1[0][1];
  // rotTran1_lin[2] = rotationTranspose1[0][2];
  // rotTran1_lin[3] = rotationTranspose1[1][0];
  // rotTran1_lin[4] = rotationTranspose1[1][1];
  // rotTran1_lin[5] = rotationTranspose1[1][2];
  // rotTran1_lin[6] = rotationTranspose1[2][0];
  // rotTran1_lin[7] = rotationTranspose1[2][1];
  // rotTran1_lin[8] = rotationTranspose1[2][2];
  //
  // float rotTran2_lin[9];
  // rotTran2_lin[0] = rotationTranspose2[0][0];
  // rotTran2_lin[1] = rotationTranspose2[0][1];
  // rotTran2_lin[2] = rotationTranspose2[0][2];
  // rotTran2_lin[3] = rotationTranspose2[1][0];
  // rotTran2_lin[4] = rotationTranspose2[1][1];
  // rotTran2_lin[5] = rotationTranspose2[1][2];
  // rotTran2_lin[6] = rotationTranspose2[2][0];
  // rotTran2_lin[7] = rotationTranspose2[2][1];
  // rotTran2_lin[8] = rotationTranspose2[2][2];
  //
  // //initialize point cloud data to 0
  // //pointCloud->points = new  float3[POINT_CLOUD_SIZE];
  // float3* currentPoint;
  // for(int i = 0; i < POINT_CLOUD_SIZE; ++i)
  // {
  //   currentPoint = &(pointCloud->points[i]);
  //   currentPoint->x = 0.0f;
  //   currentPoint->y = 0.0f;
  //   currentPoint->z = 0.0f;
  // }
  //
  // //create pointers on the device. d_ indicates pointer to mem on device
  // float4* d_in_matches; 		 //where feature matches data is stored
  // float* d_in_cam1C; 		 //where camera data is stored
  // float* d_in_cam1V; 		 //where camera data is stored
  // float* d_in_cam2C; 		 //where camera data is stored
  // float* d_in_cam2V; 		 //where camera data is stored
  // float* d_in_k_inv;
  // float* d_in_rotTran1;
  //
  // float* d_in_rotTran2;
  // float3* d_out_pointCloud; //where point cloud output is stored
  //
  // //allocate the mem on the gpu
  // hipMalloc((void**) &d_in_matches, FEATURE_DATA_BYTES);
  // hipMalloc((void**) &d_in_cam1C, CAMERA_DATA_BYTES);
  // hipMalloc((void**) &d_in_cam1V, CAMERA_DATA_BYTES);
  // hipMalloc((void**) &d_in_cam2C, CAMERA_DATA_BYTES);
  // hipMalloc((void**) &d_in_cam2V, CAMERA_DATA_BYTES);
  // hipMalloc((void**) &d_in_k_inv, MATRIX_DAYA_BYTES);
  // hipMalloc((void**) &d_in_rotTran1, MATRIX_DAYA_BYTES);
  // hipMalloc((void**) &d_in_rotTran2, MATRIX_DAYA_BYTES);
  // hipMalloc((void**) &d_out_pointCloud, POINT_CLOUD_BYTES);
  //
  // //transfer input data to mem on the gpu
  // hipMemcpy(d_in_matches, fMatches->matches, FEATURE_DATA_BYTES, hipMemcpyHostToDevice);
  // hipMemcpy(d_in_cam1C, cam1C, CAMERA_DATA_BYTES, hipMemcpyHostToDevice);
  // hipMemcpy(d_in_cam1V, cam1V, CAMERA_DATA_BYTES, hipMemcpyHostToDevice);
  // hipMemcpy(d_in_cam2C, cam2C, CAMERA_DATA_BYTES, hipMemcpyHostToDevice);
  // hipMemcpy(d_in_cam2V, cam2V, CAMERA_DATA_BYTES, hipMemcpyHostToDevice);
  // hipMemcpy(d_in_k_inv, K_inv_lin, MATRIX_DAYA_BYTES, hipMemcpyHostToDevice);
  // hipMemcpy(d_in_rotTran1, rotTran1_lin, MATRIX_DAYA_BYTES, hipMemcpyHostToDevice);
  // hipMemcpy(d_in_rotTran2, rotTran2_lin, MATRIX_DAYA_BYTES, hipMemcpyHostToDevice);
  // hipMemcpy(d_out_pointCloud, pointCloud->points, POINT_CLOUD_BYTES, hipMemcpyHostToDevice);
  //
  // //block and thread count
  // dim3 THREAD_COUNT = {512, 1, 1};
  // dim3 BLOCK_COUNT = {(unsigned int)ceil((POINT_CLOUD_SIZE+512)/512),1, 1}; //(unsigned int)ceil(POINT_CLOUD_SIZE/512)
  //
  // //call kernel
  // two_view_reproject<<<BLOCK_COUNT, THREAD_COUNT>>>(POINT_CLOUD_SIZE, d_in_matches, d_in_cam1C, d_in_cam1V, d_in_cam2C, d_in_cam2V, d_in_k_inv, d_in_rotTran1, d_in_rotTran2, d_out_pointCloud);
  //
  // //error check
  // CudaCheckError();
  //
  // //get result
  // hipMemcpy(pointCloud->points, d_out_pointCloud, POINT_CLOUD_BYTES, hipMemcpyDeviceToHost);
  //
  // pointCloud->numPoints = fMatches->numMatches;
  // //free mem on gpu
  // hipFree(d_in_matches);
  // hipFree(d_in_cam1C);
  // hipFree(d_in_cam1V);
  // hipFree(d_in_cam2C);
  // hipFree(d_in_cam2V);
  // hipFree(d_out_pointCloud);
  return pointCloud;
}
*/

ssrlcv::BundleSet ssrlcv::PointCloudFactory::generateBundles(MatchSet* matchSet, std::vector<ssrlcv::Image*> images){


  Unity<Bundle>* bundles = new Unity<Bundle>(nullptr,matchSet->matches->numElements,gpu);
  Unity<Bundle::Line>* lines = new Unity<Bundle::Line>(nullptr,matchSet->keyPoints->numElements,gpu);

  std::cout << "starting bundle generation ..." << std::endl;
  MemoryState origin[2] = {matchSet->matches->state,matchSet->keyPoints->state};
  if(origin[0] == cpu) matchSet->matches->transferMemoryTo(gpu);
  if(origin[1] == cpu) matchSet->keyPoints->transferMemoryTo(gpu);
  // the cameras
  size_t cam_bytes = images.size()*sizeof(ssrlcv::Image::Camera);
  // fill the cam boi
  ssrlcv::Image::Camera* h_cameras;
  h_cameras = (ssrlcv::Image::Camera*) malloc(cam_bytes);
  for(int i = 0; i < images.size(); i++){
    h_cameras[i] = images.at(i)->camera;
  }
  ssrlcv::Image::Camera* d_cameras;
  CudaSafeCall(hipMalloc(&d_cameras, cam_bytes));
  // copy the othe guy
  CudaSafeCall(hipMemcpy(d_cameras, h_cameras, cam_bytes, hipMemcpyHostToDevice));

  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(bundles->numElements,grid,block);

  //in this kernel fill lines and bundles from keyPoints and matches
  std::cout << "calling kernel ..." << std::endl;
  generateBundle<<<grid, block>>>(bundles->numElements,bundles->device, lines->device, matchSet->matches->device, matchSet->keyPoints->device, d_cameras);
  std::cout << "returned from kernel ..." << std::endl;

  hipDeviceSynchronize();
  CudaCheckError();


  // call the boi
  bundles->transferMemoryTo(cpu);
  bundles->clear(gpu);
  lines->transferMemoryTo(cpu);
  lines->clear(gpu);

  BundleSet bundleSet = {lines,bundles};

  if(origin[0] == cpu) matchSet->matches->setMemoryState(cpu);
  if(origin[1] == cpu) matchSet->keyPoints->setMemoryState(cpu);

  return bundleSet;
}


// TODO fillout
/**
* Preforms a Stereo Disparity
* @param matches0
* @param matches1
* @param points assumes this has been allocated prior to method call
* @param n the number of matches
* @param scale the scale factor that is multiplied
*/
ssrlcv::Unity<float3>* ssrlcv::PointCloudFactory::stereo_disparity(Unity<Match>* matches, float scale){

  MemoryState origin = matches->state;
  if(origin == cpu) matches->transferMemoryTo(gpu);

  // depth points
  float3 *points_device = nullptr;

  hipMalloc((void**) &points_device, matches->numElements*sizeof(float3));

  //
  int blockSize = 1024;
  int gridSize = (int) ceil((float) matches->numElements / blockSize);

  //
  computeStereo<<<gridSize, blockSize>>>(matches->numElements, matches->device, points_device, scale);

  Unity<float3>* points = new Unity<float3>(points_device, matches->numElements,gpu);
  if(origin == cpu) matches->setMemoryState(cpu);

  return points;
}


// device methods


__global__ void ssrlcv::generateBundle(unsigned int numBundles, Bundle* bundles, Bundle::Line* lines, MultiMatch* matches, KeyPoint* keyPoints, Image::Camera* cameras){
  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  MultiMatch match = matches[globalID];
  float3* kp = new float3[match.numKeyPoints]();
  int end =  (int)match.numKeyPoints + match.index;
  KeyPoint currentKP = {-1,{0.0f,0.0f}};
  bundles[globalID] = {match.numKeyPoints,match.index};
  for (int i = match.index, k = 0; i < end; i++,k++){
    currentKP = keyPoints[i];
    printf("[%lu][%d] camera vec: <%f,%f,%f>\n", globalID,k, cameras[currentKP.parentId].cam_vec.x,cameras[currentKP.parentId].cam_vec.y,cameras[currentKP.parentId].cam_vec.z);
    cameras[currentKP.parentId].cam_vec = normalizeVector(cameras[currentKP.parentId].cam_vec);
    printf("[%lu][%d] norm camera vec: <%f,%f,%f>\n", globalID,k, cameras[currentKP.parentId].cam_vec.x,cameras[currentKP.parentId].cam_vec.y,cameras[currentKP.parentId].cam_vec.z);
    // set dpix values
    printf("[%lu][%d] dpix calc dump: (foc: %f) (fov: %f) (tanf: %f) (size: %d) \n", globalID,k, cameras[currentKP.parentId].foc, cameras[currentKP.parentId].fov, tanf(cameras[currentKP.parentId].fov / 2.0f), cameras[currentKP.parentId].size.x);
    cameras[currentKP.parentId].dpix.x = (cameras[currentKP.parentId].foc * tanf(cameras[currentKP.parentId].fov / 2.0f)) / (cameras[currentKP.parentId].size.x / 2.0f );
    cameras[currentKP.parentId].dpix.y = cameras[currentKP.parentId].dpix.x; // assume square pixel for now
    // temp
    printf("[%lu][%d] dpix calculated as: %f \n", globalID,k, cameras[currentKP.parentId].dpix.x);

    kp[k] = {
      cameras[currentKP.parentId].dpix.x * ((currentKP.loc.x) - (cameras[currentKP.parentId].size.x / 2.0f)),
      cameras[currentKP.parentId].dpix.y * ((-1.0f * currentKP.loc.y) - (cameras[currentKP.parentId].size.y / 2.0f)),
      0.0f
    }; // set the key point

    printf("[%lu][%d] kp, pre-rotation: (%f,%f,%f) \n", globalID,k, kp[k].x, kp[k].y, kp[k].z);
    kp[k] = rotatePoint(kp[k], getVectorAngles(cameras[currentKP.parentId].cam_vec));
    printf("[%lu][%d] kp, angles: (%f,%f,%f) \n", globalID,k, getVectorAngles(cameras[currentKP.parentId].cam_vec).x, getVectorAngles(cameras[currentKP.parentId].cam_vec).y, getVectorAngles(cameras[currentKP.parentId].cam_vec).z);
    printf("[%lu][%d] kp, post-rotation: (%f,%f,%f) \n", globalID,k, kp[k].x, kp[k].y, kp[k].z);
    // NOTE: will need to adjust foc with scale or x/y component here in the future
    kp[k].x = cameras[currentKP.parentId].cam_pos.x - (kp[k].x + (cameras[currentKP.parentId].cam_vec.x * cameras[currentKP.parentId].foc));
    kp[k].y = cameras[currentKP.parentId].cam_pos.y - (kp[k].y + (cameras[currentKP.parentId].cam_vec.y * cameras[currentKP.parentId].foc));
    kp[k].z = cameras[currentKP.parentId].cam_pos.z - (kp[k].z + (cameras[currentKP.parentId].cam_vec.z * cameras[currentKP.parentId].foc));
    printf("[%lu][%d] kp in R3: (%f,%f,%f)\n", globalID,k, kp[k].x, kp[k].y, kp[k].z);
    lines[i].vec = {
      cameras[currentKP.parentId].cam_pos.x - kp[k].x,
      cameras[currentKP.parentId].cam_pos.y - kp[k].y,
      cameras[currentKP.parentId].cam_pos.z - kp[k].z
    };
    lines[i].vec = normalizeVector(lines[i].vec);
    printf("[%lu][%d] %f,%f,%f\n",globalID,k,lines[i].vec.x,lines[i].vec.y,lines[i].vec.z);
    lines[i].pnt = cameras[currentKP.parentId].cam_pos;
  }
}

__global__ void ssrlcv::computeStereo(unsigned int numMatches, Match* matches, float3* points, float scale){
  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  if (globalID < numMatches) {
    Match match = matches[globalID];
    float3 point = {match.keyPoints[0].loc.x,match.keyPoints[0].loc.y,0.0f};
    point.z = sqrtf(scale*dotProduct({match.keyPoints[0].loc-match.keyPoints[1].loc},{match.keyPoints[0].loc-match.keyPoints[1].loc}));
    points[globalID] = point;
  }
}

__global__ void ssrlcv::two_view_reproject(int numMatches, float4* matches, float cam1C[3], float cam1V[3],float cam2C[3], float cam2V[3], float K_inv[9], float rotationTranspose1[9], float rotationTranspose2[9], float3* points){

  if(!(getGlobalIdx_1D_1D()<numMatches))return;
	//check out globalID cheat sheet jackson gave you for this
	int matchIndex = getGlobalIdx_1D_1D(); //need to define once I calculate grid/block size
	//printf("thread index %d", getGlobalIdx_1D_1D());
	float4 match = matches[getGlobalIdx_1D_1D()];


	float pix1[3] =
	{
		match.x, match.y, 1
	};
	float pix2[3] =
	{
		match.z, match.w, 1
	};


	float inter1[3];
	float inter2[3];

	float temp[3];
	multiply3x3x1_gpu(K_inv, pix1, temp);
	multiply3x3x1_gpu(rotationTranspose1, temp, inter1);
	multiply3x3x1_gpu(K_inv, pix2, temp);
	multiply3x3x1_gpu(rotationTranspose2, temp, inter2);

	float worldP1[3] =
	{
		inter1[0]+cam1C[0], inter1[1]+cam1C[1], inter1[2]+cam1C[2]
	};

	float worldP2[3] =
	{
		inter2[0]+cam2C[0], inter2[1]+cam2C[1], inter2[2]+cam2C[2]
	};

	float v1[3] =
	{
		worldP1[0] - cam1C[0], worldP1[1] - cam1C[1], worldP1[2] - cam1C[2]
	};

	float v2[3] =
	{
		worldP2[0] - cam2C[0], worldP2[1] - cam2C[1], worldP2[2] - cam2C[2]
	};

	normalize_gpu(v1);
	normalize_gpu(v2);



	//match1 and match2?
	float M1[3][3] =
	{
		{ 1-(v1[0]*v1[0]), 0-(v1[0]*v1[1]), 0-(v1[0]*v1[2]) },
		{ 0-(v1[0]*v1[1]), 1-(v1[1]*v1[1]), 0-(v1[1]*v1[2]) },
		{ 0-(v1[0]*v1[2]), 0-(v1[1]*v1[2]), 1-(v1[2]*v1[2]) }
	};

	float M2[3][3] =
	{
		{ 1-(v2[0]*v2[0]), 0-(v2[0]*v2[1]), 0-(v2[0]*v2[2]) },
		{ 0-(v2[0]*v2[1]), 1-(v2[1]*v2[1]), 0-(v2[1]*v2[2]) },
		{ 0-(v2[0]*v2[2]), 0-(v2[1]*v2[2]), 1-(v2[2]*v2[2]) }
	};

	float q1[3];
	float q2[3];
	float Q[3];

	multiply3x3x1_gpu( M1, worldP1, q1);
	multiply3x3x1_gpu( M2, worldP2, q2);

	float M[3][3];
	float M_inv[3][3];

	for(int r = 0; r < 3; ++r)
	{
		for(int c = 0; c < 3; ++c)
		{
			M[r][c] = M1[r][c] + M2[r][c];
		}
		Q[r] = q1[r] + q2[r];
	}

	float solution[3];
	inverse3x3_gpu(M, M_inv);
	multiply3x3x1_gpu(M_inv, Q, solution);



  	points[matchIndex].x = solution[0];
  	points[matchIndex].y = solution[1];
  	points[matchIndex].z = solution[2];

}


























































// yee
