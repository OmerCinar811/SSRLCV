#include "hip/hip_runtime.h"
#include "PointCloudFactory.cuh"

ssrlcv::PointCloudFactory::PointCloudFactory(){

}

ssrlcv::BundleSet ssrlcv::PointCloudFactory::generateBundles(MatchSet* matchSet, std::vector<ssrlcv::Image*> images){


  Unity<Bundle>* bundles = new Unity<Bundle>(nullptr,matchSet->matches->numElements,gpu);
  Unity<Bundle::Line>* lines = new Unity<Bundle::Line>(nullptr,matchSet->keyPoints->numElements,gpu);

  std::cout << "starting bundle generation ..." << std::endl;
  MemoryState origin[2] = {matchSet->matches->state,matchSet->keyPoints->state};
  if(origin[0] == cpu) matchSet->matches->transferMemoryTo(gpu);
  if(origin[1] == cpu) matchSet->keyPoints->transferMemoryTo(gpu);
  // the cameras
  size_t cam_bytes = images.size()*sizeof(ssrlcv::Image::Camera);
  // fill the cam boi
  ssrlcv::Image::Camera* h_cameras;
  h_cameras = (ssrlcv::Image::Camera*) malloc(cam_bytes);
  for(int i = 0; i < images.size(); i++){
    h_cameras[i] = images.at(i)->camera;
  }
  ssrlcv::Image::Camera* d_cameras;
  CudaSafeCall(hipMalloc(&d_cameras, cam_bytes));
  // copy the othe guy
  CudaSafeCall(hipMemcpy(d_cameras, h_cameras, cam_bytes, hipMemcpyHostToDevice));

  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(bundles->numElements,grid,block);

  //in this kernel fill lines and bundles from keyPoints and matches
  std::cout << "calling kernel ..." << std::endl;
  generateBundle<<<grid, block>>>(bundles->numElements,bundles->device, lines->device, matchSet->matches->device, matchSet->keyPoints->device, d_cameras);
  std::cout << "returned from kernel ..." << std::endl;

  hipDeviceSynchronize();
  CudaCheckError();


  // call the boi
  bundles->transferMemoryTo(cpu);
  bundles->clear(gpu);
  lines->transferMemoryTo(cpu);
  lines->clear(gpu);

  BundleSet bundleSet = {lines,bundles};

  if(origin[0] == cpu) matchSet->matches->setMemoryState(cpu);
  if(origin[1] == cpu) matchSet->keyPoints->setMemoryState(cpu);

  return bundleSet;
}

/**
* Preforms a Stereo Disparity with the correct scalar, calcualated form camera
* parameters
* @param matches0
* @param matches1
* @param points assumes this has been allocated prior to method call
* @param n the number of matches
* @param cameras a camera array of only 2 Image::Camera structs. This is used to
* dynamically calculate a scaling factor
*/
ssrlcv::Unity<float3>* ssrlcv::PointCloudFactory::stereo_disparity(Unity<Match>* matches, Image::Camera* cameras){

  float baseline = sqrtf( (cameras[0].cam_pos.x - cameras[1].cam_pos.x)*(cameras[0].cam_pos.x - cameras[1].cam_pos.x)
                        + (cameras[0].cam_pos.y - cameras[1].cam_pos.y)*(cameras[0].cam_pos.y - cameras[1].cam_pos.y)
                        + (cameras[0].cam_pos.z - cameras[1].cam_pos.z)*(cameras[0].cam_pos.z - cameras[1].cam_pos.z));
  float scale = (baseline * cameras[0].foc )/(cameras[0].dpix.x);

  std::cout << "Stereo Baseline: " << baseline << ", Stereo Scale Factor: " << scale <<  ", Inverted Stereo Scale Factor: " << (1.0/scale) << std::endl;

  MemoryState origin = matches->state;
  if(origin == cpu) matches->transferMemoryTo(gpu);

  // depth points
  float3 *points_device = nullptr;

  hipMalloc((void**) &points_device, matches->numElements*sizeof(float3));

  //
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(matches->numElements,grid,block);
  //
  computeStereo<<<grid, block>>>(matches->numElements, matches->device, points_device, 8.0);
  // focal lenth / baseline

  // computeStereo<<<grid, block>>>(matches->numElements, matches->device, points_device, 64.0);

  Unity<float3>* points = new Unity<float3>(points_device, matches->numElements,gpu);
  if(origin == cpu) matches->setMemoryState(cpu);

  return points;
}

/**
* Preforms a Stereo Disparity, this SHOULD NOT BE THE DEFAULT as the scale is not
* dyamically calculated
* @param matches0
* @param matches1
* @param points assumes this has been allocated prior to method call
* @param n the number of matches
* @param scale the scale factor that is multiplied
*/
ssrlcv::Unity<float3>* ssrlcv::PointCloudFactory::stereo_disparity(Unity<Match>* matches, float scale){

  MemoryState origin = matches->state;
  if(origin == cpu) matches->transferMemoryTo(gpu);

  // depth points
  float3 *points_device = nullptr;

  hipMalloc((void**) &points_device, matches->numElements*sizeof(float3));

  //
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(matches->numElements,grid,block);
  //
  computeStereo<<<grid, block>>>(matches->numElements, matches->device, points_device, scale);

  Unity<float3>* points = new Unity<float3>(points_device, matches->numElements,gpu);
  if(origin == cpu) matches->setMemoryState(cpu);

  return points;
}

ssrlcv::Unity<float3>* ssrlcv::PointCloudFactory::stereo_disparity(Unity<Match>* matches, float foc, float baseline, float doffset){

  MemoryState origin = matches->state;
  if(origin == cpu) matches->transferMemoryTo(gpu);


  Unity<float3>* points = new Unity<float3>(nullptr, matches->numElements,gpu);
  //
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  getFlatGridBlock(matches->numElements,grid,block);
  //
  computeStereo<<<grid, block>>>(matches->numElements, matches->device, points->device, foc, baseline, doffset);

  if(origin == cpu) matches->setMemoryState(cpu);

  return points;
}

uchar3 ssrlcv::heatMap(float value){
  uchar3 rgb;
  // float3 colorMap[5] = {
  //   {255.0f,0,0},
  //   {127.5f,127.5f,0},
  //   {0,255.0f,0.0f},
  //   {0,127.7f,127.5f},
  //   {0,0,255.0f},
  // };
  // float temp = colors->host[i];
  // colors->host[i] *= 5.0f;
  // colors->host[i] = floor(colors->host[i]);
  // if(colors->host[i] == 5.0f) colors->host[i] = 4.0f;
  // if(colors->host[i] == 0.0f) colors->host[i] = 1.0f;
  // rgb.x = (1-temp)*colorMap[(int)colors->host[i]-1].x + (temp*colorMap[(int)colors->host[i]].x);
  // rgb.y = (1-temp)*colorMap[(int)colors->host[i]-1].y + (temp*colorMap[(int)colors->host[i]].y);
  // rgb.z = (1-temp)*colorMap[(int)colors->host[i]-1].z + (temp*colorMap[(int)colors->host[i]].z);


  if(value <= 0.5f){
    value *= 2.0f;
    rgb.x = (unsigned char) 255*(1-value) + 0.5;
    rgb.y = (unsigned char) 255*value + 0.5;
    rgb.z = 0;
  }
  else{
    value = value*2.0f - 1;
    rgb.x = 0;
    rgb.y = (unsigned char) 255*(1-value) + 0.5;
    rgb.z = (unsigned char) 255*value + 0.5;
  }
  return rgb;
}

void ssrlcv::writeDisparityImage(Unity<float3>* points, unsigned int interpolationRadius, std::string pathToFile){
  MemoryState origin = points->state;
  if(origin == gpu) points->transferMemoryTo(cpu);
  float3 min = {FLT_MAX,FLT_MAX,FLT_MAX};
  float3 max = {-FLT_MAX,-FLT_MAX,-FLT_MAX};
  for(int i = 0; i < points->numElements; ++i){
    if(points->host[i].x < min.x) min.x = points->host[i].x;
    if(points->host[i].x > max.x) max.x = points->host[i].x;
    if(points->host[i].y < min.y) min.y = points->host[i].y;
    if(points->host[i].y > max.y) max.y = points->host[i].y;
    if(points->host[i].z < min.z) min.z = points->host[i].z;
    if(points->host[i].z > max.z) max.z = points->host[i].z;
  }
  uint2 imageDim = {(unsigned int)ceil(max.x-min.x)+1,(unsigned int)ceil(max.y-min.y)+1};
  unsigned char* disparityImage = new unsigned char[imageDim.x*imageDim.y*3];
  Unity<float>* colors = new Unity<float>(nullptr,imageDim.x*imageDim.y,cpu);
  for(int i = 0; i < imageDim.x*imageDim.y*3; ++i){
    disparityImage[i] = 0;
  }
  for(int i = 0; i < points->numElements; ++i){
    float3 temp = points->host[i] - min;
    if(ceil(temp.x) != temp.x || ceil(temp.y) != temp.y){
      colors->host[((int)ceil(temp.y)*imageDim.x) + (int)ceil(temp.x)] += (1-ceil(temp.x)-temp.x)*(1-ceil(temp.y)-temp.y)*temp.z/(max.z-min.z);
      colors->host[((int)ceil(temp.y)*imageDim.x) + (int)floor(temp.x)] += (1-temp.x-floor(temp.x))*(1-ceil(temp.y)-temp.y)*temp.z/(max.z-min.z);
      colors->host[((int)floor(temp.y)*imageDim.x) + (int)ceil(temp.x)] += (1-ceil(temp.x)-temp.x)*(1-temp.y-floor(temp.y))*temp.z/(max.z-min.z);
      colors->host[((int)floor(temp.y)*imageDim.x) + (int)floor(temp.x)] += (1-temp.x-floor(temp.x))*(1-temp.y-floor(temp.y))*temp.z/(max.z-min.z);
    }
    else{
      colors->host[(int)temp.y*imageDim.x + (int)temp.x] += temp.z/(max.z-min.z);
    }
  }

  /*
  INTERPOLATE
  */
  if(interpolationRadius != 0){
    colors->setMemoryState(gpu);
    float* interpolated = nullptr;
    CudaSafeCall(hipMalloc((void**)&interpolated,imageDim.x*imageDim.y*sizeof(float)));
    dim3 block = {1,1,1};
    dim3 grid = {1,1,1};
    getFlatGridBlock(imageDim.x*imageDim.y,grid,block);
    interpolateDepth<<<grid,block>>>(imageDim,interpolationRadius,colors->device,interpolated);
    hipDeviceSynchronize();
    CudaCheckError();
    colors->setData(interpolated,colors->numElements,gpu);
    colors->setMemoryState(cpu);
  }

  min.z = FLT_MAX;
  max.z = -FLT_MAX;
  for(int i = 0; i < imageDim.x*imageDim.y; ++i){
    if(min.z > colors->host[i]) min.z = colors->host[i];
    if(max.z < colors->host[i]) max.z = colors->host[i];
  }


  uchar3 rgb;
  for(int i = 0; i < imageDim.x*imageDim.y; ++i){
    colors->host[i] -= min.z;
    colors->host[i] /= (max.z-min.z);
    rgb = heatMap(colors->host[i]);
    disparityImage[i*3] = rgb.x;
    disparityImage[i*3 + 1] = rgb.y;
    disparityImage[i*3 + 2] = rgb.z;
  }
  delete colors;
  writePNG(pathToFile.c_str(),disparityImage,3,imageDim.x,imageDim.y);
  delete disparityImage;
}


// device methods


__global__ void ssrlcv::generateBundle(unsigned int numBundles, Bundle* bundles, Bundle::Line* lines, MultiMatch* matches, KeyPoint* keyPoints, Image::Camera* cameras){
  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  MultiMatch match = matches[globalID];
  float3* kp = new float3[match.numKeyPoints]();
  int end =  (int)match.numKeyPoints + match.index;
  KeyPoint currentKP = {-1,{0.0f,0.0f}};
  bundles[globalID] = {match.numKeyPoints,match.index};
  for (int i = match.index, k = 0; i < end; i++,k++){
    currentKP = keyPoints[i];
    printf("[%lu][%d] camera vec: <%f,%f,%f>\n", globalID,k, cameras[currentKP.parentId].cam_vec.x,cameras[currentKP.parentId].cam_vec.y,cameras[currentKP.parentId].cam_vec.z);
    // may not be needed w new method??
    normalize(cameras[currentKP.parentId].cam_vec);
    printf("[%lu][%d] norm camera vec: <%f,%f,%f>\n", globalID,k, cameras[currentKP.parentId].cam_vec.x,cameras[currentKP.parentId].cam_vec.y,cameras[currentKP.parentId].cam_vec.z);
    printf("[%lu][%d] camera axangle: %f \n",globalID,k,cameras[currentKP.parentId].axangle);
    // set dpix values
    printf("[%lu][%d] dpix calc dump: (foc: %f) (fov: %f) (tanf: %f) (size: %d) \n", globalID,k, cameras[currentKP.parentId].foc, cameras[currentKP.parentId].fov, tanf(cameras[currentKP.parentId].fov / 2.0f), cameras[currentKP.parentId].size.x);
    cameras[currentKP.parentId].dpix.x = (cameras[currentKP.parentId].foc * tanf(cameras[currentKP.parentId].fov / 2.0f)) / (cameras[currentKP.parentId].size.x / 2.0f );
    cameras[currentKP.parentId].dpix.y = cameras[currentKP.parentId].dpix.x; // assume square pixel for now
    // temp
    printf("[%lu][%d] dpix calculated as: %f \n", globalID,k, cameras[currentKP.parentId].dpix.x);

    // here we imagine the image plane is in the X Y plane AT a particular Z value, which is the focal length
    // We need to slowly transform this later so that it has the correct orientation
    kp[k] = {
      // NOTE: This is when dpix was supposed to be used, but does dpix just end up adding more
      // floating point errors? instead it might still be best to live within the image space
      // cameras[currentKP.parentId].dpix.x * ((currentKP.loc.x) - (cameras[currentKP.parentId].size.x / 2.0f)),
      // cameras[currentKP.parentId].dpix.y * ((currentKP.loc.y) - (cameras[currentKP.parentId].size.y / 2.0f)),
      ((currentKP.loc.x) - (cameras[currentKP.parentId].size.x / 2.0f)),
      ((currentKP.loc.y) - (cameras[currentKP.parentId].size.y / 2.0f)),
      cameras[currentKP.parentId].foc // this is the focal length
    }; // set the key point

    printf("[%lu][%d] kp, pre-rotation: (%f,%f,%f) \n", globalID,k, kp[k].x, kp[k].y, kp[k].z);

    // old thing
    // kp[k] = rotatePoint(kp[k], getVectorAngles(cameras[currentKP.parentId].cam_vec));

    // attempting new thing
    // kp[k] = rotatePointKP(kp[k], cameras[currentKP.parentId].cam_vec, cameras[currentKP.parentId].axangle);
    kp[k] = rotatePoint(kp[k], cameras[currentKP.parentId].cam_vec);
    printf("[%lu][%d] kp, post-rotation: (%f,%f,%f) \n", globalID,k, kp[k].x, kp[k].y, kp[k].z);

    kp[k].x = cameras[currentKP.parentId].cam_pos.x - (kp[k].x);
    kp[k].y = cameras[currentKP.parentId].cam_pos.y - (kp[k].y);
    kp[k].z = cameras[currentKP.parentId].cam_pos.z - (kp[k].z);
    printf("[%lu][%d] kp in R3: (%f,%f,%f)\n", globalID,k, kp[k].x, kp[k].y, kp[k].z);
    lines[i].vec = {
      cameras[currentKP.parentId].cam_pos.x - kp[k].x,
      cameras[currentKP.parentId].cam_pos.y - kp[k].y,
      cameras[currentKP.parentId].cam_pos.z - kp[k].z
    };
    normalize(lines[i].vec);
    printf("[%lu][%d] %f,%f,%f\n",globalID,k,lines[i].vec.x,lines[i].vec.y,lines[i].vec.z);
    lines[i].pnt = cameras[currentKP.parentId].cam_pos;
  }
}

__global__ void ssrlcv::computeStereo(unsigned int numMatches, Match* matches, float3* points, float scale){
  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  if (globalID < numMatches) {
    Match match = matches[globalID];
    float3 point = {match.keyPoints[0].loc.x,match.keyPoints[0].loc.y,0.0f};
    point.z = scale / sqrtf( dotProduct(match.keyPoints[0].loc-match.keyPoints[1].loc,match.keyPoints[0].loc-match.keyPoints[1].loc)) ;
    points[globalID] = point;
  }
}

__global__ void ssrlcv::computeStereo(unsigned int numMatches, Match* matches, float3* points, float foc, float baseLine, float doffset){
  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  if (globalID < numMatches) {
    Match match = matches[globalID];
    float3 point = {match.keyPoints[1].loc.x,match.keyPoints[1].loc.y,0.0f};
    //point.z = sqrtf(dotProduct(match.keyPoints[1].loc-match.keyPoints[0].loc,match.keyPoints[1].loc-match.keyPoints[0].loc));
    //with non parrallel or nonrecitified then replace .x - .x below with above
    point.z = foc*baseLine/(match.keyPoints[0].loc.x-match.keyPoints[1].loc.x+doffset);
    points[globalID] = point;
  }
}

__global__ void ssrlcv::interpolateDepth(uint2 disparityMapSize, int influenceRadius, float* disparities, float* interpolated){
  unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
  if(globalID < (disparityMapSize.x-(2*influenceRadius))*(disparityMapSize.y-(2*influenceRadius))){
    float disparity = disparities[globalID];
    int2 loc = {globalID%disparityMapSize.x + influenceRadius,globalID/disparityMapSize.x + influenceRadius};
    for(int y = loc.y - influenceRadius; y >= loc.y + influenceRadius; ++y){
      for(int x = loc.x - influenceRadius; x >= loc.x + influenceRadius; ++x){
        disparity += disparities[y*disparityMapSize.x + x]*(1 - abs((x-loc.x)/influenceRadius))*(1 - abs((y-loc.y)/influenceRadius));
      }
    }
    interpolated[globalID] = disparity;
  }
}


__global__ void ssrlcv::two_view_reproject(int numMatches, float4* matches, float cam1C[3], float cam1V[3],float cam2C[3], float cam2V[3], float K_inv[9], float rotationTranspose1[9], float rotationTranspose2[9], float3* points){
   unsigned long globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;

  if(!(globalID<numMatches))return;
	//check out globalID cheat sheet jackson gave you for this
	int matchIndex = globalID; //need to define once I calculate grid/block size
	float4 match = matches[globalID];


	float pix1[3] =
	{
		match.x, match.y, 1
	};
	float pix2[3] =
	{
		match.z, match.w, 1
  };
  float K_inv_reg[3][3];
  for(int r = 0; r < 3; ++r){
    for(int c = 0; c < 3; ++c){
      K_inv_reg[r][c] = K_inv[r*3 + c];
    }
  }
  float rotationTranspose1_reg[3][3];
   for(int r = 0; r < 3; ++r){
    for(int c = 0; c < 3; ++c){
      rotationTranspose1_reg[r][c] = rotationTranspose1[r*3 + c];
    }
  }
  float rotationTranspose2_reg[3][3];
   for(int r = 0; r < 3; ++r){
    for(int c = 0; c < 3; ++c){
      rotationTranspose2_reg[r][c] = rotationTranspose2[r*3 + c];
    }
  }

	float inter1[3];
	float inter2[3];

	float temp[3];
	multiply(K_inv_reg, pix1, temp);
	multiply(rotationTranspose1_reg, temp, inter1);
	multiply(K_inv_reg, pix2, temp);
	multiply(rotationTranspose2_reg, temp, inter2);

	float worldP1[3] =
	{
		inter1[0]+cam1C[0], inter1[1]+cam1C[1], inter1[2]+cam1C[2]
	};

	float worldP2[3] =
	{
		inter2[0]+cam2C[0], inter2[1]+cam2C[1], inter2[2]+cam2C[2]
	};

	float v1[3] =
	{
		worldP1[0] - cam1C[0], worldP1[1] - cam1C[1], worldP1[2] - cam1C[2]
	};

	float v2[3] =
	{
		worldP2[0] - cam2C[0], worldP2[1] - cam2C[1], worldP2[2] - cam2C[2]
	};

	normalize(v1);
	normalize(v2);



	//match1 and match2?
	float M1[3][3] =
	{
		{ 1-(v1[0]*v1[0]), 0-(v1[0]*v1[1]), 0-(v1[0]*v1[2]) },
		{ 0-(v1[0]*v1[1]), 1-(v1[1]*v1[1]), 0-(v1[1]*v1[2]) },
		{ 0-(v1[0]*v1[2]), 0-(v1[1]*v1[2]), 1-(v1[2]*v1[2]) }
	};

	float M2[3][3] =
	{
		{ 1-(v2[0]*v2[0]), 0-(v2[0]*v2[1]), 0-(v2[0]*v2[2]) },
		{ 0-(v2[0]*v2[1]), 1-(v2[1]*v2[1]), 0-(v2[1]*v2[2]) },
		{ 0-(v2[0]*v2[2]), 0-(v2[1]*v2[2]), 1-(v2[2]*v2[2]) }
	};

	float q1[3];
	float q2[3];
	float Q[3];

	multiply( M1, worldP1, q1);
	multiply( M2, worldP2, q2);

	float M[3][3];
	float M_inv[3][3];

	for(int r = 0; r < 3; ++r)
	{
		for(int c = 0; c < 3; ++c)
		{
			M[r][c] = M1[r][c] + M2[r][c];
		}
		Q[r] = q1[r] + q2[r];
	}

	float solution[3];
	inverse(M, M_inv);
	multiply(M_inv, Q, solution);



  	points[matchIndex].x = solution[0];
  	points[matchIndex].y = solution[1];
  	points[matchIndex].z = solution[2];

}


























































// yee
