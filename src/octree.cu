#include "hip/hip_runtime.h"
#include "octree.cuh"

using namespace std;

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )


inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  if (hipSuccess != err) {
      fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
      file, line, hipGetErrorString(err));
      exit(-1);
  }
#endif

  return;
}
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
    file, line, hipGetErrorString(err));
    exit(-1);
  }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    //err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
    file, line, hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}

/*
OCTREE NODE array
*/
__device__ __host__ Node::Node(){
  this->pointIndex = -1;
  this->center = {0.0f,0.0f,0.0f};
  this->key = 0;
  this->numPoints = 0;
  this->parent = -1;
  for(int i = 0; i < 27; ++i){
    if(i < 6){
      this->faces[i] = -1;
    }
    if(i < 8){
      this->children[i] = -1;
      this->vertices[i] = -1;
    }
    if(i < 12){
      this->edges[i] = -1;
    }
    this->neighbors[i] = -1;
  }
}

/*
HELPER METHODS AND CUDA KERNELS
*/
__device__ __host__ void printBits(size_t const size, void const * const ptr){
    unsigned char *b = (unsigned char*) ptr;
    unsigned char byte;
    int i, j;
    printf("bits - ");
    for (i=size-1;i>=0;i--)
    {
        for (j=7;j>=0;j--)
        {
            byte = (b[i] >> j) & 1;
            printf("%u", byte);
        }
    }
    printf("\n");
}
__global__ void getNodeKeys(float3* points, float3* nodeCenters, int* nodeKeys, float3 c, float W, int numPoints, int D){
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int globalID = bx * blockDim.x + tx;
  if(globalID < numPoints){
    float x = points[globalID].x;
    float y = points[globalID].y;
    float z = points[globalID].z;
    float leftx = c.x-W/2.0f, rightx = c.x + W/2.0f;
    float lefty = c.y-W/2.0f, righty = c.y + W/2.0f;
    float leftz = c.z-W/2.0f, rightz = c.z + W/2.0f;
    int key = 0;
    int depth = 1;
    while(depth <= D){
      if(x < c.x){
        key <<= 1;
        rightx = c.x;
        c.x = (leftx + rightx)/2.0f;
      }
      else{
        key = (key << 1) + 1;
        leftx = c.x;
        c.x = (leftx + rightx)/2.0f;
      }
      if(y < c.y){
        key <<= 1;
        righty = c.y;
        c.y = (lefty + righty)/2.0f;
      }
      else{
        key = (key << 1) + 1;
        lefty = c.y;
        c.y = (lefty + righty)/2.0f;
      }
      if(z < c.z){
        key <<= 1;
        rightz = c.z;
        c.z = (leftz + rightz)/2.0f;
      }
      else{
        key = (key << 1) + 1;
        leftz = c.z;
        c.z = (leftz + rightz)/2.0f;
      }
      depth++;
    }
    nodeKeys[globalID] = key;
    nodeCenters[globalID].x = c.x;
    nodeCenters[globalID].y = c.y;
    nodeCenters[globalID].z = c.z;
  }
}

__global__ void findAllNodes(int numUniqueNodes, int* nodeNumbers, Node* uniqueNodes){
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int globalID = bx * blockDim.x + tx;
  int tempCurrentKey = 0;
  int tempPrevKey = 0;
  if(globalID < numUniqueNodes){
    if(globalID == 0){
      nodeNumbers[globalID] = 0;
      return;
    }

    tempCurrentKey = uniqueNodes[globalID].key>>3;
    tempPrevKey = uniqueNodes[globalID - 1].key>>3;
    if(tempPrevKey == tempCurrentKey){
      nodeNumbers[globalID] = 0;
    }
    else{
      nodeNumbers[globalID] = 8;
    }
  }
}

void calculateNodeAddresses(int numUniqueNodes, Node* uniqueNodesDevice, int* nodeAddressesDevice, int* nodeNumbersDevice){
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  if(numUniqueNodes < 65535) grid.x = (unsigned int) numUniqueNodes;
  else{
    grid.x = 65535;
    while(grid.x*block.x < numUniqueNodes){
      ++block.x;
    }
    while(grid.x*block.x > numUniqueNodes){
      --grid.x;
    }
  }

  findAllNodes<<<grid,block>>>(numUniqueNodes, nodeNumbersDevice, uniqueNodesDevice);
  CudaCheckError();
  hipDeviceSynchronize();
  thrust::device_ptr<int> nN(nodeNumbersDevice);
  thrust::device_ptr<int> nA(nodeAddressesDevice);
  thrust::inclusive_scan(nN, nN + numUniqueNodes, nA);


}

__global__ void fillBlankNodeArray(Node* uniqueNodes, int* nodeNumbers, int* nodeAddresses, Node* outputNodeArray, int numUniqueNodes){
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int globalID = bx * blockDim.x + tx;
  int address = 0;
  if(globalID < numUniqueNodes && (globalID == 0 || nodeNumbers[globalID] == 8)){
    int siblingKey = uniqueNodes[globalID].key;
    siblingKey &= ~(1u);
    siblingKey &= ~(1u << 1);
    siblingKey &= ~(1u << 2);
    for(int i = 0; i < 8; ++i){
      Node currentNode = Node();
      currentNode.key = siblingKey + i;
      address = nodeAddresses[globalID] + i;
      outputNodeArray[address] = currentNode;
      //centers?
    }
  }
}

__global__ void fillFinestNodeArrayWithUniques(Node* uniqueNodes, int* nodeAddresses, Node* outputNodeArray, int numUniqueNodes, int* pointNodeIndex){
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int globalID = bx * blockDim.x + tx;
  int address = 0;
  int currentDKey = 0;
  Node currentNode = uniqueNodes[globalID];
  if(globalID < numUniqueNodes){
    currentDKey = currentNode.key&((1<<3)-1);
    address = nodeAddresses[globalID] + currentDKey;
    for(int i = currentNode.pointIndex; i < currentNode.numPoints + currentNode.pointIndex; ++i){
      pointNodeIndex[i] = address;
    }
    outputNodeArray[address] = currentNode;
  }
}

__global__ void fillNodeArrayWithUniques(Node* uniqueNodes, int* nodeAddresses, Node* outputNodeArray, int numUniqueNodes){
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int globalID = bx * blockDim.x + tx;
  int address = 0;
  int currentDKey = 0;
  Node currentNode = uniqueNodes[globalID];
  if(globalID < numUniqueNodes){
    currentDKey = currentNode.key&((1<<3)-1);
    address = nodeAddresses[globalID] + currentDKey;
    outputNodeArray[address] = currentNode;
  }
}

__global__ void generateParentalUniqueNodes(Node* uniqueNodes, Node* nodeArrayD, int numNodesAtDepth){
  int numUniqueNodesAtDepth = numNodesAtDepth / 8;
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int globalID = bx * blockDim.x + tx;
  int nodeArrayIndex = globalID*8;
  Node parentNode = Node();
  int parentKey;
  if(globalID < numUniqueNodesAtDepth){
    parentKey = nodeArrayD[nodeArrayIndex].key;
    parentKey >>= 3;
    parentNode.key = parentKey;
    parentNode.pointIndex = nodeArrayD[nodeArrayIndex].pointIndex;
    for(int i = 0; i < 8; ++i){
      parentNode.numPoints += nodeArrayD[nodeArrayIndex + i].numPoints;
      nodeArrayD[nodeArrayIndex + i].parent = globalID;
      parentNode.children[i] = nodeArrayIndex + i;//index of NodeArry(d+1)
      //center?
      uniqueNodes[globalID] = parentNode;
    }
  }
}

//each node should have atleast 8 neighbors
__global__ void computeNeighboringNodes(Node* nodeArray, int numNodes, int depthIndex, int childDepthIndex, int depthDistance,int* parentLUT, int* childLUT, int* numNeighbors){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numNodes){
    blockID += depthIndex;
    nodeArray[blockID].neighbors[13] = blockID;
    int parentIndex = nodeArray[blockID].parent;
    if(childDepthIndex != -1){
      for(int i = 0; i < 8; ++i){
        if(nodeArray[blockID].children[i] != -1) nodeArray[blockID].children[i] += childDepthIndex;
      }
    }
    if(parentIndex != -1){
       nodeArray[blockID].parent += (depthIndex + numNodes);
       parentIndex += (depthIndex + numNodes);
    }
    else{
      //then it is highest node
      return;
    }
    __syncthreads();
    int depthKey = nodeArray[blockID].key&((1<<3)-1);
    int parentLUTIndex = parentLUT[depthKey*8 + threadIdx.x];
    int childLUTIndex = childLUT[depthKey*8 + threadIdx.x];
    if(depthKey > 7 || depthKey < 0){
      printf("ERROR in depthKey generation...depthKey = %d\n",depthKey);
    }
    int neighborParentIndex = 0;
    neighborParentIndex = nodeArray[parentIndex].neighbors[parentLUTIndex];
    if(neighborParentIndex != -1){
      //printf("depth = %d-%d, current = %d, parent = %d, %d, %d\n",depthIndex, depthIndex + numNodes, blockID, parentIndex,neighborParentIndex,nodeArray[neighborParentIndex].children[childLUTIndex]);
      if(nodeArray[neighborParentIndex].children[childLUTIndex] != -1 ){
        nodeArray[blockID].neighbors[threadIdx.x] = nodeArray[neighborParentIndex].children[childLUTIndex];
        atomicAdd(numNeighbors, 1);

      }

    }
    else{
      nodeArray[blockID].neighbors[threadIdx.x] = -1;
    }
    //if(blockID == depthIndex && threadIdx.x == 26){
    //  printBits(sizeof(int), &nodeArray[blockID].key);
    //  printf("%d,%d,%d,%d\n", parentIndex,blockID,parentLUTIndex,neighborParentIndex);
    //}
  }

}



/*
OCTREE CLASS FUNCTIONS
*/
Octree::Octree(){

}

Octree::~Octree(){

}

void Octree::parsePLY(string pathToFile){
  cout<<pathToFile + "'s data to be transfered to an empty octree."<<endl;
	ifstream plystream(pathToFile);
	string currentLine;
  vector<float3> points;
  vector<float3> normals;
  float minX = 0, minY = 0, minZ = 0, maxX = 0, maxY = 0, maxZ = 0;
	if (plystream.is_open()) {
		while (getline(plystream, currentLine)) {
      stringstream getMyFloats = stringstream(currentLine);
      float value = 0.0;
      int index = 0;
      float3 point;
      float3 normal;
      bool lineIsDone = false;
      int numPoints= 0;
      while(getMyFloats >> value){
        switch(index){
          case 0:
            point.x = value;
            if(value > maxX) maxX = value;
            if(value < minX) minX = value;
            break;
          case 1:
            point.y = value;
            if(value > maxY) maxY = value;
            if(value < minY) minY = value;
            break;
          case 2:
            point.z = value;
            if(value > maxZ) maxZ = value;
            if(value < minZ) minZ = value;
            break;
          case 3:
            normal.x = value;
            break;
          case 4:
            normal.y = value;
            break;
          case 5:
            normal.z = value;
            break;
          default:
            numPoints++;
            lineIsDone = true;
            points.push_back(point);
            normals.push_back(normal);
            break;
        }
        if(lineIsDone) break;
        ++index;
      }
		}

    cout<<numPoints<<endl;
    this->min = {minX,minY,minZ};
    this->max = {maxX,maxY,maxZ};

    this->center.x = (maxX + minX)/2;
    this->center.y = (maxY + minY)/2;
    this->center.z = (maxZ + minZ)/2;

    this->width = maxX - minX;
    if(this->width < maxY - minY) this->width = maxY - minY;
    if(this->width < maxZ - minZ) this->width = maxZ - minZ;

    this->numPoints = (int) points.size();
    this->points = new float3[this->numPoints];
    this->normals = new float3[this->numPoints];
    this->finestNodeCenters = new float3[this->numPoints];
    this->finestNodePointIndexes = new int[this->numPoints];
    this->finestNodeKeys = new int[this->numPoints];
    this->pointNodeIndex = new int[this->numPoints];
    this->totalNodes = 0;
    this->numFinestUniqueNodes = 0;

    for(int i = 0; i < points.size(); ++i){
      this->points[i] = points[i];
      this->normals[i] = normals[i];
      this->finestNodeCenters[i] = {0.0f,0.0f,0.0f};
      this->finestNodeKeys[i] = 0;
      this->pointNodeIndex[i] = -1;
      //initializing here even though points are not sorted yet
      this->finestNodePointIndexes[i] = i;
    }
    printf("\nmin = %f,%f,%f\n",this->min.x,this->min.y,this->min.z);
    printf("max = %f,%f,%f\n",this->max.x,this->max.y,this->max.z);
    printf("bounding box width = %f\n", this->width);
    printf("center = %f,%f,%f\n",this->center.x,this->center.y,this->center.z);
    printf("number of points = %d\n\n", this->numPoints);
    cout<<pathToFile + "'s data has been transfered to an initialized octree.\n"<<endl;
	}
	else{
    cout << "Unable to open: " + pathToFile<< endl;
    exit(1);
  }
}

Octree::Octree(string pathToFile, int depth){
  this->parsePLY(pathToFile);
  this->depth = depth;
}

void Octree::copyPointsToDevice(){
  CudaSafeCall(hipMemcpy(this->pointsDevice, this->points, this->numPoints * sizeof(float3), hipMemcpyHostToDevice));
}
void Octree::copyPointsToHost(){
  CudaSafeCall(hipMemcpy(this->points, this->pointsDevice, this->numPoints * sizeof(float3), hipMemcpyDeviceToHost));

}
void Octree::copyNormalsToDevice(){
  CudaSafeCall(hipMemcpy(this->normalsDevice, this->normals, this->numPoints * sizeof(float3), hipMemcpyHostToDevice));
}
void Octree::copyNormalsToHost(){
  CudaSafeCall(hipMemcpy(this->normals, this->normalsDevice, this->numPoints * sizeof(float3), hipMemcpyDeviceToHost));
}

void Octree::copyFinestNodeCentersToDevice(){
  CudaSafeCall(hipMemcpy(this->finestNodeCentersDevice, this->finestNodeCenters, this->numPoints * sizeof(float3), hipMemcpyHostToDevice));
}
void Octree::copyFinestNodeCentersToHost(){
  CudaSafeCall(hipMemcpy(this->finestNodeCenters, this->finestNodeCentersDevice, this->numPoints * sizeof(float3), hipMemcpyDeviceToHost));

}
void Octree::copyFinestNodeKeysToDevice(){
  CudaSafeCall(hipMemcpy(this->finestNodeKeysDevice, this->finestNodeKeys, this->numPoints * sizeof(int), hipMemcpyHostToDevice));
}
void Octree::copyFinestNodeKeysToHost(){
  CudaSafeCall(hipMemcpy(this->finestNodeKeys, this->finestNodeKeysDevice, this->numPoints * sizeof(int), hipMemcpyDeviceToHost));
}
void Octree::copyFinestNodePointIndexesToDevice(){
  CudaSafeCall(hipMemcpy(this->finestNodePointIndexesDevice, this->finestNodePointIndexes, this->numPoints * sizeof(int), hipMemcpyHostToDevice));
}
void Octree::copyFinestNodePointIndexesToHost(){
  CudaSafeCall(hipMemcpy(this->finestNodePointIndexes, this->finestNodePointIndexesDevice, this->numPoints * sizeof(int), hipMemcpyDeviceToHost));
}

void Octree::executeKeyRetrieval(dim3 grid, dim3 block){

  getNodeKeys<<<grid,block>>>(this->pointsDevice, this->finestNodeCentersDevice, this->finestNodeKeysDevice, this->center, this->width, this->numPoints, this->depth);
  CudaCheckError();

}

void Octree::sortByKey(){
  int* keyTemp = new int[this->numPoints];
  int* keyTempDevice;
  CudaSafeCall(hipMalloc((void**)&keyTempDevice, this->numPoints*sizeof(int)));

  for(int array = 0; array < 2; ++array){
    for(int i = 0; i < this->numPoints; ++i){
      keyTemp[i] = this->finestNodeKeys[i];
    }
    thrust::device_ptr<float3> P(this->pointsDevice);
    thrust::device_ptr<float3> C(this->finestNodeCentersDevice);
    thrust::device_ptr<float3> N(this->normalsDevice);
    thrust::device_ptr<int> K(this->finestNodeKeysDevice);

    CudaSafeCall(hipMemcpy(keyTempDevice, keyTemp, this->numPoints * sizeof(int), hipMemcpyHostToDevice));
    thrust::device_ptr<int> KT(keyTempDevice);
    if(array == 0){
      thrust::sort_by_key(KT, KT + this->numPoints, P);
    }
    else if(array == 1){
      thrust::sort_by_key(KT, KT + this->numPoints, C);
      thrust::sort_by_key(K, K + this->numPoints, N);
    }
  }
}

void Octree::compactData(){
  thrust::pair<int*, float3*> nodeKeyCenters;//the last value of these node arrays
  thrust::pair<int*, int*> nodeKeyPointIndexes;//the last value of these node arrays

  int* keyTemp = new int[this->numPoints];
  for(int i = 0; i < this->numPoints; ++i){
    keyTemp[i] = this->finestNodeKeys[i];
  }
  nodeKeyCenters = thrust::unique_by_key(keyTemp, keyTemp + this->numPoints, this->finestNodeCenters);
  nodeKeyPointIndexes = thrust::unique_by_key(this->finestNodeKeys, this->finestNodeKeys + this->numPoints, this->finestNodePointIndexes);
  int numUniqueNodes = 0;
  for(int i = 0; this->finestNodeKeys[i] != *nodeKeyPointIndexes.first; ++i,++numUniqueNodes);
  this->numFinestUniqueNodes = numUniqueNodes;

}

void Octree::fillUniqueNodesAtFinestLevel(){
  //we have keys, centers, numpoints, point indexes
  this->uniqueNodesAtFinestLevel = new Node[this->numFinestUniqueNodes];
  for(int i = 0; i < this->numFinestUniqueNodes; ++i){
    Node currentNode;
    currentNode.key = this->finestNodeKeys[i];
    currentNode.center = this->finestNodeCenters[i];
    currentNode.pointIndex = this->finestNodePointIndexes[i];
    if(i + 1 != this->numFinestUniqueNodes){
      currentNode.numPoints = this->finestNodePointIndexes[i + 1] - this->finestNodePointIndexes[i];
    }
    else{
      currentNode.numPoints = this->numPoints - this->finestNodePointIndexes[i] - 1;
    }

    this->uniqueNodesAtFinestLevel[i] = currentNode;
  }
}

void Octree::createFinalNodeArray(){
  cout<<this->numFinestUniqueNodes<<"\n"<<endl;

  Node* uniqueNodesDevice;
  CudaSafeCall(hipMalloc((void**)&uniqueNodesDevice, this->numFinestUniqueNodes*sizeof(Node)));
  CudaSafeCall(hipMemcpy(uniqueNodesDevice, this->uniqueNodesAtFinestLevel, this->numFinestUniqueNodes*sizeof(Node), hipMemcpyHostToDevice));

  Node** nodeArray2DDevice;
  CudaSafeCall(hipMalloc((void**)&nodeArray2DDevice, (this->depth + 1)*sizeof(Node*)));
  Node** nodeArray2D = new Node*[this->depth + 1];
  CudaSafeCall(hipMemcpy(nodeArray2D, nodeArray2DDevice, (this->depth + 1)*sizeof(Node*), hipMemcpyDeviceToHost));

  int* nodeAddressesDevice;
  int* nodeNumbersDevice;

  this->depthIndex = new int[this->depth + 1];
  CudaSafeCall(hipMalloc((void**)&this->depthIndexDevice, (this->depth + 1)*sizeof(int)));

  int numUniqueNodes = this->numFinestUniqueNodes;

  for(int d = this->depth; d >= 0; --d){
    cout<<"depth "<<d<<endl;
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    if(numUniqueNodes < 65535) grid.x = (unsigned int) numUniqueNodes;
    else{
      grid.x = 65535;
      while(grid.x*block.x < numUniqueNodes){
        ++block.x;
      }
      while(grid.x*block.x > numUniqueNodes){
        --grid.x;
        if(grid.x*block.x < numUniqueNodes){
          ++grid.x;//to ensure that numThreads > numUniqueNodes
          break;
        }
      }
    }
    int* nodeAddressesHost = new int[numUniqueNodes];

    for(int i = 0; i < numUniqueNodes; ++i){
      nodeAddressesHost[i] = 0;
    }

    CudaSafeCall(hipMalloc((void**)&nodeNumbersDevice, numUniqueNodes * sizeof(int)));
    CudaSafeCall(hipMalloc((void**)&nodeAddressesDevice, numUniqueNodes*sizeof(int)));
    //this is just to fill the arrays with 0s
    CudaSafeCall(hipMemcpy(nodeNumbersDevice, nodeAddressesHost, numUniqueNodes * sizeof(int), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(nodeAddressesDevice, nodeAddressesHost, numUniqueNodes * sizeof(int), hipMemcpyHostToDevice));

    calculateNodeAddresses(numUniqueNodes, uniqueNodesDevice, nodeAddressesDevice, nodeNumbersDevice);
    CudaSafeCall(hipMemcpy(nodeAddressesHost, nodeAddressesDevice, numUniqueNodes* sizeof(int), hipMemcpyDeviceToHost));

    int numNodesAtDepth = d > 0 ? nodeAddressesHost[numUniqueNodes - 1] + 8: 1;
    cout<<"NUM NODES AT DEPTH = "<<numNodesAtDepth<<"\nNUM UNIQUE NODES AT DEPTH = "<<numUniqueNodes<<endl;
    delete[] nodeAddressesHost;

    CudaSafeCall(hipMalloc((void**)&nodeArray2D[this->depth - d], numNodesAtDepth* sizeof(Node)));

    fillBlankNodeArray<<<grid,block>>>(uniqueNodesDevice, nodeNumbersDevice,  nodeAddressesDevice, nodeArray2D[this->depth - d], numUniqueNodes);
    CudaCheckError();
    hipDeviceSynchronize();
    if(this->depth == d){
      int* pointNodeIndexDevice;
      CudaSafeCall(hipMalloc((void**)&pointNodeIndexDevice, this->numPoints*sizeof(int)));
      CudaSafeCall(hipMemcpy(pointNodeIndexDevice, this->pointNodeIndex, numUniqueNodes* sizeof(int), hipMemcpyHostToDevice));
      fillFinestNodeArrayWithUniques<<<grid,block>>>(uniqueNodesDevice, nodeAddressesDevice, nodeArray2D[this->depth - d], numUniqueNodes, pointNodeIndexDevice);
      CudaCheckError();
      CudaSafeCall(hipMemcpy(this->pointNodeIndex, pointNodeIndexDevice, this->numPoints*sizeof(int), hipMemcpyDeviceToHost));
      CudaSafeCall(hipFree(pointNodeIndexDevice));
    }
    else{
      fillNodeArrayWithUniques<<<grid,block>>>(uniqueNodesDevice, nodeAddressesDevice, nodeArray2D[this->depth - d], numUniqueNodes);
      CudaCheckError();
    }
    CudaSafeCall(hipFree(uniqueNodesDevice));
    CudaSafeCall(hipFree(nodeAddressesDevice));
    CudaSafeCall(hipFree(nodeNumbersDevice));

    numUniqueNodes = numNodesAtDepth / 8;

    //get unique nodes at next depth
    if(d > 0){
      CudaSafeCall(hipMalloc((void**)&uniqueNodesDevice, numUniqueNodes*sizeof(Node)));
      if(numUniqueNodes < 65535) grid.x = (unsigned int) numUniqueNodes;
      else{
        grid.x = 65535;
        while(grid.x*block.x < numUniqueNodes){
          ++block.x;
        }
        while(grid.x*block.x > numUniqueNodes){
          --grid.x;
          if(grid.x*block.x < numUniqueNodes){
            ++grid.x;//to ensure that numThreads > numUniqueNodes
            break;
          }
        }
      }
      generateParentalUniqueNodes<<<grid,block>>>(uniqueNodesDevice, nodeArray2D[this->depth - d], numNodesAtDepth);
      CudaCheckError();
    }
    this->depthIndex[this->depth - d] = this->totalNodes;
    cout<<"DEPTH INDEX = "<<this->totalNodes<<endl;
    this->totalNodes += numNodesAtDepth;
  }
  cout<<"2D NODE ARRAY COMPLETED\n"<<endl;
  this->finalNodeArray = new Node[this->totalNodes];
  CudaSafeCall(hipMalloc((void**)&this->finalNodeArrayDevice, this->totalNodes*sizeof(Node)));
  for(int i = 0; i <= this->depth; ++i){
    if(i < this->depth){
      CudaSafeCall(hipMemcpy(this->finalNodeArrayDevice + this->depthIndex[i], nodeArray2D[i], (this->depthIndex[i+1]-this->depthIndex[i])*sizeof(Node), hipMemcpyDeviceToDevice));
    }
    else{
      CudaSafeCall(hipMemcpy(this->finalNodeArrayDevice + this->depthIndex[i], nodeArray2D[i], sizeof(Node), hipMemcpyDeviceToDevice));
    }
    CudaSafeCall(hipFree(nodeArray2D[i]));
  }
  CudaSafeCall(hipMemcpy(this->finalNodeArray, this->finalNodeArrayDevice, this->totalNodes*sizeof(Node), hipMemcpyDeviceToHost));
  CudaSafeCall(hipMemcpy(this->depthIndexDevice, this->depthIndex, (this->depth + 1)*sizeof(int), hipMemcpyHostToDevice));
  //for(int i =this->depthIndex[0]; i < this->depthIndex[1] - this->depthIndex[0]; ++i){
  //for(int i = 0 ; i < 64; ++i){
  //  printBits(sizeof(int), &this->finalNodeArray[i].key);
  //}

  delete[] nodeArray2D;
  cout<<"NODE ARRAY FLATTENED AND COMPLETED"<<endl;
}

void Octree::fillLUTs(){
  int c[6][6][6];
  int p[6][6][6];

  int numbParent = 0;
  for (int k = 5; k >= 0; k -= 2){
    for (int i = 0; i < 6; i += 2){
    	for (int j = 5; j >= 0; j -= 2){
    		int numb = 0;
    		for (int l = 0; l < 2; l++){
    		  for (int m = 0; m < 2; m++){
    				for (int n = 0; n < 2; n++){
    					c[i+m][j-n][k-l] = numb++;
    					p[i+m][j-n][k-l] = numbParent;
    				}
    			}
        }
        numbParent++;
      }
    }
  }

  int numbLUT = 0;
  for (int k = 3; k > 1; k--){
    for (int i = 2; i < 4; i++){
    	for (int j = 3; j > 1; j--){
    		int numb = 0;
    		for (int n = 1; n >= -1; n--){
    			for (int l = -1; l <= 1; l++){
    				for (int m = 1; m >= -1; m--){
    					this->parentLUT[numbLUT][numb] = p[i+l][j+m][k+n];
    					this->childLUT[numbLUT][numb++] = c[i+l][j+m][k+n];
    				}
    			}
        }
        numbLUT++;
      }
    }
  }
  int flatParentLUT[8*27];
  int flatChildLUT[8*27];
  int flatCounter = 0;
  for(int row = 0; row < 8; ++row){
    for(int col = 0; col < 27; ++col){
      flatParentLUT[flatCounter] = this->parentLUT[row][col];
      flatChildLUT[flatCounter] = this->childLUT[row][col];
      flatCounter++;
    }
  }
  CudaSafeCall(hipMalloc((void**)&this->parentLUTDevice, 8*27*sizeof(int)));
  CudaSafeCall(hipMalloc((void**)&this->childLUTDevice, 8*27*sizeof(int)));
  CudaSafeCall(hipMemcpy(this->parentLUTDevice, flatParentLUT, 8*27*sizeof(int), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(this->childLUTDevice, flatChildLUT, 8*27*sizeof(int), hipMemcpyHostToDevice));

}

void Octree::printLUTs(){
  cout<<"\nPARENT LUT"<<endl;
  for(int row = 0; row <  8; ++row){
    for(int col = 0; col < 27; ++col){
      cout<<this->parentLUT[row][col]<<" ";
    }
    cout<<endl;
  }
  cout<<"\nCHILD LUT"<<endl;
  for(int row = 0; row <  8; ++row){
    for(int col = 0; col < 27; ++col){
      cout<<this->childLUT[row][col]<<" ";
    }
    cout<<endl;
  }

}

void Octree::fillNeighborhoods(){

  //need to use highest number of nodes in a depth instead of totalNodes
  dim3 grid = {1,1,1};
  dim3 block = {27,1,1};
  int numNodesAtDepth;
  int depthStartingIndex;
  int childDepthStartingIndex;
  int atomicCounter = 0;
  int* atomicCounterDevice;
  CudaSafeCall(hipMalloc((void**)&atomicCounterDevice, sizeof(int)));
  for(int i = this->depth; i >= 0 ; --i){
    numNodesAtDepth = 1;
    childDepthStartingIndex = -1;
    depthStartingIndex = this->depthIndex[i];
    if(i != depth){
      numNodesAtDepth = this->depthIndex[i + 1] - depthStartingIndex;
    }
    if(i != 0){
      childDepthStartingIndex = this->depthIndex[i-1];
    }
    if(numNodesAtDepth < 65535) grid.x = (unsigned int) numNodesAtDepth;
    else{
      grid.x = 65535;
      while(grid.x*grid.y < numNodesAtDepth){
        ++grid.y;
      }
      while(grid.x*grid.y > numNodesAtDepth){
        --grid.x;
        if(grid.x*grid.y < numNodesAtDepth){
          ++grid.x;//to ensure that numThreads > totalNodes
          break;
        }
      }
    }
    atomicCounter = 0;
    CudaSafeCall(hipMemcpy(atomicCounterDevice, &atomicCounter, sizeof(int), hipMemcpyHostToDevice));
    cout<<"COMPUTE NEIGHBORHOOD ON DEPTH "<<this->depth - i<<" HAS INITIATED AND INCLUDES "<<numNodesAtDepth<<" NODES STARTING AT "<<depthStartingIndex<<endl;
    computeNeighboringNodes<<<grid, block>>>(this->finalNodeArrayDevice, numNodesAtDepth, depthStartingIndex, childDepthStartingIndex,this->depth - i, this->parentLUTDevice, this->childLUTDevice, atomicCounterDevice);
    CudaCheckError();
    CudaSafeCall(hipMemcpy(&atomicCounter, atomicCounterDevice, sizeof(int), hipMemcpyDeviceToHost));
    cout<<"NUM NEIGHBORS = "<<atomicCounter<<endl;
  }
  CudaSafeCall(hipMemcpy(this->finalNodeArray, this->finalNodeArrayDevice, this->totalNodes * sizeof(Node), hipMemcpyDeviceToHost));

}
void Octree::computeVertexArray(){

}
void Octree::computeEdgeArray(){

}
void Octree::computeFaceArray(){

}
