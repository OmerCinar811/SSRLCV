#include "hip/hip_runtime.h"
#include "octree.cuh"

using namespace std;

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )


inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  if (hipSuccess != err) {
      fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
      file, line, hipGetErrorString(err));
      exit(-1);
  }
#endif

  return;
}
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
    file, line, hipGetErrorString(err));
    exit(-1);
  }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    //err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
    file, line, hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}


//pretty much just a binary search in each dimension performed by threads
__global__ void getNodeKeys(float3* points, float3* nodeCenters, int* nodeKeys, float3 c, float W, int numPoints, int D){
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int globalID = bx * blockDim.x + tx;
  if(globalID < numPoints){
    float x = points[globalID].x;
    float y = points[globalID].y;
    float z = points[globalID].z;
    float leftx = c.x-W/2.0f, rightx = c.x + W/2.0f;
    float lefty = c.y-W/2.0f, righty = c.y + W/2.0f;
    float leftz = c.z-W/2.0f, rightz = c.z + W/2.0f;
    int key = 0;
    int depth = 1;
    while(depth <= D){
      if(x < c.x){
        key <<= 1;
        rightx = c.x;
        c.x = (leftx + rightx)/2.0f;
      }
      else{
        key = (key << 1) + 1;
        leftx = c.x;
        c.x = (leftx + rightx)/2.0f;
      }
      if(y < c.y){
        key <<= 1;
        righty = c.y;
        c.y = (lefty + righty)/2.0f;
      }
      else{
        key = (key << 1) + 1;
        lefty = c.y;
        c.y = (lefty + righty)/2.0f;
      }
      if(z < c.z){
        key <<= 1;
        rightz = c.z;
        c.z = (leftz + rightz)/2.0f;
      }
      else{
        key = (key << 1) + 1;
        leftz = c.z;
        c.z = (leftz + rightz)/2.0f;
      }
      depth++;
    }
    nodeKeys[globalID] = key;
    nodeCenters[globalID].x = c.x;
    nodeCenters[globalID].y = c.y;
    nodeCenters[globalID].z = c.z;
  }
}

__global__ void findAllNodes(int numUniqueNodes, int* nodeNumbers, Node* uniqueNodes){
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int globalID = bx * blockDim.x + tx;
  int tempCurrentKey = 0;
  int tempPrevKey = 0;
  if(globalID < numUniqueNodes){
    if(globalID == 0){
      nodeNumbers[globalID] = 0;
      return;
    }
    tempCurrentKey = uniqueNodes[globalID].key >> 3;
    tempPrevKey = uniqueNodes[globalID - 1].key >> 3;
    if(tempPrevKey == tempCurrentKey){
      nodeNumbers[globalID] = 0;
    }
    else{
      nodeNumbers[globalID] = 8;
    }
  }
}

__global__ void fill1NodeArray(Node* uniqueNodes, int* nodeAddresses, Node* finalNodeArray, int numUniqueNodes){
  int tx = threadIdx.x;
  int bx = blockIdx.x;
  int globalID = bx * blockDim.x + tx;
  Node currentNode;
  int address = 0;
  int currentDKey = 0;
  if(globalID < numUniqueNodes){
    currentNode = uniqueNodes[globalID];
    currentDKey = currentNode.key&((1<<3)-1);
    address = nodeAddresses[globalID] + currentDKey;//actually last three printBits
    finalNodeArray[address] = currentNode;
  }
}


Octree::Octree(){

}

Octree::~Octree(){

}

void Octree::parsePLY(string pathToFile){
  cout<<pathToFile + "'s data to be transfered to an empty octree."<<endl;
	ifstream plystream(pathToFile);
	string currentLine;
  vector<float3> points;
  vector<float3> normals;
  float minX = 0, minY = 0, minZ = 0, maxX = 0, maxY = 0, maxZ = 0;
	if (plystream.is_open()) {
		while (getline(plystream, currentLine)) {
      stringstream getMyFloats = stringstream(currentLine);
      float value = 0.0;
      int index = 0;
      float3 point;
      float3 normal;
      bool lineIsDone = false;
      while(getMyFloats >> value){
        switch(index){
          case 0:
            point.x = value;
            if(value > maxX) maxX = value;
            if(value < minX) minX = value;
            break;
          case 1:
            point.y = value;
            if(value > maxY) maxY = value;
            if(value < minY) minY = value;
            break;
          case 2:
            point.z = value;
            if(value > maxZ) maxZ = value;
            if(value < minZ) minZ = value;
            break;
          case 3:
            normal.x = value;
            break;
          case 4:
            normal.y = value;
            break;
          case 5:
            normal.z = value;
            break;
          default:
            lineIsDone = true;
            points.push_back(point);
            normals.push_back(normal);
            break;
        }
        if(lineIsDone) break;
        ++index;
      }
		}
    this->min = {minX,minY,minZ};
    this->max = {maxX,maxY,maxZ};

    this->center.x = (maxX + minX)/2;
    this->center.y = (maxY + minY)/2;
    this->center.z = (maxZ +  minZ)/2;

    this->width = maxX - minX;
    if(this->width < maxY - minY) this->width = maxY - minY;
    if(this->width < maxZ - minZ) this->width = maxZ - minZ;

    this->numPoints = (int) points.size();
    this->points = new float3[this->numPoints];
    this->normals = new float3[this->numPoints];
    this->nodeCenters = new float3[this->numPoints];
    this->nodePointIndexes = new int[this->numPoints];
    this->nodeKeys = new int[this->numPoints];
    this->totalNodes = 0;
    this->numUniqueNodes = 0;

    for(int i = 0; i < points.size(); ++i){
      this->points[i] = points[i];
      this->normals[i] = normals[i];
      this->nodeCenters[i] = {0.0f,0.0f,0.0f};
      this->nodeKeys[i] = 0;
      //initializing here even though points are not sorted yet
      this->nodePointIndexes[i] = i;
    }
    printf("\nmin = %f,%f,%f\n",this->min.x,this->min.y,this->min.z);
    printf("max = %f,%f,%f\n",this->max.x,this->max.y,this->max.z);
    printf("center = %f,%f,%f\n",this->center.x,this->center.y,this->center.z);
    printf("number of points = %d\n\n", this->numPoints);
    cout<<pathToFile + "'s data has been transfered to an initialized octree.\n"<<endl;
	}
	else{
    cout << "Unable to open: " + pathToFile<< endl;
    exit(1);
  }
}

Octree::Octree(string pathToFile, int depth){
  this->parsePLY(pathToFile);
  this->depth = depth;
}

void Octree::copyPointsToDevice(){
  CudaSafeCall(hipMemcpy(this->pointsDevice, this->points, this->numPoints * sizeof(float3), hipMemcpyHostToDevice));
}
void Octree::copyPointsToHost(){
  CudaSafeCall(hipMemcpy(this->points, this->pointsDevice, this->numPoints * sizeof(float3), hipMemcpyDeviceToHost));

}
void Octree::copyNormalsToDevice(){
  CudaSafeCall(hipMemcpy(this->normalsDevice, this->normals, this->numPoints * sizeof(float3), hipMemcpyHostToDevice));
}
void Octree::copyNormalsToHost(){
  CudaSafeCall(hipMemcpy(this->normals, this->normalsDevice, this->numPoints * sizeof(float3), hipMemcpyDeviceToHost));

}
void Octree::copyNodeCentersToDevice(){
  CudaSafeCall(hipMemcpy(this->nodeCentersDevice, this->nodeCenters, this->numPoints * sizeof(float3), hipMemcpyHostToDevice));
}
void Octree::copyNodeCentersToHost(){
  CudaSafeCall(hipMemcpy(this->nodeCenters, this->nodeCentersDevice, this->numPoints * sizeof(float3), hipMemcpyDeviceToHost));

}
void Octree::copyNodeKeysToDevice(){
  CudaSafeCall(hipMemcpy(this->nodeKeysDevice, this->nodeKeys, this->numPoints * sizeof(int), hipMemcpyHostToDevice));
}
void Octree::copyNodeKeysToHost(){
  CudaSafeCall(hipMemcpy(this->nodeKeys, this->nodeKeysDevice, this->numPoints * sizeof(int), hipMemcpyDeviceToHost));

}
void Octree::copyNodePointIndexesToDevice(){
  CudaSafeCall(hipMemcpy(this->nodePointIndexesDevice, this->nodePointIndexes, this->numPoints * sizeof(int), hipMemcpyHostToDevice));
}
void Octree::copyNodePointIndexesToHost(){
  CudaSafeCall(hipMemcpy(this->nodePointIndexes, this->nodePointIndexesDevice, this->numPoints * sizeof(int), hipMemcpyDeviceToHost));
}
void Octree::copyFinalNodeArrayToDevice(){
  CudaSafeCall(hipMemcpy(this->finalNodeArrayDevice, this->finalNodeArray, this->totalNodes * sizeof(Node), hipMemcpyHostToDevice));

}
void Octree::copyFinalNodeArrayToHost(){
  CudaSafeCall(hipMemcpy(this->finalNodeArray, this->finalNodeArrayDevice, this->totalNodes * sizeof(Node), hipMemcpyDeviceToHost));

}

void Octree::executeKeyRetrieval(dim3 grid, dim3 block){

  getNodeKeys<<<grid,block>>>(this->pointsDevice, this->nodeCentersDevice, this->nodeKeysDevice, this->center, this->width, this->numPoints, this->depth);
  CudaCheckError();

}

void Octree::sortByKey(){
  int* keyTemp = new int[this->numPoints];
  int* keyTempDevice;
  CudaSafeCall(hipMalloc((void**)&keyTempDevice, this->numPoints*sizeof(int)));

  for(int array = 0; array < 2; ++array){
    for(int i = 0; i < this->numPoints; ++i){
      keyTemp[i] = this->nodeKeys[i];
    }
    thrust::device_ptr<float3> P(this->pointsDevice);
    thrust::device_ptr<float3> C(this->nodeCentersDevice);
    thrust::device_ptr<float3> N(this->normalsDevice);
    thrust::device_ptr<int> K(this->nodeKeysDevice);

    CudaSafeCall(hipMemcpy(keyTempDevice, keyTemp, this->numPoints * sizeof(int), hipMemcpyHostToDevice));
    thrust::device_ptr<int> KT(keyTempDevice);
    if(array == 0){
      thrust::sort_by_key(KT, KT + this->numPoints, P);
    }
    else if(array == 1){
      thrust::sort_by_key(KT, KT + this->numPoints, C);
      thrust::sort_by_key(K, K + this->numPoints, N);
    }
  }
}

//three new node arrays are instantiated here once numUniqueNodes is found out
void Octree::compactData(){
  thrust::pair<int*, float3*> nodeKeyCenters;//the last value of these node arrays
  thrust::pair<int*, int*> nodeKeyPointIndexes;//the last value of these node arrays

  int* keyTemp = new int[this->numPoints];
  for(int i = 0; i < this->numPoints; ++i){
    keyTemp[i] = this->nodeKeys[i];
  }
  nodeKeyCenters = thrust::unique_by_key(keyTemp, keyTemp + this->numPoints, this->nodeCenters);
  nodeKeyPointIndexes = thrust::unique_by_key(this->nodeKeys, this->nodeKeys + this->numPoints, this->nodePointIndexes);
  int numUniqueNodes = 0;
  for(int i = 0; this->nodeKeys[i] != *nodeKeyPointIndexes.first; ++i){
    ++numUniqueNodes;
  }
  this->numUniqueNodes = numUniqueNodes;

}

void Octree::fillInUniqueNodes(){
  this->uniqueNodeArray = new Node[this->numUniqueNodes];
  for(int i = 0; i < this->numUniqueNodes; ++i){
    this->uniqueNodeArray[i].center = this->nodeCenters[i];
    this->uniqueNodeArray[i].pointIndex = this->nodePointIndexes[i];
    if(i + 1 == this->numUniqueNodes){
      this->uniqueNodeArray[i].numPoints = this->nodePointIndexes[i + 1] - this->nodePointIndexes[i];
    }
    else{
      this->uniqueNodeArray[i].numPoints = this->numPoints - this->nodePointIndexes[i];

    }
    this->uniqueNodeArray[i].key = this->nodeKeys[i];
  }
}

void Octree::executeFindAllNodes(dim3 grid, dim3 block, int numUniqueNodes, int* nodeNumbers, Node* uniqueNodes){

  findAllNodes<<<grid,block>>>(numUniqueNodes, nodeNumbers, uniqueNodes);
  CudaCheckError();
}
