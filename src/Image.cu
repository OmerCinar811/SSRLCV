#include "hip/hip_runtime.h"
#include "Image.cuh"


__device__ __forceinline__ unsigned long getGlobalIdx_2D_1D(){
  unsigned long blockId = blockIdx.y * gridDim.x + blockIdx.x;
  unsigned long threadId = blockId * blockDim.x + threadIdx.x;
  return threadId;
}
__device__ __forceinline__ unsigned char bwaToBW(const uchar2 &color){
  return color.x;
}
__device__ __forceinline__ unsigned char rgbToBW(const uchar3 &color){
  return (color.x/4) + (color.y/2) + (color.z/4);
}
__device__ __forceinline__ unsigned char rgbaToBW(const uchar4 &color){
  return rgbToBW({color.x,color.y,color.z});
}

__global__ void generateBW(int numPixels, unsigned int colorDepth, unsigned char* colorPixels, unsigned char* pixels){
  unsigned long globalID = getGlobalIdx_2D_1D();
  if(globalID < numPixels){
    int numValues = colorDepth;
    switch(numValues){
      case 2:
        pixels[globalID] = bwaToBW({colorPixels[globalID*numValues],colorPixels[globalID*numValues + 1]});
        break;
      case 3:
        pixels[globalID] = rgbToBW({colorPixels[globalID*numValues],colorPixels[globalID*numValues + 1], colorPixels[globalID*numValues + 2]});
        break;
      case 4:
        pixels[globalID] = rgbToBW({colorPixels[globalID*numValues],colorPixels[globalID*numValues + 1], colorPixels[globalID*numValues + 2]});
        break;
      default:
        printf("ERROR colorDepth of %u is not supported\n",numValues);
        asm("trap;");
    }
  }
}

__device__ __host__ ssrlcv::Image_Descriptor::Image_Descriptor(){
  this->id = 0;
  this->size = {0,0};
  this->cam_vec = {0.0f,0.0f,0.0f};
  this->cam_pos = {0.0f,0.0f,0.0f};
  this->fov = 0;
  this->foc = 0;
  this->dpix = 0.0f;
}
__device__ __host__ ssrlcv::Image_Descriptor::Image_Descriptor(int id, uint2 size){
  this->id = id;
  this->size = size;
  this->cam_vec = {0.0f,0.0f,0.0f};
  this->cam_pos = {0.0f,0.0f,0.0f};
  this->fov = 0;
  this->foc = 0;
  this->dpix = 0.0f;
}
__device__ __host__ ssrlcv::Image_Descriptor::Image_Descriptor(int id, uint2 size, float3 cam_pos, float3 camp_dir){
  this->id = id;
  this->size = size;
  this->cam_pos = cam_pos;
  this->cam_vec = cam_vec;
  this->fov = 0;
  this->foc = 0;
  this->dpix = 0.0f;
}

void ssrlcv::get_cam_params2view(Image_Descriptor &cam1, Image_Descriptor &cam2, std::string infile){
  std::ifstream input(infile);
  std::string line;
  float res = 0.0f;
  while(std::getline(input, line)) {
    std::istringstream iss(line);
    std::string param;
    float arg1;
    float arg2;
    float arg3;
    iss >> param >> arg1;
    if(param.compare("foc") == 0) {
      cam1.foc = arg1;
      cam2.foc = arg1;
    }
    else if(param.compare("fov") == 0) {
      cam1.fov = arg1;
      cam2.fov = arg1;
    }
    else if(param.compare("res") == 0) {
      res = arg1;
    }
    else if(param.compare("cam1C") == 0) {
      iss >> arg2 >> arg3;
      cam1.cam_pos.x = arg1;
      cam1.cam_pos.y = arg2;
      cam1.cam_pos.z = arg3;
    }
    else if(param.compare("cam1V") == 0) {
      iss >> arg2 >> arg3;
      cam1.cam_vec.x = arg1;
      cam1.cam_vec.y = arg2;
      cam1.cam_vec.z = arg3;
    }
    else if(param.compare("cam2C") == 0) {
      iss >> arg2 >> arg3;
      cam2.cam_pos.x = arg1;
      cam2.cam_pos.y = arg2;
      cam2.cam_pos.z = arg3;
    }
    else if(param.compare("cam2V") == 0) {
      iss >> arg2 >> arg3;
      cam2.cam_vec.x = arg1;
      cam2.cam_vec.y = arg2;
      cam2.cam_vec.z = arg3;
    }
  }
  cam1.dpix = (cam1.foc*tan(cam1.fov/2))/(res/2);
  cam2.dpix = (cam2.foc*tan(cam2.fov/2))/(res/2);
}

ssrlcv::Image::Image(){
  this->colorDepth = 0;
  this->descriptor.id = -1;
  this->pixels = nullptr;
}

ssrlcv::Image::Image(std::string filePath, int id){
  this->filePath = filePath;

  unsigned char* pixels_host = readPNG(filePath.c_str(), this->descriptor.size.y, this->descriptor.size.x, this->colorDepth);
  this->pixels = new Unity<unsigned char>(pixels_host,this->descriptor.size.y*this->descriptor.size.x*this->colorDepth,cpu);

  this->descriptor.id = id;
}

ssrlcv::Image::~Image(){

}

void ssrlcv::Image::convertToBW(){
  if(this->colorDepth == 1){
    std::cout<<"Pixels are already bw"<<std::endl;
    return;
  }

  MemoryState origin = this->pixels->state;
  this->pixels->transferMemoryTo(gpu);

  unsigned int numPixels = (this->pixels->numElements/this->colorDepth);

  unsigned char* bwPixels_device;
  CudaSafeCall(hipMalloc((void**)&bwPixels_device, numPixels*sizeof(unsigned char)));

  dim3 grid;
  dim3 block;
  getFlatGridBlock(numPixels, grid, block);
  generateBW<<<grid,block>>>(numPixels, this->colorDepth, this->pixels->device, bwPixels_device);
  CudaCheckError();

  this->pixels->setData(bwPixels_device, numPixels, gpu);
  this->pixels->transferMemoryTo(origin);
  if(origin == cpu){
    this->pixels->clear(gpu);
  }
  this->colorDepth = 1;
}
