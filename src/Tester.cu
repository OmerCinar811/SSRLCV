#include "hip/hip_runtime.h"
#include "common_includes.h"
#include "Image.cuh"
#include "io_util.h"
#include "SIFT_FeatureFactory.cuh"
#include "MatchFactory.cuh"
#include "PointCloudFactory.cuh"
#include "MeshFactory.cuh"

int main(int argc, char *argv[]){
  try{
    //CUDA INITIALIZATION
    hipInit(0);
    clock_t totalTimer = clock();
    clock_t partialTimer = clock();

    std::cout << "=========================== TEST 01 ===========================" << std::endl;
    std::cout << "Creating Fake Images" << std::endl;
    std::vector<ssrlcv::Image*> images_vec;

    // If you want to test N many matched keypoints, you need at least N many fake cameras
    ssrlcv::Image* image0 = new ssrlcv::Image();
    ssrlcv::Image* image1 = new ssrlcv::Image();
    ssrlcv::Image* image2 = new ssrlcv::Image();
    ssrlcv::Image* image3 = new ssrlcv::Image();
    ssrlcv::Image* image4 = new ssrlcv::Image();
    images_vec.push_back(image0);
    images_vec.push_back(image1);
    images_vec.push_back(image2);
    images_vec.push_back(image3);
    images_vec.push_back(image4);

    // Test Camera Parameters
    // If you want to test N many matched keypoints, you need at least N many fake cameras
    // be sure that you index them correctly in the images_vec
    // be sure that the id matches the index
    std::cout << "Filling in Test Camera Params ..." << std::endl;
    images_vec[0]->id = 0;
    images_vec[0]->camera.size = {2,2};
    images_vec[0]->camera.cam_pos = {1.0,0.0,0.0};
    images_vec[0]->camera.cam_rot = {M_PI/2.0,0.0,0.0};
    images_vec[0]->camera.fov = {10.0 * (M_PI/180.0), 10.0 * (M_PI/180.0)}; // 10 degrees x and y fov
    images_vec[0]->camera.foc = 0.25;

    images_vec[1]->id = 1;
    images_vec[1]->camera.size = {2,2};
    images_vec[1]->camera.cam_pos = {-1.0,0.0,0.0};
    images_vec[1]->camera.cam_rot = {0.0, M_PI/2.0, 0.0};
    images_vec[1]->camera.fov = {10.0 * (M_PI/180.0), 10.0 * (M_PI/180.0)};
    images_vec[1]->camera.foc = 0.25;

    images_vec[2]->id = 2;
    images_vec[2]->camera.size = {2,2};
    images_vec[2]->camera.cam_pos = {0.0,0.0,1.0};
    images_vec[2]->camera.cam_rot = {0.0, 0.0, M_PI/4.0};
    images_vec[2]->camera.fov = {10.0 * (M_PI/180.0), 10.0 * (M_PI/180.0)};
    images_vec[2]->camera.foc = 0.25;

    images_vec[3]->id = 3;
    images_vec[3]->camera.size = {2,2};
    images_vec[3]->camera.cam_pos = {1.0,0.0,1.0};
    images_vec[3]->camera.cam_rot = {M_PI/3.0, M_PI/3.0, 0.0};
    images_vec[3]->camera.fov = {10.0 * (M_PI/180.0), 10.0 * (M_PI/180.0)};
    images_vec[3]->camera.foc = 0.25;

    images_vec[4]->id = 4;
    images_vec[4]->camera.size = {2,2};
    images_vec[4]->camera.cam_pos = {0.0,1.0,1.0};
    images_vec[4]->camera.cam_rot = {0.0, M_PI/3.0, M_PI/3.0};
    images_vec[4]->camera.fov = {10.0 * (M_PI/180.0), 10.0 * (M_PI/180.0)};
    images_vec[4]->camera.foc = 0.25;

    // Test Match Points
    std::cout << "Filling in Matches ..." << std::endl;
    ssrlcv::MatchSet matchSet;

    // lets say we want the following:
    //    4 sets of matches, so we need to have a count of that
    int matchesnum = 4;
    // then, in terms of our matches we want:
    //    * first match connects 2 keypoints
    //    * second match connects 3 keypoints
    //    * third match connets 2 keypoints
    //    * forth match connets 5 keypoints
    // we need to calcualte how many keypoitns that is, so:
    int keypointnum = 2 + 3 + 2 + 5;

    // next we need to allocate memory for these guys

    // matches contain groups of keypoints, which are just the R2 coordinates tha correspond
    // here is where we use matches num
    matchSet.matches = new ssrlcv::Unity<ssrlcv::MultiMatch>(nullptr,matchesnum,ssrlcv::cpu);
    // this is the list of R2 coodrindates
    // here we want to use the keypoint num
    matchSet.keyPoints = new ssrlcv::Unity<ssrlcv::KeyPoint>(nullptr,keypointnum,ssrlcv::cpu);

    // now we need to fill in what our matches are



    // note that   *->host[#] is the memory location of the match information, that is sequential in RAM and counts up
    matchSet.matches->host[0] = {2,0}; // here we say the number of matches and the starting index of those matches in the keypoints
                                       // that ends up looking like = {number keypoints in the match, where those keyPoints start}
                                       // for us we have 2 matches and the matches start at index 0
    // note that the *->host[#] is the memory location, that's sequential in RAM, so those always count up
    matchSet.keyPoints->host[0] = {{0}, {1.0, 1.0}}; // { {image number}, {x-y match location} }
    matchSet.keyPoints->host[1] = {{1}, {1.0, 1.0}};



    // note that   *->host[#] is the memory location of the match information, that is sequential in RAM and counts up
    matchSet.matches->host[1] = {3,2}; // here we say the number of matches and the starting index of those matches in the keypoints
                                       // that ends up looking like = {number keypoints in the match, where those keyPoints start}
                                       // for us we have 3 matches and the matches start at index 3
    // note that the *->host[#] is the memory location, that's sequential in RAM, so those always count up
    matchSet.keyPoints->host[2] = {{0}, {1.0, 1.0}};
    matchSet.keyPoints->host[3] = {{1}, {1.0, 1.0}};
    matchSet.keyPoints->host[4] = {{2}, {1.0, 1.0}};



    // note that   *->host[#] is the memory location of the match information, that is sequential in RAM and counts up
    matchSet.matches->host[2] = {2,5}; // here we say the number of matches and the starting index of those matches in the keypoints
                                       // that ends up looking like = {number keypoints in the match, where those keyPoints start}
                                       // for us we have 2 matches and the matches start at index 5
    // note that the *->host[#] is the memory location, that's sequential in RAM, so those always count up
                                       // let's say we want to match the last 2 images this time, let's use image indexes 3 and 4
    matchSet.keyPoints->host[5] = {{3}, {1.0, 1.0}}; // { {image number}, {x-y match location} }
    matchSet.keyPoints->host[6] = {{4}, {1.0, 1.0}};



    // note that   *->host[#] is the memory location of the match information, that is sequential in RAM and counts up
    matchSet.matches->host[3] = {5,7}; // here we say the number of matches and the starting index of those matches in the keypoints
                                       // that ends up looking like = {number keypoints in the match, where those keyPoints start}
                                       // for us we have 5 matches and the matches start at index 7
    // note that the *->host[#] is the memory location, that's sequential in RAM, so those always count up
    matchSet.keyPoints->host[7]  = {{0}, {1.0, 1.0}}; // { {image number}, {x-y match location} }
    matchSet.keyPoints->host[8]  = {{1}, {1.0, 1.0}};
    matchSet.keyPoints->host[9]  = {{2}, {1.0, 1.0}};
    matchSet.keyPoints->host[10] = {{3}, {1.0, 1.0}};
    matchSet.keyPoints->host[11] = {{4}, {1.0, 1.0}};

    // now we can try to make lines

    // Line Generation Test
    ssrlcv::PointCloudFactory demPoints = ssrlcv::PointCloudFactory();
    
    std::cout << "Bundles: " << std::endl;
    ssrlcv::BundleSet bundleSet = demPoints.generateBundles(&matchSet,images_vec);
    
    // prints out generated lines in point vector format
    std::cout << "<lines start>" << std::endl;
    for(int i = 0; i < bundleSet.bundles->numElements; i ++){
      for (int j = bundleSet.bundles->host[i].index; j < bundleSet.bundles->host[i].index + bundleSet.bundles->host[i].numLines; j++){
        std::cout << "(" << bundleSet.lines->host[j].pnt.x << "," << bundleSet.lines->host[j].pnt.y << "," << bundleSet.lines->host[j].pnt.z << ")    ";
        std::cout << "<" << bundleSet.lines->host[j].vec.x << "," << bundleSet.lines->host[j].vec.y << "," << bundleSet.lines->host[j].vec.z << ">" << std::endl;
      }
      std::cout << std::endl;
    }
    
    //N-View Point Cloud
    ssrlcv::Unity<float3> *pointcloud;
    pointcloud = demPoints.nViewTriangulate(bundleSet);
    ssrlcv::writePLY("out/test.ply",pointcloud);
  }
  catch (const std::exception &e){
    std::cerr << "Caught exception: " << e.what() << '\n';
    std::exit(1);
  }
  catch (...){
    std::cerr << "Caught unknown exception\n";
    std::exit(1);
  }
}
