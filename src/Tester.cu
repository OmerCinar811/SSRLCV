#include "hip/hip_runtime.h"

// _______________________________________________________________________________________________________________
//  _____/\\\\\\\\\\\_______/\\\\\\\\\\\______/\\\\\\\\\______/\\\____________________/\\\\\\\\\__/\\\________/\\\_
//   ___/\\\/////////\\\___/\\\/////////\\\__/\\\///////\\\___\/\\\_________________/\\\////////__\/\\\_______\/\\\_
//    __\//\\\______\///___\//\\\______\///__\/\\\_____\/\\\___\/\\\_______________/\\\/___________\//\\\______/\\\__
//     ___\////\\\___________\////\\\_________\/\\\\\\\\\\\/____\/\\\______________/\\\______________\//\\\____/\\\___
//      ______\////\\\___________\////\\\______\/\\\//////\\\____\/\\\_____________\/\\\_______________\//\\\__/\\\____
//       _________\////\\\___________\////\\\___\/\\\____\//\\\___\/\\\_____________\//\\\_______________\//\\\/\\\_____
//        __/\\\______\//\\\___/\\\______\//\\\__\/\\\_____\//\\\__\/\\\______________\///\\\______________\//\\\\\______
//         _\///\\\\\\\\\\\/___\///\\\\\\\\\\\/___\/\\\______\//\\\_\/\\\\\\\\\\\\\\\____\////\\\\\\\\\______\//\\\_______
//          ___\///////////_______\///////////_____\///________\///__\///////////////________\/////////________\///________
//           _______________________________________________________________________________________________________________



#include "common_includes.h"
#include "Image.cuh"
#include "io_util.h"
#include "SIFT_FeatureFactory.cuh"
#include "MatchFactory.cuh"
#include "PointCloudFactory.cuh"
#include "MeshFactory.cuh"

//TODO fix gaussian operators - currently creating very low values


int main(int argc, char *argv[]){
  try{

    //CUDA INITIALIZATION
    hipInit(0);
    clock_t totalTimer = clock();
    clock_t partialTimer = clock();

    //ARG PARSING

    std::map<std::string,ssrlcv::arg*> args = ssrlcv::parseArgs(argc,argv);
    if(args.find("dir") == args.end()){
      std::cerr<<"ERROR: SFM executable requires a directory of images"<<std::endl;
      exit(-1);
    }

    // fake 2 images

    std::vector<ssrlcv::Image*> images;
    ssrlcv::Image* image0 = new ssrlcv::Image();
    images.push_back(image0);
    ssrlcv::Image* image1 = new ssrlcv::Image();
    images.push_back(image1);

    // cube camera params

    images[0]->id = 0;
    images[0]->camera.size = {1024,1024};
    images[0]->camera.cam_pos = {0.000000000000,0.000000000000,-400.000000000000};
    images[0]->camera.cam_rot = {0.0, 0.0, 0.0};
    images[0]->camera.fov = {0.0593411945678,0.0593411945678};
    images[0]->camera.foc = 0.160000000000;
    images[1]->id = 1;
    images[1]->camera.size = {1024,1024};
    images[1]->camera.cam_pos = {0.000000000000,69.459271066772,-393.923101204883};
    images[1]->camera.cam_rot = {0.174532925199, 0.0, 0.0};
    images[1]->camera.fov = {0.0593411945678,0.0593411945678};
    images[1]->camera.foc = 0.160000000000;



    // fake 2-view cube

    ssrlcv::Match* matches_host = new ssrlcv::Match[9];
    ssrlcv::Unity<ssrlcv::Match>* matches = new ssrlcv::Unity<ssrlcv::Match>(matches_host, 9, ssrlcv::cpu);
    matches->host[0].keyPoints[0].parentId = 0;
    matches->host[0].keyPoints[0].loc = {468.764219112,555.235780888};
    matches->host[0].keyPoints[1].parentId = 1;
    matches->host[0].keyPoints[1].loc = {468.784672247,562.063052731};
    matches->host[1].keyPoints[0].parentId = 0;
    matches->host[1].keyPoints[0].loc = {555.235780888,555.235780888};
    matches->host[1].keyPoints[1].parentId = 1;
    matches->host[1].keyPoints[1].loc = {555.215327753,562.063052731};
    matches->host[2].keyPoints[0].parentId = 0;
    matches->host[2].keyPoints[0].loc = {555.235780888,468.764219112};
    matches->host[2].keyPoints[1].parentId = 1;
    matches->host[2].keyPoints[1].loc = {555.25295805,476.914948916};
    matches->host[3].keyPoints[0].parentId = 0;
    matches->host[3].keyPoints[0].loc = {468.764219112,468.764219112};
    matches->host[3].keyPoints[1].parentId = 1;
    matches->host[3].keyPoints[1].loc = {468.74704195,476.914948916};
    matches->host[4].keyPoints[0].parentId = 0;
    matches->host[4].keyPoints[0].loc = {468.979858917,555.020141083};
    matches->host[4].keyPoints[1].parentId = 1;
    matches->host[4].keyPoints[1].loc = {468.996851695,546.882415518};
    matches->host[5].keyPoints[0].parentId = 0;
    matches->host[5].keyPoints[0].loc = {555.020141083,555.020141083};
    matches->host[5].keyPoints[1].parentId = 1;
    matches->host[5].keyPoints[1].loc = {555.003148305,546.882415518};
    matches->host[6].keyPoints[0].parentId = 0;
    matches->host[6].keyPoints[0].loc = {555.020141083,468.979858917};
    matches->host[6].keyPoints[1].parentId = 1;
    matches->host[6].keyPoints[1].loc = {555.040409834,462.139581969};
    matches->host[7].keyPoints[0].parentId = 0;
    matches->host[7].keyPoints[0].loc = {468.979858917,468.979858917};
    matches->host[7].keyPoints[1].parentId = 1;
    matches->host[7].keyPoints[1].loc = {468.959590166,462.139581969};
    matches->host[8].keyPoints[0].parentId = 0;
    matches->host[8].keyPoints[0].loc = {512.0,512.0};
    matches->host[8].keyPoints[1].parentId = 1;
    matches->host[8].keyPoints[1].loc = {512.0,512.0};


    // center point tests
    /*
    ssrlcv::Match* matches_host = new ssrlcv::Match[1];
    ssrlcv::Unity<ssrlcv::Match>* matches = new ssrlcv::Unity<ssrlcv::Match>(matches_host, 1, ssrlcv::cpu);
    matches->host[0].keyPoints[0].parentId = 0;
    matches->host[0].keyPoints[0].loc = {512.0,512.0};
    matches->host[0].keyPoints[1].parentId = 1;
    matches->host[0].keyPoints[1].loc = {512.0,512.0};
    */

    //
    // 2 View Case
    //

    ssrlcv::MatchSet matchSet;
    matchSet.keyPoints = new ssrlcv::Unity<ssrlcv::KeyPoint>(nullptr,matches->size()*2,ssrlcv::cpu);
    matchSet.matches = new ssrlcv::Unity<ssrlcv::MultiMatch>(nullptr,matches->size(),ssrlcv::cpu);
    matches->setMemoryState(ssrlcv::cpu);
    matchSet.matches->setMemoryState(ssrlcv::cpu);
    matchSet.keyPoints->setMemoryState(ssrlcv::cpu);
    for(int i = 0; i < matchSet.matches->size(); i++){
      matchSet.keyPoints->host[i*2] = matches->host[i].keyPoints[0];
      matchSet.keyPoints->host[i*2 + 1] = matches->host[i].keyPoints[1];
      matchSet.matches->host[i] = {2,i*2};
    }
    std::cout << "Generated MatchSet ..." << std::endl << "Total Matches: " << matches->size() << std::endl << std::endl;

    // the point boi
    ssrlcv::PointCloudFactory demPoints = ssrlcv::PointCloudFactory();
    ssrlcv::Unity<float3>* points;
    ssrlcv::BundleSet bundleSet;


    //
    // 2-view test
    //

    std::cout << "Attempting 2-view Triangulation" << std::endl;

    float* linearError = (float*)malloc(sizeof(float));

    bundleSet = demPoints.generateBundles(&matchSet,images);
    points = demPoints.twoViewTriangulate(bundleSet, linearError);

    std::cout << "initial linearError: " << std::fixed << std::setprecision(12) << *linearError << std::endl;
    std::cout << "\t writing initial PLY ..." << std::endl;
    demPoints.saveDebugCloud(points, bundleSet, images, "initial");

    /*
    // Test SENSITIVITY
    std::string temp_filename1 = "pre-noise";
    demPoints.generateSensitivityFunctions(&matchSet,images,temp_filename1);
    */

    /*
    // add some noisey stuff to the image to see the heck is up
    images[1]->camera.cam_pos.x += 1.0;
    images[1]->camera.cam_pos.y += 0.4;
    images[1]->camera.cam_pos.z -= 0.7;
    images[1]->camera.cam_rot.x += (PI/16);
    images[1]->camera.cam_rot.y -= (PI/20);
    images[1]->camera.cam_rot.z += (PI/40);

    // Test SENSITIVITY
    std::string temp_filename2 = "post-noise";
    demPoints.generateSensitivityFunctions(&matchSet,images,temp_filename2);
    */



    //
    // now start a test of bundle adjustment
    //

    // Save for a before and after:
    // this temp vector is only used for the +/- h steps when calculating the gradients
    std::vector<ssrlcv::Image*> temp;
    for (int i = 0; i < images.size(); i++){
      ssrlcv::Image* t = new ssrlcv::Image();
      t->camera = images[i]->camera;
      temp.push_back(t); // fill in the initial images
    }

    // start by messing up the initial paramters
    // test moving the camera slightly
    float noise[6] = {1.0,  1.0,  1.0,  0.001,  0.001,  0.001};
    images[1]->camera.cam_pos.x += noise[0];
    images[1]->camera.cam_pos.y += noise[1];
    images[1]->camera.cam_pos.z += noise[2];
    images[1]->camera.cam_rot.x += noise[3];
    images[1]->camera.cam_rot.y += noise[4];
    images[1]->camera.cam_rot.z += noise[5];
    bundleSet = demPoints.generateBundles(&matchSet,images);
    points = demPoints.twoViewTriangulate(bundleSet, linearError);
    std::cout << "simulated with noise linearError: " << std::fixed << std::setprecision(12) << *linearError << std::endl;
    std::cout << "\t writing noisy PLY ..." << std::endl;
    demPoints.saveDebugCloud(points, bundleSet, images, "noisey");

    std::cout << "Starting Bundle Adjustment Loop ..." << std::endl;
    // now start the bundle adjustment 2-view loop
    points = demPoints.BundleAdjustTwoView(&matchSet,images, 3);
    // points = demPoints.twoViewTriangulate(bundleSet, linearError); // one last time!
    // std::cout << "final adjusted cloud has linearError: " << *linearError << std::endl;
    std::cout << "\t writing adjusted PLY ..." << std::endl;
    demPoints.saveDebugCloud(points, bundleSet, images, "adjusted");

    // print off the befores and afters of image params
    for (int i = 0; i < images.size(); i++){
      std::cout << "Cam " << i << " locations:" << std::endl;
      std::cout << std::fixed << std::setprecision(20);
      std::cout << "[" << temp[i]->camera.cam_pos.x << ", " << temp[i]->camera.cam_pos.y << ", " << temp[i]->camera.cam_pos.z << ", ";
      std::cout << temp[i]->camera.cam_rot.x << ", " << temp[i]->camera.cam_rot.y << ", " << temp[i]->camera.cam_rot.z << " ]" << std::endl;
      if (i == 1){
        std::cout << "[" << (temp[i]->camera.cam_pos.x + noise[0]) << ", " << (temp[i]->camera.cam_pos.y + noise[1]) << ", " << (temp[i]->camera.cam_pos.z + noise[2]) << ", ";
        std::cout << (temp[i]->camera.cam_rot.x + noise[3])<< ", " << (temp[i]->camera.cam_rot.y + noise[4]) << ", " << (temp[i]->camera.cam_rot.z + noise[5])<< " ]" << std::endl;
      } else {
        std::cout << "[" << (temp[i]->camera.cam_pos.x) << ", " << (temp[i]->camera.cam_pos.y) << ", " << temp[i]->camera.cam_pos.z << ", ";
        std::cout << temp[i]->camera.cam_rot.x << ", " << temp[i]->camera.cam_rot.y << ", " << temp[i]->camera.cam_rot.z << " ]" << std::endl;
      }
      std::cout << "[" << images[i]->camera.cam_pos.x << ", " << images[i]->camera.cam_pos.y << ", " << images[i]->camera.cam_pos.z << " ";
      std::cout << images[i]->camera.cam_rot.x << ", " << images[i]->camera.cam_rot.y << ", " << images[i]->camera.cam_rot.z << " ]" << std::endl << std::endl;
    }

    ssrlcv::Unity<float>* diff1 = temp[0]->getExtrinsicDifference(temp[1]->camera);
    ssrlcv::Unity<float>* diff2 = images[0]->getExtrinsicDifference(images[1]->camera);

    std::cout << std::endl << "Goal:" << std::cout;
    for (int i = 0; i < diff1->size(); i++){
      std::cout << diff1->host[i] << "  ";
    }
    std::cout << std::endl << "Result:" << std::cout;
    for (int i = 0; i < diff2->size(); i++){
      std::cout << diff2->host[i] << "  ";
    }

    // cleanup
    delete points;
    delete matches;
    delete matchSet.matches;
    delete matchSet.keyPoints;
    delete bundleSet.bundles;
    delete bundleSet.lines;
    // for(int i = 0; i < imagePaths.size(); ++i){
    //   delete images[i];
    //   delete allFeatures[i];
    // }

    return 0;
  }
  catch (const std::exception &e){
      std::cerr << "Caught exception: " << e.what() << '\n';
      std::exit(1);
  }
  catch (...){
      std::cerr << "Caught unknown exception\n";
      std::exit(1);
  }

}

























































// yeet
