#include "hip/hip_runtime.h"
#include "common_includes.h"
#include "Image.cuh"
#include "io_util.h"
#include "SIFT_FeatureFactory.cuh"
#include "MatchFactory.cuh"
#include "PointCloudFactory.cuh"
#include "MeshFactory.cuh"

//TODO fix gaussian operators - currently creating very low values

int main(int argc, char *argv[]){
  try{
    //CUDA INITIALIZATION
    hipInit(0);
    clock_t totalTimer = clock();
    clock_t partialTimer = clock();


    // test bundle adjustment here

    // ====================== FOR MANUAL TESTING

    std::cout << "=========================== TEST 01 ===========================" << std::endl;
    std::cout << "Making fake image guys ..." << std::endl;
    std::vector<ssrlcv::Image*> images;

    ssrlcv::Image* image0 = new ssrlcv::Image();
    ssrlcv::Image* image1 = new ssrlcv::Image();
    images.push_back(image0);
    images.push_back(image1);

    // fill the test camera params
    std::cout << "Filling in Test Camera Params ..." << std::endl;

    // ===== IF loading real image stuff

    std::map<std::string,ssrlcv::arg*> args = ssrlcv::parseArgs(argc,argv);
    if(args.find("dir") == args.end()){
      std::cerr<<"ERROR: SFM executable requires a directory of images"<<std::endl;
      exit(-1);
    }

    std::vector<std::string> imagePaths = ((ssrlcv::img_dir_arg*)args["dir"])->paths;
    int numImages = (int) imagePaths.size();
    std::cout<<"found "<<numImages<<" in directory given"<<std::endl;

    std::vector<ssrlcv::Image*> images;
    std::vector<ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>*> allFeatures;
    for(int i = 0; i < numImages; ++i){
      ssrlcv::Image* image = new ssrlcv::Image(imagePaths[i],i);
      ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>* features = featureFactory.generateFeatures(image,false,2,0.8);
      features->transferMemoryTo(ssrlcv::cpu);
      images.push_back(image);
      allFeatures.push_back(features);
    }

    //  X        Y      Z        rot_x    rot_y      rot_z    fov_x           fov_y         foc  dpix  dpix   time        x   y
    // 781.417, 0.0,  4436.30,  0.0,  0.1745329252, 0.0,  0.19933754453,  0.19933754453,  0.16, 0.4,  0.4,  1580766557, 1024,1024,
    // 0.0,     0.0,  4500.0,   0.0,  0.0,          0.0,  0.19933754453,  0.19933754453,  0.16, 0.4,  0.4,  1580766557, 1024,1024,


    // images[0]->id = 0;
    // images[0]->camera.size = {2,2};
    // images[0]->camera.cam_pos = {0.000000000000,0.000000000000,-100.000000000000};
    // images[0]->camera.cam_rot = {0.0, 0.0, 0.0};
    // images[0]->camera.fov = {0.174532925199,0.174532925199};
    // images[0]->camera.foc = 0.000025000000;
    // images[1]->id = 1;
    // images[1]->camera.size = {2,2};
    // images[1]->camera.cam_pos = {0.000000000000,100.000000000000,-0.000000000000};
    // images[1]->camera.cam_rot = {1.57079632679, 0.0, 0.0};
    // images[1]->camera.fov = {0.174532925199,0.174532925199};
    // images[1]->camera.foc = 0.000025000000;

    // fill the test match points
    std::cout << "Filling in Matches ..." << std::endl;

    ssrlcv::Match* matches_host = new ssrlcv::Match[9];
    ssrlcv::Unity<ssrlcv::Match>* matches = new ssrlcv::Unity<ssrlcv::Match>(matches_host, 9, ssrlcv::cpu);
    matches->host[0].keyPoints[0].parentId = 0;
    matches->host[0].keyPoints[1].parentId = 1;
    matches->host[0].keyPoints[0].loc = {513.138029297,513.138029297};
    matches->host[0].keyPoints[1].loc = {512.923155875,512.923155875};
    matches->host[1].keyPoints[0].parentId = 0;
    matches->host[1].keyPoints[1].parentId = 1;
    matches->host[1].keyPoints[0].loc = {513.138029297,513.138029297};
    matches->host[1].keyPoints[1].loc = {512.923155875,512.923155875};
    matches->host[2].keyPoints[0].parentId = 0;
    matches->host[2].keyPoints[1].parentId = 1;
    matches->host[2].keyPoints[0].loc = {510.861970703,510.861970703};
    matches->host[2].keyPoints[1].loc = {510.681698546,510.681698546};
    matches->host[3].keyPoints[0].parentId = 0;
    matches->host[3].keyPoints[1].parentId = 1;
    matches->host[3].keyPoints[0].loc = {510.861970703,510.861970703};
    matches->host[3].keyPoints[1].loc = {510.681698546,510.681698546};
    matches->host[4].keyPoints[0].parentId = 0;
    matches->host[4].keyPoints[1].parentId = 1;
    matches->host[4].keyPoints[0].loc = {513.137523618,513.137523618};
    matches->host[4].keyPoints[1].loc = {513.317826272,513.317826272};
    matches->host[5].keyPoints[0].parentId = 0;
    matches->host[5].keyPoints[1].parentId = 1;
    matches->host[5].keyPoints[0].loc = {513.137523618,513.137523618};
    matches->host[5].keyPoints[1].loc = {513.317826272,513.317826272};
    matches->host[6].keyPoints[0].parentId = 0;
    matches->host[6].keyPoints[1].parentId = 1;
    matches->host[6].keyPoints[0].loc = {510.862476382,510.862476382};
    matches->host[6].keyPoints[1].loc = {511.077319307,511.077319307};
    matches->host[7].keyPoints[0].parentId = 0;
    matches->host[7].keyPoints[1].parentId = 1;
    matches->host[7].keyPoints[0].loc = {510.862476382,510.862476382};
    matches->host[7].keyPoints[1].loc = {511.077319307,511.077319307};
    matches->host[8].keyPoints[0].parentId = 0;
    matches->host[8].keyPoints[1].parentId = 1;
    matches->host[8].keyPoints[0].loc = {512.0,512.0};
    matches->host[8].keyPoints[1].loc = {512.0,512.0};

    // ====================== END FOR MANUAL TESTING

    // ====================== FOR TESTING WITH INPUT IMAGES

    // std::map<std::string,ssrlcv::arg*> args = ssrlcv::parseArgs(argc,argv);
    // if(args.find("dir") == args.end()){
    //   std::cerr<<"ERROR: SFM executable requires a directory of images"<<std::endl;
    //   exit(-1);
    // }
    // ssrlcv::SIFT_FeatureFactory featureFactory = ssrlcv::SIFT_FeatureFactory(1.5f,6.0f);
    // ssrlcv::MatchFactory<ssrlcv::SIFT_Descriptor> matchFactory = ssrlcv::MatchFactory<ssrlcv::SIFT_Descriptor>(0.6f,250.0f*250.0f);
    // bool seedProvided = false;
    // ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>* seedFeatures = nullptr;
    // if(args.find("seed") != args.end()){
    //   seedProvided = true;
    //   std::string seedPath = ((ssrlcv::img_arg*)args["seed"])->path;
    //   ssrlcv::Image* seed = new ssrlcv::Image(seedPath,-1);
    //   seedFeatures = featureFactory.generateFeatures(seed,false,2,0.8);
    //   matchFactory.setSeedFeatures(seedFeatures);
    //   delete seed;
    // }
    // std::vector<std::string> imagePaths = ((ssrlcv::img_dir_arg*)args["dir"])->paths;
    // int numImages = (int) imagePaths.size();
    // std::cout<<"found "<<numImages<<" in directory given"<<std::endl;
    //
    // std::vector<ssrlcv::Image*> images;
    // std::vector<ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>*> allFeatures;
    // for(int i = 0; i < numImages; ++i){
    //   ssrlcv::Image* image = new ssrlcv::Image(imagePaths[i],i);
    //   ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>* features = featureFactory.generateFeatures(image,false,2,0.8);
    //   features->transferMemoryTo(ssrlcv::cpu);
    //   images.push_back(image);
    //   allFeatures.push_back(features);
    // }
    //
    // /*
    // MATCHING
    // */
    // //seeding with false photo
    //
    // std::cout << "Starting matching..." << std::endl;
    // ssrlcv::Unity<float>* seedDistances = (seedProvided) ? matchFactory.getSeedDistances(allFeatures[0]) : nullptr;
    // ssrlcv::Unity<ssrlcv::DMatch>* distanceMatches = matchFactory.generateDistanceMatches(images[0],allFeatures[0],images[1],allFeatures[1],seedDistances);
    // if(seedDistances != nullptr) delete seedDistances;
    //
    // distanceMatches->transferMemoryTo(ssrlcv::cpu);
    // float maxDist = 0.0f;
    // for(int i = 0; i < distanceMatches->size(); ++i){
    //   if(maxDist < distanceMatches->host[i].distance) maxDist = distanceMatches->host[i].distance;
    // }
    // printf("max euclidean distance between features = %f\n",maxDist);
    // if(distanceMatches->getMemoryState() != ssrlcv::gpu) distanceMatches->setMemoryState(ssrlcv::gpu);
    // ssrlcv::Unity<ssrlcv::Match>* matches = matchFactory.getRawMatches(distanceMatches);
    // delete distanceMatches;
    // std::string delimiter = "/";
    // std::string matchFile = imagePaths[0].substr(0,imagePaths[0].rfind(delimiter)) + "/matches.txt";
    // ssrlcv::writeMatchFile(matches, matchFile);

    // ====================== FOR TESTING WITH INPUT IMAGES

    // HARD CODED FOR 2 VIEW
    // Need to fill into to MatchSet boi
    std::cout << "Generating MatchSet ..." << std::endl;
    ssrlcv::MatchSet matchSet;
    matchSet.keyPoints = new ssrlcv::Unity<ssrlcv::KeyPoint>(nullptr,matches->size()*2,ssrlcv::cpu);
    matchSet.matches = new ssrlcv::Unity<ssrlcv::MultiMatch>(nullptr,matches->size(),ssrlcv::cpu);
    matches->setMemoryState(ssrlcv::cpu);
    for(int i = 0; i < matchSet.matches->size(); i++){
      matchSet.keyPoints->host[i*2] = matches->host[i].keyPoints[0];
      matchSet.keyPoints->host[i*2 + 1] = matches->host[i].keyPoints[1];
      matchSet.matches->host[i] = {2,i*2};
    }
    std::cout << "Generated MatchSet ..." << std::endl << "Total Matches: " << matches->size() << std::endl << std::endl;

    // start testing reprojection
    ssrlcv::PointCloudFactory demPoints = ssrlcv::PointCloudFactory();

    // //match interpolation method will take the place of this here.
    // ssrlcv::MatchSet matchSet;
    // matchSet.keyPoints = new ssrlcv::Unity<ssrlcv::KeyPoint>(nullptr,matches->size()*2,ssrlcv::cpu);
    // matchSet.matches = new ssrlcv::Unity<ssrlcv::MultiMatch>(nullptr,matches->size(),ssrlcv::cpu);
    // for(int i = 0; i < matches->size(); ++i){
    //   matchSet.keyPoints->host[i*2] = matches->host[i].keyPoints[0];
    //   matchSet.keyPoints->host[i*2 + 1] = matches->host[i].keyPoints[1];
    //   matchSet.matches->host[i] = {2,i*2};
    // }

    // test the prefect case
    std::cout << "Testing perfect case ..." << std::endl;

    ssrlcv::Unity<float>* errors       = new ssrlcv::Unity<float>(nullptr,matchSet.matches->size(),ssrlcv::cpu);
    float* linearError                 = (float*) malloc(sizeof(float));
    float* linearErrorCutoff           = (float*) malloc(sizeof(float));
    *linearError                       = 0;
    *linearErrorCutoff                 = 9001;
    ssrlcv::BundleSet bundleSet        = demPoints.generateBundles(&matchSet,images);
    ssrlcv::Unity<float3>* test_points = demPoints.twoViewTriangulate(bundleSet, errors, linearError, linearErrorCutoff);

    std::cout << "<lines start>" << std::endl;
    for(int i = 0; i < bundleSet.bundles->size(); i ++){
      for (int j = bundleSet.bundles->host[i].index; j < bundleSet.bundles->host[i].index + bundleSet.bundles->host[i].numLines; j++){
        std::cout << "(" << bundleSet.lines->host[j].pnt.x << "," << bundleSet.lines->host[j].pnt.y << "," << bundleSet.lines->host[j].pnt.z << ")\t\t";
        std::cout << "<" << bundleSet.lines->host[j].vec.x << "," << bundleSet.lines->host[j].vec.y << "," << bundleSet.lines->host[j].vec.z << ">" << std::endl;
      }
      std::cout << std::endl;
    }
    std::cout << "</lines end>" << std::endl;

    // std::cout << "Prefect points:" << std::endl;
    // //std::cout << "\t( " << test_point->host[0].x << ",  " << test_point->host[0].y << ", " << test_point->host[0].z << " )" << std::endl;
    // //std::cout << "\t( " << test_point->host[1].x << ",  " << test_point->host[1].y << ", " << test_point->host[1].z << " )" << std::endl;
    // std::cout << "\tLinear Error: " << *linearError << std::endl;

    ssrlcv::writePLY("out/test_points.ply",test_points);

    // a test of the new PLY writer
    // int colorPoint_size = 10;
    // // struct colorPoint* cpoints = (colorPoint*)  malloc(colorPoint_size * sizeof(struct colorPoint));
    // ssrlcv::Unity<colorPoint>* cpoints = new ssrlcv::Unity<float>(nullptr,matchSet.matches->size(),ssrlcv::cpu);
    // for (int k = 0; k < colorPoint_size; k++){
    //   //cpoints[k] = {(float) k,1.0,1.0,0,255,32};
    //   cpoints[k].x = (float) k;
    //   cpoints[k].y = (float) 0.0;
    //   cpoints[k].z = (float) 0.0;
    //   cpoints[k].r = 0;
    //   cpoints[k].g = 255;
    //   cpoints[k].b = 32;
    // }
    // std::string fname = "colorPointTest";
    // ssrlcv::writePLY("fuck", cpoints, colorPoint_size);

    std::cout << "Cube Points: " << std::endl;
    for (int i = 0; i < test_points->size(); i++){
      std::cout << "\t(" << test_points->host[i].x << "," << test_points->host[i].y << "," << test_points->host[i].z << ")" << std::endl;
    }

    // test output of all the boiz
    // ssrlcv::Unity<colorPoint>* cpoints = new ssrlcv::Unity<float>(nullptr, bundleSet.bundles->size() + test_points->size() + 2,ssrlcv::cpu);
    int colorPoint_size = test_points->size() + 2*bundleSet.lines->size() + 2;
    struct colorPoint* cpoints = (colorPoint*)  malloc(colorPoint_size * sizeof(struct colorPoint));
    // fill in the camera points
    cpoints[0].x = images[0]->camera.cam_pos.x;
    cpoints[0].y = images[0]->camera.cam_pos.y;
    cpoints[0].z = images[0]->camera.cam_pos.z;
    cpoints[0].r = 255;
    cpoints[0].g = 32;
    cpoints[0].b = 32;
    cpoints[1].x = images[1]->camera.cam_pos.x;
    cpoints[1].y = images[1]->camera.cam_pos.y;
    cpoints[1].z = images[1]->camera.cam_pos.z;
    cpoints[1].r = 255;
    cpoints[1].g = 32;
    cpoints[1].b = 32;
    // fill in the first bundles
    for (int i = 2; i < bundleSet.lines->size() + 2; i++){
      std::cout << "bundle point index: " << i - 2 << "\ti: "<< i << std::endl;
      cpoints[i].x = bundleSet.lines->host[i - 2].pnt.x;
      cpoints[i].y = bundleSet.lines->host[i - 2].pnt.y;
      cpoints[i].z = bundleSet.lines->host[i - 2].pnt.z;
      cpoints[i].r = 0;
      cpoints[i].g = 255;
      cpoints[i].b = 10;
    }
    // fill in the vectors
    for (int i = bundleSet.lines->size() + 2; i < 2*bundleSet.lines->size() + 2; i++) {
      std::cout << "bundle vector index: " << i - bundleSet.lines->size() - 2 << "\ti: " << i<< std::endl;
      cpoints[i].x = bundleSet.lines->host[i -  bundleSet.lines->size() - 2].pnt.x + bundleSet.lines->host[i -  bundleSet.lines->size() - 2].vec.x;
      cpoints[i].y = bundleSet.lines->host[i -  bundleSet.lines->size() - 2].pnt.y + bundleSet.lines->host[i -  bundleSet.lines->size() - 2].vec.y;
      cpoints[i].z = bundleSet.lines->host[i -  bundleSet.lines->size() - 2].pnt.z + bundleSet.lines->host[i -  bundleSet.lines->size() - 2].vec.z;
      cpoints[i].r = 30;
      cpoints[i].g = 10;
      cpoints[i].b = 127;
    }
    // fill in the point cloud
    for (int i = 2*bundleSet.lines->size() + 2; i < test_points->size() + 2*bundleSet.lines->size() + 2; i++){
      std::cout << "point clound index: " << i - 2*bundleSet.lines->size() - 2 << "\ti: " << i << std::endl;
      cpoints[i].x = test_points->host[i - 2*bundleSet.lines->size() - 2].x;
      cpoints[i].y = test_points->host[i - 2*bundleSet.lines->size() - 2].y;
      cpoints[i].z = test_points->host[i - 2*bundleSet.lines->size() - 2].z;
      cpoints[i].r = 0;
      cpoints[i].g = 32;
      cpoints[i].b = 255;
    }
    // now save it
    ssrlcv::writePLY("cubeTest", cpoints, colorPoint_size);

    // // add some random errors into the camera stuff
    // std::vector<ssrlcv::Image*> images_vec_err;
    //
    // ssrlcv::Image* image0_err = new ssrlcv::Image();
    // ssrlcv::Image* image1_err = new ssrlcv::Image();
    // images_vec_err.push_back(image0_err);
    // images_vec_err.push_back(image1_err);
    //
    // std::default_random_engine generator;
    // std::normal_distribution<float> distribution(0.0,0.00001);
    //
    // // std::cout << "Sample Errors to add:" << std::endl;
    // // for (int i = 0; i < 5; i ++){
    // //   float n = distribution(generator);
    // //   std::cout << n << ", ";
    // // }
    // // std::cout << std::endl;
    //
    // // addint noise to camera
    // std::cout << "Filling in Test Camera Params ..." << std::endl;
    // images_vec_err[0]->id = images_vec[0]->id;
    // images_vec_err[0]->camera.size = images_vec[0]->camera.size;
    // float3 err0 = {0.0001,0.0,0.0};
    // images_vec_err[0]->camera.cam_pos = images_vec[0]->camera.cam_pos + err0;
    // images_vec_err[0]->camera.cam_rot = images_vec[0]->camera.cam_rot;
    // images_vec_err[0]->camera.fov = images_vec[0]->camera.fov;
    // images_vec_err[0]->camera.foc = images_vec[0]->camera.foc;
    //
    // images_vec_err[1]->id = images_vec[1]->id;
    // images_vec_err[1]->camera.size = images_vec[1]->camera.size;
    // images_vec_err[1]->camera.cam_pos = images_vec[1]->camera.cam_pos;
    // float3 err1 = {0.0000001,0.0,0.0};
    // images_vec_err[1]->camera.cam_rot = images_vec[1]->camera.cam_rot + err1;
    // images_vec_err[1]->camera.fov = images_vec[1]->camera.fov;
    // images_vec_err[1]->camera.foc = images_vec[1]->camera.foc;
    //
    // // test the prefect case
    // std::cout << "Testing error case ..." << std::endl;
    //
    // ssrlcv::Unity<float>* errors_err      = new ssrlcv::Unity<float>(nullptr,matchSet.matches->size(),ssrlcv::cpu);
    // float* linearError_err                = (float*) malloc(sizeof(float));
    // float* linearErrorCutoff_err          = (float*) malloc(sizeof(float));
    // *linearError_err                      = 0;
    // *linearErrorCutoff_err                = 9001;
    // ssrlcv::BundleSet bundleSet_err       = demPoints.generateBundles(&matchSet,images_vec_err);
    // ssrlcv::Unity<float3>* test_point_err = demPoints.twoViewTriangulate(bundleSet_err, errors_err, linearError_err, linearErrorCutoff_err);
    //
    // std::cout << "Errored points:" << std::endl;
    // //std::cout << "\t( " << test_point_err->host[0].x << ",  " << test_point_err->host[0].y << ", " << test_point_err->host[0].z << " )" << std::endl;
    // //std::cout << "\t( " << test_point_err->host[1].x << ",  " << test_point_err->host[1].y << ", " << test_point_err->host[1].z << " )" << std::endl;
    // std::cout << "\tLinear Error: " << *linearError_err << std::endl;



    //std::cout << "Attempting Bundle Adjustment ..." << std::endl;
    // ssrlcv::Unity<float3>* bundleAdjustedPoints = demPoints.BundleAdjustTwoView(&matchSet,images_vec_err);

    //ARG PARSING

    // // ========================== REAL BUNDLE ADJUSTMENT ATTEMPT START
    // std::map<std::string,ssrlcv::arg*> args = ssrlcv::parseArgs(argc,argv);
    // if(args.find("dir") == args.end()){
    //   std::cerr<<"ERROR: SFM executable requires a directory of images"<<std::endl;
    //   exit(-1);
    // }
    // ssrlcv::SIFT_FeatureFactory featureFactory = ssrlcv::SIFT_FeatureFactory(1.5f,6.0f);
    // ssrlcv::MatchFactory<ssrlcv::SIFT_Descriptor> matchFactory = ssrlcv::MatchFactory<ssrlcv::SIFT_Descriptor>(0.6f,250.0f*250.0f);
    // bool seedProvided = false;
    // ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>* seedFeatures = nullptr;
    // if(args.find("seed") != args.end()){
    //   seedProvided = true;
    //   std::string seedPath = ((ssrlcv::img_arg*)args["seed"])->path;
    //   ssrlcv::Image* seed = new ssrlcv::Image(seedPath,-1);
    //   seedFeatures = featureFactory.generateFeatures(seed,false,2,0.8);
    //   matchFactory.setSeedFeatures(seedFeatures);
    //   delete seed;
    // }
    // std::vector<std::string> imagePaths = ((ssrlcv::img_dir_arg*)args["dir"])->paths;
    // int numImages = (int) imagePaths.size();
    // std::cout<<"found "<<numImages<<" in directory given"<<std::endl;
    //
    // std::vector<ssrlcv::Image*> images;
    // std::vector<ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>*> allFeatures;
    // for(int i = 0; i < numImages; ++i){
    //   ssrlcv::Image* image = new ssrlcv::Image(imagePaths[i],i);
    //   ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>* features = featureFactory.generateFeatures(image,false,2,0.8);
    //   features->transferMemoryTo(ssrlcv::cpu);
    //   images.push_back(image);
    //   allFeatures.push_back(features);
    // }
    //
    // /*
    // MATCHING
    // */
    // //seeding with false photo
    //
    // std::cout << "Starting matching..." << std::endl;
    // ssrlcv::Unity<float>* seedDistances = (seedProvided) ? matchFactory.getSeedDistances(allFeatures[0]) : nullptr;
    // ssrlcv::Unity<ssrlcv::DMatch>* distanceMatches = matchFactory.generateDistanceMatches(images[0],allFeatures[0],images[1],allFeatures[1],seedDistances);
    // if(seedDistances != nullptr) delete seedDistances;
    //
    // distanceMatches->transferMemoryTo(ssrlcv::cpu);
    // float maxDist = 0.0f;
    // for(int i = 0; i < distanceMatches->size(); ++i){
    //   if(maxDist < distanceMatches->host[i].distance) maxDist = distanceMatches->host[i].distance;
    // }
    // printf("max euclidean distance between features = %f\n",maxDist);
    // if(distanceMatches->getMemoryState() != ssrlcv::gpu) distanceMatches->setMemoryState(ssrlcv::gpu);
    // ssrlcv::Unity<ssrlcv::Match>* matches = matchFactory.getRawMatches(distanceMatches);
    // delete distanceMatches;
    // std::string delimiter = "/";
    // std::string matchFile = imagePaths[0].substr(0,imagePaths[0].rfind(delimiter)) + "/matches.txt";
    // ssrlcv::writeMatchFile(matches, matchFile);
    //
    // // HARD CODED FOR 2 VIEW
    // // Need to fill into to MatchSet boi
    // std::cout << "Generating MatchSet ..." << std::endl;
    // ssrlcv::MatchSet matchSet;
    // matchSet.keyPoints = new ssrlcv::Unity<ssrlcv::KeyPoint>(nullptr,matches->size()*2,ssrlcv::cpu);
    // matchSet.matches = new ssrlcv::Unity<ssrlcv::MultiMatch>(nullptr,matches->size(),ssrlcv::cpu);
    // matches->setMemoryState(ssrlcv::cpu);
    // for(int i = 0; i < matchSet.matches->size(); i++){
    //   matchSet.keyPoints->host[i*2] = matches->host[i].keyPoints[0];
    //   matchSet.keyPoints->host[i*2 + 1] = matches->host[i].keyPoints[1];
    //   matchSet.matches->host[i] = {2,i*2};
    // }
    // std::cout << "Generated MatchSet ..." << std::endl << "Total Matches: " << matches->size() << std::endl << std::endl;
    //
    // /*
    // attempted bundle adjustment
    // */
    //
    // ssrlcv::PointCloudFactory pc = ssrlcv::PointCloudFactory();
    //
    //
    // ssrlcv::Unity<float3>* points = pc.BundleAdjustTwoView(&matchSet,images);
    //
    // ssrlcv::writePLY("out/bundleAdjustedPoints.ply",points);
    // // points->clear();
    // // ========================== REAL BUNDLE ADJUSTMENT ATTEMPT START


    /*
    2 View Reprojection
    */
    // ssrlcv::PointCloudFactory demPoints = ssrlcv::PointCloudFactory();
    //
    // // bunlde adjustment loop would be here. images_vec woudl be modified to minimize the boi
    // unsigned long long int* linearError = (unsigned long long int*) malloc(sizeof(unsigned long long int));
    // float* linearErrorCutoff = (float*) malloc(sizeof(float));
    // ssrlcv::BundleSet bundleSet = demPoints.generateBundles(&matchSet,images);
    //
    // // the version that will be used normally
    // ssrlcv::Unity<float3>* points = demPoints.twoViewTriangulate(bundleSet, linearError);
    // std::cout << "Total Linear Error: " << *linearError << std::endl;
    //
    // // here is a version that will give me individual linear errors
    // ssrlcv::Unity<float>* errors = new ssrlcv::Unity<float>(nullptr,matches->numElements,ssrlcv::cpu);
    // *linearErrorCutoff = 620.0;
    // ssrlcv::Unity<float3>* points2 = demPoints.twoViewTriangulate(bundleSet, errors, linearError, linearErrorCutoff);
    // // then I write them to a csv to see what to heck is goin on
    // ssrlcv::writeCSV(errors->host, (int) errors->numElements, "individualLinearErrors");

    // optional stereo disparity here
    // /*
    // STEREODISPARITY
    // */
    // ssrlcv::PointCloudFactory demPoints = ssrlcv::PointCloudFactory();
    // ssrlcv::Unity<float3>* points = demPoints.stereo_disparity(matches,8.0);
    //

    // delete matches;
    // ssrlcv::writePLY("out/unfiltered.ply",points);
    // delete points;
    // ssrlcv::writePLY("out/filtered.ply",points2);
    // delete points2;

    // clean up the images
    // for(int i = 0; i < imagePaths.size(); ++i){
    //   delete images[i];
    //   delete allFeatures[i];
    // }

    return 0;
  }
  catch (const std::exception &e){
      std::cerr << "Caught exception: " << e.what() << '\n';
      std::exit(1);
  }
  catch (...){
      std::cerr << "Caught unknown exception\n";
      std::exit(1);
  }

}

























































// yeet
