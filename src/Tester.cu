#include "hip/hip_runtime.h"
#include "common_includes.h"
#include "Image.cuh"
#include "io_util.h"
#include "SIFT_FeatureFactory.cuh"
#include "MatchFactory.cuh"
#include "PointCloudFactory.cuh"
#include "MeshFactory.cuh"

int main(int argc, char *argv[]){
  try{
    //CUDA INITIALIZATION
    hipInit(0);
    clock_t totalTimer = clock();
    clock_t partialTimer = clock();

    //ARG PARSING
    if(argc < 2 || argc > 4){
      std::cout<<"USAGE ./bin/Tester </path/to/image/directory/>"<<std::endl;
      exit(-1);
    }
    std::string path = argv[1];
    std::vector<std::string> imagePaths = ssrlcv::findFiles(path);

    int numImages = (int) imagePaths.size();

    std::cout << "test code running... " << std::endl;
    // ===================================================================
    // test code goes below here
    // Code within these comment blocks can always be deteled and
    // should only be used when you're testing your new stuff
    // ===================================================================

    // fake cameras
    // doesn't matter what the images are
    std::cout << "Making fake image guys ..." << std::endl;
    std::vector<ssrlcv::Image*> images;
    ssrlcv::Image* image0 = new ssrlcv::Image();
    ssrlcv::Image* image1 = new ssrlcv::Image();
    images.push_back(image0);
    images.push_back(image1);

    // fill the test camera params
    std::cout << "Filling in test Camera Params ..." << std::endl;
    images[0]->camera.cam_pos = {0.0,0.0,0.0};
    images[0]->camera.cam_vec = {1.0,1.0,1.0};
    images[0]->camera.fov = 30.0;
    images[0]->camera.foc = 0.25;
    images[1]->camera.cam_pos = {0.0,-1.0,0.0};
    images[1]->camera.cam_vec = {1.0, 0.0,0.0};
    images[1]->camera.fov = 30.0;
    images[1]->camera.foc = 0.25;


    // ===================================================================
    // test code goes ends above here
    // ===================================================================
    std::cout << "done running test code ... " << std::endl;

    return 0;
  }
  catch (const std::exception &e){
      std::cerr << "Caught exception: " << e.what() << '\n';
      std::exit(1);
  }
  catch (const ssrlcv::UnityException &e){
      std::cerr << "Caught exception: " << e.what() << '\n';
      std::exit(1);
  }
  catch (...){
      std::cerr << "Caught unknown exception\n";
      std::exit(1);
  }

}
