#include "hip/hip_runtime.h"
#include "FeatureFactory.cuh"



ssrlcv::FeatureFactory::ScaleSpace::Octave::Blur::Blur(){
    this->sigma = 0.0f;
    this->pixels = nullptr;
    this->gradients = nullptr;
    this->size = {0,0};
}
ssrlcv::FeatureFactory::ScaleSpace::Octave::Blur::Blur(float sigma, int2 kernelSize, Unity<float>* pixels, uint2 size, float pixelWidth) : 
sigma(sigma),size(size){
    MemoryState origin = pixels->state;
    if(origin == cpu || pixels->fore == cpu) pixels->transferMemoryTo(gpu);
    kernelSize.x = ceil((float)kernelSize.x*this->sigma/pixelWidth);
    kernelSize.y = ceil((float)kernelSize.y*this->sigma/pixelWidth);
    if(kernelSize.x%2 == 0)kernelSize.x++;
    if(kernelSize.y%2 == 0)kernelSize.y++;
    float* gaussian = new float[kernelSize.y*kernelSize.x]();
    for(int y = -kernelSize.y/2, i = 0; y <= kernelSize.y/2; ++y){
        for(int x = -kernelSize.x/2; x <= kernelSize.x/2; ++x){
            gaussian[i++] = expf(-(((x*x) + (y*y))/2.0f/this->sigma/this->sigma))/2.0f/PI/this->sigma/this->sigma;
        }
    }
    pixels->setData(convolve(this->size,pixels,1,kernelSize,gaussian,true)->device,pixels->numElements,gpu);
    pixels->fore = gpu;
    this->pixels = new Unity<float>(nullptr,pixels->numElements,gpu);
    CudaSafeCall(hipMemcpy(this->pixels->device,pixels->device,pixels->numElements*sizeof(float),hipMemcpyDeviceToDevice));
    if(origin == cpu) pixels->setMemoryState(cpu);
    this->gradients = nullptr;
}
void ssrlcv::FeatureFactory::ScaleSpace::Octave::Blur::computeGradients(){
    MemoryState origin = this->pixels->state;
    if(origin == cpu || this->pixels->fore == cpu) this->pixels->transferMemoryTo(gpu);
    this->gradients = generatePixelGradients(this->size, this->pixels);
    if(origin == cpu){
        this->pixels->setMemoryState(cpu);
        this->gradients->setMemoryState(cpu);
    } 
}
ssrlcv::FeatureFactory::ScaleSpace::Octave::Blur::~Blur(){
    if(this->pixels != nullptr) delete this->pixels;
    if(this->gradients != nullptr) delete this->gradients;
}

ssrlcv::FeatureFactory::ScaleSpace::Octave::Octave(){
    this->numBlurs = 0;
    this->blurs = nullptr;
    this->pixelWidth = 0.0f;
    this->extrema = nullptr;
    this->extremaBlurIndices = nullptr;
    this->id = -1;
}
ssrlcv::FeatureFactory::ScaleSpace::Octave::Octave(int id, unsigned int numBlurs, int2 kernelSize, float* sigmas, Unity<float>* pixels, uint2 size, float pixelWidth) : 
numBlurs(numBlurs),pixelWidth(pixelWidth),id(id){
    this->extrema = nullptr;
    this->extremaBlurIndices = nullptr;
    printf("creating octave[%d] with %d blurs of size {%d,%d}\n",this->id,this->numBlurs,size.x,size.y);
    MemoryState origin = pixels->state;
    if(origin == cpu || pixels->fore == cpu) pixels->transferMemoryTo(gpu);

    this->blurs = new Blur*[this->numBlurs]();

    for(int i = 0; i < this->numBlurs; ++i){
        this->blurs[i] = new Blur(sigmas[i],kernelSize,pixels,size,pixelWidth);
    }
    if(origin == cpu) pixels->setMemoryState(cpu);
}
ssrlcv::FeatureFactory::ScaleSpace::Octave::~Octave(){
    if(this->blurs != nullptr){
        for(int i = 0; i < this->numBlurs; ++i){
            delete this->blurs[i];
        }
        delete[] this->blurs;
    } 
    if(this->extrema != nullptr) delete this->extrema;
}
//NOTE THIS PRODUCES BLUR 3 KEY POINTS
void ssrlcv::FeatureFactory::ScaleSpace::Octave::searchForExtrema(){
    Unity<float>* pixelsUpper = nullptr;
    Unity<float>* pixelsMiddle = nullptr;
    Unity<float>* pixelsLower = nullptr;
    dim3 grid2D = {1,1,1};
    dim3 block2D = {3,3,3};
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    MemoryState origin[3];
    int* extremaAddresses = nullptr;
    int totalExtrema = 0;
    SSKeyPoint** extrema2D = new SSKeyPoint*[this->numBlurs - 2]();

    this->extremaBlurIndices = new int[this->numBlurs]();
    this->extremaBlurIndices[0] = 0;
    int extremaAtDepth = 0;

    pixelsLower = this->blurs[0]->pixels;
    getGrid(pixelsLower->numElements,grid2D);
    int* temp = new int[pixelsLower->numElements];
    for(int i = 0; i < pixelsLower->numElements; ++i){
        temp[i] = -1;
    }
    CudaSafeCall(hipMalloc((void**)&extremaAddresses,pixelsLower->numElements*sizeof(int)));
    for(int b = 1; b < this->numBlurs - 1; ++b){
        CudaSafeCall(hipMemcpy(extremaAddresses,temp,pixelsLower->numElements*sizeof(int),hipMemcpyHostToDevice));
        pixelsMiddle = this->blurs[b]->pixels;
        pixelsUpper = this->blurs[b+1]->pixels;
        origin[0] = pixelsLower->state;
        origin[1] = pixelsMiddle->state;
        origin[2] = pixelsUpper->state;
        if(origin[0] == cpu) pixelsLower->transferMemoryTo(gpu);
        if(origin[1] == cpu) pixelsMiddle->transferMemoryTo(gpu);
        if(origin[2] == cpu) pixelsUpper->transferMemoryTo(gpu);
        findExtrema<<<grid2D,block2D>>>(this->blurs[b]->size,pixelsUpper->device,pixelsMiddle->device,pixelsLower->device,extremaAddresses);
        hipDeviceSynchronize();
        CudaCheckError();

        thrust::device_ptr<int> addr(extremaAddresses);

        thrust::device_ptr<int> new_end = thrust::remove(addr, addr + pixelsLower->numElements,-1);
        hipDeviceSynchronize();
        CudaCheckError();
        extremaAtDepth = new_end - addr;

        this->extremaBlurIndices[b] = totalExtrema;
        totalExtrema += extremaAtDepth;

        if(extremaAtDepth != 0){
            //std::cout<<this->id<<" "<<b<<" "<<extremaAtDepth<<std::endl;
            CudaSafeCall(hipMalloc((void**)&extrema2D[b-1],extremaAtDepth*sizeof(ScaleSpace::SSKeyPoint)));
            grid = {1,1,1}; block = {1,1,1};
            getFlatGridBlock(extremaAtDepth,grid,block);
            fillExtrema<<<grid,block>>>(extremaAtDepth,this->blurs[b]->size,this->pixelWidth,{this->id,b},extremaAddresses,pixelsMiddle->device,extrema2D[b-1]);
            CudaCheckError();
        }
        else{
            extrema2D[b-1] = nullptr;
        }
        
        pixelsLower->fore = gpu;
        if(origin[0] == cpu) pixelsLower->setMemoryState(cpu);
        pixelsLower = pixelsMiddle;
        pixelsMiddle = pixelsUpper;
    }
    delete[] temp;
    CudaSafeCall(hipFree(extremaAddresses));
    this->extrema = new Unity<ScaleSpace::SSKeyPoint>(nullptr,totalExtrema,gpu);
    this->extremaBlurIndices[this->numBlurs - 1] = this->extrema->numElements;
    for(int i = 1; i < this->numBlurs - 1; ++i){
        if(extrema2D[i-1] == nullptr) continue;
        if(this->extremaBlurIndices[i+1] - this->extremaBlurIndices[i] != 0){
            CudaSafeCall(hipMemcpy(this->extrema->device + this->extremaBlurIndices[i],extrema2D[i-1],(this->extremaBlurIndices[i+1]-this->extremaBlurIndices[i])*sizeof(ScaleSpace::SSKeyPoint),hipMemcpyDeviceToDevice));
        }
        CudaSafeCall(hipFree(extrema2D[i-1]));
    }  
    delete[] extrema2D;
}
void ssrlcv::FeatureFactory::ScaleSpace::Octave::discardExtrema(){
    if(this->extrema == nullptr) return;
    MemoryState origin = this->extrema->state;
    if(origin == cpu || this->extrema->fore == cpu) this->extrema->transferMemoryTo(gpu);
    SSKeyPoint** temp = new SSKeyPoint*[this->numBlurs];
    int* numExtrema = new int[this->numBlurs];
    int numExtremaAtBlur = 0;
    for(int i = 0; i < this->numBlurs; ++i){
        if(i < this->numBlurs - 1){
            numExtremaAtBlur = this->extremaBlurIndices[i+1] - this->extremaBlurIndices[i];
        }
        else{
            numExtremaAtBlur = this->extrema->numElements - this->extremaBlurIndices[i];
        }
        numExtrema[i] = numExtremaAtBlur;
        if(numExtremaAtBlur == 0){
            temp[i] = nullptr;
            continue;
        }
        CudaSafeCall(hipMalloc((void**)&temp[i],numExtremaAtBlur*sizeof(SSKeyPoint)));
        CudaSafeCall(hipMemcpy(temp[i],this->extrema->device + this->extremaBlurIndices[i],numExtremaAtBlur*sizeof(SSKeyPoint),hipMemcpyDeviceToDevice));
    }
    int totalKept = 0;
    for(int i = 0; i < this->numBlurs; ++i){
        numExtremaAtBlur = 0;
        if(temp[i] != nullptr){
            thrust::device_ptr<ScaleSpace::SSKeyPoint> kp(temp[i]);
            thrust::device_ptr<ScaleSpace::SSKeyPoint> new_end = thrust::remove_if(kp,kp+numExtrema[i],ScaleSpace::discard());
            hipDeviceSynchronize();
            CudaCheckError();
            numExtremaAtBlur = new_end - kp;
        }
        this->extremaBlurIndices[i] = totalKept;
        totalKept += numExtremaAtBlur;
    }
    delete[] numExtrema;

    if(totalKept != 0){
        this->extrema->setData(nullptr,totalKept,gpu);
        for(int i = 0; i < this->numBlurs; ++i){
            if(temp[i] == nullptr) continue;
            if(i == this->numBlurs - 1){
                CudaSafeCall(hipMemcpy(this->extrema->device + this->extremaBlurIndices[i],temp[i],(totalKept-this->extremaBlurIndices[i])*sizeof(SSKeyPoint),hipMemcpyDeviceToDevice));
            }
            else{
                CudaSafeCall(hipMemcpy(this->extrema->device + this->extremaBlurIndices[i],temp[i],(this->extremaBlurIndices[i+1]-this->extremaBlurIndices[i])*sizeof(ScaleSpace::SSKeyPoint),hipMemcpyDeviceToDevice));
            }
            CudaSafeCall(hipFree(temp[i]));
        }
        if(origin == cpu) this->extrema->setMemoryState(cpu);
    }
    else{ 
        delete this->extrema;
        this->extrema = nullptr;
        delete[] this->extremaBlurIndices;
        this->extremaBlurIndices = nullptr;
    }
    delete[] temp;
}

//NOTE THIS IS CURRENTLY REMOVING ALL BLUR 3 KEY POINTS
void ssrlcv::FeatureFactory::ScaleSpace::Octave::refineExtremaLocation(float minScaleSpacePixelWidth){

    MemoryState origin = this->extrema->state;
    if(origin == cpu || this->extrema->fore == cpu) this->extrema->transferMemoryTo(gpu);
    MemoryState* pixelsOrigin = new MemoryState[this->numBlurs];
    for(int i = 0; i < this->numBlurs; ++i){
        pixelsOrigin[i] = this->blurs[i]->pixels->state;
        if(pixelsOrigin[i] == cpu || this->blurs[i]->pixels->fore == cpu){
            this->blurs[i]->pixels->transferMemoryTo(gpu);
        }
    } 

    /*
    1. refine location
    2. discard extrema
    3. resort extrema
    */
    float** allPixels_device = nullptr;
    CudaSafeCall(hipMalloc((void**)&allPixels_device,this->numBlurs*sizeof(float*)));
    for(int i = 0; i < this->numBlurs; ++i){
        CudaSafeCall(hipMemcpy(allPixels_device + i,&this->blurs[i]->pixels->device,sizeof(float*),hipMemcpyHostToDevice));
    }

    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    getFlatGridBlock(this->extrema->numElements,grid,block);
    refineLocation<<<grid,block>>>(this->extrema->numElements, this->blurs[0]->size, this->blurs[0]->sigma, 
        this->pixelWidth/minScaleSpacePixelWidth, this->pixelWidth, this->numBlurs, allPixels_device, this->extrema->device);
    hipDeviceSynchronize();
    CudaCheckError();
    
    CudaSafeCall(hipFree(allPixels_device));
    this->extrema->fore = gpu;

    this->discardExtrema();

    thrust::device_ptr<SSKeyPoint> kp(this->extrema->device);
    thrust::stable_sort(kp, kp + this->extrema->numElements);
    this->extrema->transferMemoryTo(cpu);
    this->extremaBlurIndices[0] = 0;
    this->extremaBlurIndices[1] = 0;
    for(int i = 1,blur = 2; i < this->extrema->numElements && blur < this->numBlurs - 1; ++i){
        if(this->extrema->host[i-1] < this->extrema->host[i]){
            this->extremaBlurIndices[blur++] = i; 
            //std::cout<<this->id<<" "<<blur-2<<" "<<i-this->extremaBlurIndices[blur - 2]<<std::endl;
        } 
    }
    this->extremaBlurIndices[this->numBlurs - 1] = this->extrema->numElements;
    this->extrema->fore = cpu;
    if(origin == gpu) this->extrema->setMemoryState(gpu);
    for(int i = 0; i < this->numBlurs; ++i){
        if(pixelsOrigin[i] == cpu) this->blurs[i]->pixels->setMemoryState(cpu);
    }
    delete[] pixelsOrigin;
}
void ssrlcv::FeatureFactory::ScaleSpace::Octave::removeNoise(float noiseThreshold){
    MemoryState origin = this->extrema->state;
    if(origin == cpu || this->extrema->fore == cpu) this->extrema->transferMemoryTo(gpu);
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    getFlatGridBlock(this->extrema->numElements,grid,block);
    flagNoise<<<grid,block>>>(this->extrema->numElements,this->extrema->device,noiseThreshold);
    hipDeviceSynchronize();
    CudaCheckError();
    this->extrema->fore = gpu;
    this->discardExtrema();
    if(origin == cpu) this->extrema->setMemoryState(cpu);
}
void ssrlcv::FeatureFactory::ScaleSpace::Octave::removeEdges(float edgeThreshold){
    MemoryState origin = this->extrema->state;
    if(origin == cpu || this->extrema->fore == cpu) this->extrema->transferMemoryTo(gpu);
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    int numExtremaAtBlur = 0;
    MemoryState pixelOrigin;
    for(int i = 0; i < this->numBlurs; ++i){
        grid = {1,1,1};
        block = {1,1,1};
        if(i < this->numBlurs - 1){
            numExtremaAtBlur = this->extremaBlurIndices[i+1] - this->extremaBlurIndices[i];
        }   
        else{
            numExtremaAtBlur = this->extrema->numElements - this->extremaBlurIndices[i];
        }
        if(numExtremaAtBlur == 0) continue;
        pixelOrigin = this->blurs[i+1]->pixels->state;
        if(pixelOrigin == cpu || this->blurs[i+1]->pixels->fore == cpu) this->blurs[i]->pixels->transferMemoryTo(gpu);
        getFlatGridBlock(numExtremaAtBlur,grid,block);
        flagEdges<<<grid,block>>>(numExtremaAtBlur, this->extremaBlurIndices[i], this->blurs[0]->size,this->extrema->device,this->blurs[i]->pixels->device,edgeThreshold);
        hipDeviceSynchronize();
        CudaCheckError();
        if(pixelOrigin == cpu){
            this->blurs[i]->pixels->setMemoryState(cpu);
        }
    }
    this->extrema->fore = gpu;
    this->discardExtrema();
    if(origin == cpu) this->extrema->setMemoryState(cpu);
}
void ssrlcv::FeatureFactory::ScaleSpace::Octave::removeBorder(float2 border){
    if(this->extrema == nullptr) return;
    MemoryState origin = this->extrema->state;
    if(origin == cpu || this->extrema->fore == cpu) this->extrema->transferMemoryTo(gpu);
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    int numExtremaAtBlur = 0;
    getFlatGridBlock(numExtremaAtBlur,grid,block);
    flagBorder<<<grid,block>>>(numExtremaAtBlur, this->blurs[0]->size,this->extrema->device,border);
    hipDeviceSynchronize();
    CudaCheckError();
       
    this->extrema->fore = gpu;
    this->discardExtrema();
    if(origin == cpu) this->extrema->setMemoryState(cpu);
}


ssrlcv::FeatureFactory::ScaleSpace::ScaleSpace(){
    this->depth = {0,0};
    this->octaves = nullptr;

}
ssrlcv::FeatureFactory::ScaleSpace::ScaleSpace(Image* image, int startingOctave, uint2 depth, float initialSigma, float2 sigmaMultiplier, int2 kernelSize) : 
depth(depth){ 

    if(image->size.x/powf(2, startingOctave+depth.x) == 0 || image->size.x/powf(2, startingOctave+depth.x) == 0){
        std::cerr<<"This image is too small to make a ScaleSpace of the specified depth"<<std::endl;
        exit(-1);
    }
    printf("creating scalespace with depth {%d,%d}\n",this->depth.x,this->depth.y);
    Unity<float>* pixels = nullptr;
    
    if(image->colorDepth != 1){
        Unity<unsigned char>* charPixels = new Unity<unsigned char>(nullptr,image->pixels->numElements,gpu);
        MemoryState origin = image->pixels->state;
        if(origin == cpu || image->pixels->fore == cpu) image->pixels->transferMemoryTo(gpu);
        CudaSafeCall(hipMemcpy(charPixels->device, image->pixels->device, pixels->numElements*sizeof(unsigned char),hipMemcpyDeviceToDevice));
        convertToBW(charPixels,image->colorDepth);
        pixels = convertImageToFlt(charPixels);
        if(origin == cpu) image->pixels->setMemoryState(cpu);
        delete charPixels;
    }
    else{
        pixels = convertImageToFlt(image->pixels);
    }

    normalizeImage(pixels);

    uint2 imageSize = image->size;
    uint2 scalar = {2,2};
   
    float pixelWidth = 1.0f;

    for(int i = startingOctave; i < 0; ++i){
        pixels->setData(upsample(imageSize,1,pixels)->device,pixels->numElements*4,gpu);   
        imageSize = imageSize*scalar;
        pixelWidth /= 2.0f;
    }
    for(int i = 0; i < startingOctave; ++i){
        pixels->setData(bin(imageSize,1,pixels)->device,pixels->numElements/4,gpu);   
        imageSize = imageSize/scalar;
        pixelWidth *= 2.0f;
    }   
    float* sigmas = new float[this->depth.y]();
    sigmas[0] = initialSigma;
    for(int i = 1; i < this->depth.y; ++i){
        sigmas[i] = sigmas[i-1]*sigmaMultiplier.y;
    }
    
    this->octaves = new Octave*[this->depth.x]();
    for(int i = 0; i < this->depth.x; ++i){
        this->octaves[i] = new Octave(i,this->depth.y,kernelSize,sigmas,pixels,imageSize,pixelWidth);
        if(i + 1 < this->depth.x){
            pixels->setData(bin(imageSize,1,pixels)->device,pixels->numElements/4,gpu);
            imageSize = imageSize/scalar;
            pixelWidth *= 2.0f;
            for(int b = 0; b < this->depth.y; ++b){
                sigmas[b]*=sigmaMultiplier.x;    
            }
        }
        hipDeviceSynchronize();
        CudaCheckError();
    }
    delete pixels;
    delete[] sigmas;

}
void ssrlcv::FeatureFactory::ScaleSpace::convertToDOG(){
    Unity<float>* pixelsUpper = nullptr;
    Unity<float>* pixelsLower = nullptr;
    MemoryState origin[2];
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    uint2 dogDepth = {this->depth.x,this->depth.y - 1};
    Octave** dogOctaves = new Octave*[dogDepth.x]();
    for(int o = 0; o < dogDepth.x; o++){
        dogOctaves[o] = new Octave();
        dogOctaves[o]->blurs = new Octave::Blur*[dogDepth.y]();
        dogOctaves[o]->numBlurs = dogDepth.y;
        dogOctaves[o]->pixelWidth = this->octaves[o]->pixelWidth;
        pixelsLower = this->octaves[o]->blurs[0]->pixels;
        getFlatGridBlock(pixelsLower->numElements,grid,block);
        for(int b = 0; b < dogDepth.y; ++b){
            dogOctaves[o]->blurs[b] = new Octave::Blur();
            dogOctaves[o]->id = o;
            dogOctaves[o]->blurs[b]->size = this->octaves[o]->blurs[0]->size;
            dogOctaves[o]->blurs[b]->sigma = (this->octaves[o]->pixelWidth/this->octaves[0]->pixelWidth)*this->octaves[0]->blurs[0]->sigma*pow(2,b/3);//TODO check these sigmas
            dogOctaves[o]->blurs[b]->pixels = new Unity<float>(nullptr,pixelsLower->numElements,gpu);
            pixelsUpper = this->octaves[o]->blurs[b+1]->pixels;
            origin[0] = pixelsLower->state;
            origin[1] = pixelsUpper->state;
            if(origin[0] == cpu) pixelsLower->transferMemoryTo(gpu);
            if(origin[1] == cpu) pixelsUpper->transferMemoryTo(gpu);
            subtractImages<<<grid,block>>>(pixelsLower->numElements,pixelsUpper->device,pixelsLower->device,dogOctaves[o]->blurs[b]->pixels->device);
            hipDeviceSynchronize();
            CudaCheckError();
            normalizeImage(dogOctaves[o]->blurs[b]->pixels);
            if(origin[0] == cpu) pixelsLower->setMemoryState(cpu);
            pixelsLower = pixelsUpper;
        }
    }
    for(int i = 0; i < this->depth.x; ++i){
        delete this->octaves[i];
    }
    delete[] this->octaves;
    this->depth = dogDepth;
    this->octaves = dogOctaves;
}
ssrlcv::FeatureFactory::ScaleSpace::~ScaleSpace(){
    if(this->octaves != nullptr){
        for(int i = 0; i < this->depth.x; ++i){
            delete this->octaves[i];
        }
        delete[] this->octaves;
    }
 
}
void ssrlcv::FeatureFactory::ScaleSpace::dumpData(std::string filePath){
    for(int o = 0; o < this->depth.x; ++o){
        for(int b = 0; b < this->depth.y; ++b){
            Unity<unsigned char>* writable = convertImageToChar(this->octaves[o]->blurs[b]->pixels);
            writable->transferMemoryTo(cpu);
            std::string currentFile = filePath + std::to_string(o) + "_" + std::to_string(b) + ".png";
            writePNG(currentFile.c_str(), writable->host, 1, this->octaves[o]->blurs[b]->size.x, this->octaves[o]->blurs[b]->size.y);
        }
    }
}
void ssrlcv::FeatureFactory::ScaleSpace::findKeyPoints(float noiseThreshold, float edgeThreshold, bool subpixel){
    if(this->depth.y < 4){
        std::cerr<<"findKeyPoints should be done on a dog scale space - this is either not a dog or the number of blurs is insufficient"<<std::endl;
        exit(-1);
    }
    int temp = 0;
    for(int i = 0; i < this->depth.x; ++i){
        this->octaves[i]->searchForExtrema();
        temp = this->octaves[i]->extrema->numElements;
        std::cout<<"keypoints in octave["<<i<<"] = "<<temp;
        if(temp > 0){
            this->octaves[i]->removeNoise(noiseThreshold*0.8);
            std::cout<<"-"<<temp - this->octaves[i]->extrema->numElements;
            if(subpixel){
                this->octaves[i]->refineExtremaLocation(this->octaves[0]->pixelWidth);
                std::cout<<"-"<<temp - this->octaves[i]->extrema->numElements;
                this->octaves[i]->removeNoise(noiseThreshold);
                std::cout<<"-"<<temp - this->octaves[i]->extrema->numElements;
            }
            this->octaves[i]->removeEdges(edgeThreshold);
            std::cout<<"-"<<temp - this->octaves[i]->extrema->numElements;
            std::cout<<"="<<this->octaves[i]->extrema->numElements<<std::endl;
        }  
        if(this->octaves[i]->extrema->numElements == 0){
            std::cout<<std::endl;
            delete this->octaves[i]->extrema;
            delete[] this->octaves[i]->extremaBlurIndices;
            this->octaves[i]->extremaBlurIndices = nullptr;
            this->octaves[i]->extrema = nullptr;
        }
    }
}
ssrlcv::Unity<ssrlcv::FeatureFactory::ScaleSpace::SSKeyPoint>* ssrlcv::FeatureFactory::ScaleSpace::getAllKeyPoints(MemoryState destination){
    unsigned int totalKeyPoints = 0;
    if(destination != cpu || destination != gpu){
        std::cerr<<"in getAllKeyPoints, destination must be cpu or gpu"<<std::endl;
        exit(-1);
    }
    MemoryState* origin = new MemoryState[this->depth.x];
    for(int i = 0; i < this->depth.x; ++i){
        origin[i] = this->octaves[i]->extrema->state;
        if(origin[i] != destination) this->octaves[i]->extrema->transferMemoryTo(destination);
        totalKeyPoints += this->octaves[i]->extrema->numElements;
    }
    if(totalKeyPoints == 0){
        std::cerr<<"scale space has no keyPoints generated within its octaves"<<std::endl;
        exit(0);
    }
    Unity<SSKeyPoint>* aggregatedKeyPoints = new Unity<SSKeyPoint>(nullptr,totalKeyPoints,destination);
    int currentIndex = 0;
    for(int i = 0; i < this->depth.x; ++i){
        if(destination == cpu){
            std::memcpy(aggregatedKeyPoints->host + currentIndex, this->octaves[i]->extrema->host, this->octaves[i]->extrema->numElements*sizeof(SSKeyPoint));
        }
        else{
            CudaSafeCall(hipMemcpy(aggregatedKeyPoints->device + currentIndex, this->octaves[i]->extrema->device, this->octaves[i]->extrema->numElements*sizeof(SSKeyPoint),hipMemcpyDeviceToDevice));
        }
        currentIndex += this->octaves[i]->extrema->numElements;
        if(origin[i] != destination) this->octaves[i]->extrema->setMemoryState(origin[i]);
    }
    return aggregatedKeyPoints;
}

void ssrlcv::FeatureFactory::ScaleSpace::computeKeyPointOrientations(float orientationThreshold, unsigned int maxOrientations, float contributerWindowWidth, bool keepGradients){
    ScaleSpace::Octave* currentOctave = nullptr;
    ScaleSpace::Octave::Blur* currentBlur = nullptr;
    int* thetaAddresses_device = nullptr;
    float* thetas_device = nullptr;
    dim3 grid = {1,1,1};
    dim3 block = {1,1,1};
    unsigned int numKeyPointsAtBlur = 0;
    MemoryState origin;
    unsigned int numOrientedKeyPoints = 0;
    unsigned int totalKeyPoints = 0;
    ScaleSpace::SSKeyPoint** orientedKeyPoints2D = nullptr;
    unsigned int keyPointIndex = 0;
    bool gradientsExisted = false;
    for(int o = 0; o < this->depth.x; ++o){
        currentOctave = this->octaves[o];
        if(currentOctave->extrema == nullptr) continue;
        totalKeyPoints = 0;
        orientedKeyPoints2D = new ScaleSpace::SSKeyPoint*[this->depth.y];
        origin = currentOctave->extrema->state;
        if(origin == cpu || currentOctave->extrema->fore == cpu){
            currentOctave->extrema->setMemoryState(gpu);
        }
        for(int b = 0; b < this->depth.y; ++b){
            currentBlur = currentOctave->blurs[b];
            if(b + 1 != this->depth.y){
                numKeyPointsAtBlur = currentOctave->extremaBlurIndices[b + 1] - currentOctave->extremaBlurIndices[b];
            }
            else{
                numKeyPointsAtBlur = currentOctave->extrema->numElements - currentOctave->extremaBlurIndices[b];
            }
            if(numKeyPointsAtBlur == 0){
                orientedKeyPoints2D[b] = nullptr;
                currentOctave->extremaBlurIndices[b] = totalKeyPoints;
                continue;
            } 
            std::cout<<b<<" "<<numKeyPointsAtBlur<<std::endl;
            keyPointIndex = currentOctave->extremaBlurIndices[b];
            grid = {1,1,1};
            block = {1,1,1}; 
            getFlatGridBlock(numKeyPointsAtBlur, grid, block);
            
            //determine how to best record num orientations for a keypoint
            
            CudaSafeCall(hipMalloc((void**)&thetas_device, numKeyPointsAtBlur*maxOrientations*sizeof(float)));
            CudaSafeCall(hipMalloc((void**)&thetaAddresses_device, numKeyPointsAtBlur*maxOrientations*sizeof(int)));

            gradientsExisted = currentBlur->gradients != nullptr;
            if(!gradientsExisted) currentBlur->computeGradients();
            if(currentBlur->gradients->state != gpu) currentBlur->gradients->setMemoryState(gpu);
        
            computeThetas<<<grid,block>>>(numKeyPointsAtBlur,keyPointIndex,currentBlur->size, currentOctave->pixelWidth,
                contributerWindowWidth,currentOctave->extrema->device, currentBlur->gradients->device, thetaAddresses_device, maxOrientations, orientationThreshold, thetas_device);
            hipDeviceSynchronize();
            CudaCheckError();

            if(!keepGradients && !gradientsExisted){
                delete currentBlur->gradients;
                currentBlur->gradients = nullptr;
            } 

            thrust::device_ptr<float> t(thetas_device);
            thrust::device_ptr<float> new_end = thrust::remove(t, t + (numKeyPointsAtBlur*maxOrientations), -1.0f);
            thrust::device_ptr<int> tN(thetaAddresses_device);
            thrust::device_ptr<int> end = thrust::remove(tN, tN + (numKeyPointsAtBlur*maxOrientations), -1);
            numOrientedKeyPoints = end - tN;
            
            currentOctave->extremaBlurIndices[b] = totalKeyPoints;
            totalKeyPoints += numOrientedKeyPoints;

            if(numOrientedKeyPoints != 0){
                grid = {1,1,1};
                block = {1,1,1};
                getFlatGridBlock(numOrientedKeyPoints,grid,block);
                CudaSafeCall(hipMalloc((void**)&orientedKeyPoints2D[b],numOrientedKeyPoints*sizeof(ScaleSpace::SSKeyPoint)));
                expandKeyPoints<<<grid,block>>>(numOrientedKeyPoints, currentOctave->extrema->device, orientedKeyPoints2D[b], thetaAddresses_device, thetas_device);
                hipDeviceSynchronize();
                CudaCheckError();
            }
            CudaSafeCall(hipFree(thetas_device));
            CudaSafeCall(hipFree(thetaAddresses_device));
        }
        printf("after computing theta for each keyPoint octave[%d] has %d keyPoints\n",o,totalKeyPoints);
        if(totalKeyPoints != 0){
            currentOctave->extrema->setData(nullptr,totalKeyPoints,gpu);
            for(int i = 0; i < currentOctave->numBlurs; ++i){
                if(orientedKeyPoints2D[i] == nullptr) continue;
                if(i == currentOctave->numBlurs - 1 && totalKeyPoints-currentOctave->extremaBlurIndices[i] != 0){
                    CudaSafeCall(hipMemcpy(currentOctave->extrema->device + currentOctave->extremaBlurIndices[i],orientedKeyPoints2D[i],(totalKeyPoints-currentOctave->extremaBlurIndices[i])*sizeof(ScaleSpace::SSKeyPoint),hipMemcpyDeviceToDevice));
                }
                else if(i != currentOctave->numBlurs - 1 && currentOctave->extremaBlurIndices[i+1]-currentOctave->extremaBlurIndices[i] != 0){
                    CudaSafeCall(hipMemcpy(currentOctave->extrema->device + currentOctave->extremaBlurIndices[i],orientedKeyPoints2D[i],(currentOctave->extremaBlurIndices[i+1]-currentOctave->extremaBlurIndices[i])*sizeof(ScaleSpace::SSKeyPoint),hipMemcpyDeviceToDevice));
                }
                CudaSafeCall(hipFree(orientedKeyPoints2D[i]));
            }  
            if(origin == cpu) currentOctave->extrema->setMemoryState(cpu);
        }
        else{
            delete currentOctave->extrema;
            currentOctave->extrema = nullptr;
        } 
        delete[] orientedKeyPoints2D;
    }
}

ssrlcv::FeatureFactory::FeatureFactory(float orientationContribWidth, float descriptorContribWidth):
orientationContribWidth(orientationContribWidth), descriptorContribWidth(descriptorContribWidth)
{}

ssrlcv::FeatureFactory::~FeatureFactory(){

}

__constant__ float ssrlcv::pi = 3.1415927;

/*
const long double PI = 3.141592653589793238L;
const double PI = 3.141592653589793;
const float PI = 3.1415927;
*/

__device__ __forceinline__ float ssrlcv::getMagnitude(const int2 &vector){
  return sqrtf((float)dotProduct(vector, vector));
}
__device__ __forceinline__ float ssrlcv::getMagnitude(const float2 &vector){
  return sqrtf(dotProduct(vector, vector));
}
__device__ __forceinline__ float ssrlcv::getMagnitudeSq(const int2 &vector){
  return (float)dotProduct(vector, vector);
}
__device__ __forceinline__ float ssrlcv::getMagnitudeSq(const float2 &vector){
  return dotProduct(vector, vector);
}
__device__ __forceinline__ float ssrlcv::getTheta(const int2 &vector){
  return fmodf(atan2f((float)vector.y, (float)vector.x) + pi,2.0f*pi);
}
__device__ __forceinline__ float ssrlcv::getTheta(const float2 &vector){
  return fmodf(atan2f(vector.y, vector.x) + pi,2.0f*pi);
}
__device__ __forceinline__ float ssrlcv::getTheta(const float2 &vector, const float &offset){
  return fmodf((atan2f(vector.y, vector.x) + pi) - offset,2.0f*pi);
}
__device__ void ssrlcv::trickleSwap(const float2 &compareWValue, float2* arr, const int &index, const int &length){
  for(int i = index; i < length; ++i){
    if(compareWValue.x > arr[i].x){
      float2 temp = arr[i];
      arr[i] = compareWValue;
      if((temp.x == 0.0f && temp.y == 0.0f)|| index + 1 == length) return;
      return trickleSwap(temp, arr, index + 1, length);
    }
  }
}
__device__ __forceinline__ long4 ssrlcv::getOrientationContributers(const long2 &loc, const uint2 &imageSize){
  long4 orientationContributers;
  long pixelIndex = loc.y*imageSize.x + loc.x;
  orientationContributers.x = (loc.x == imageSize.x - 1) ? -1 : pixelIndex + 1;
  orientationContributers.y = (loc.x == 0) ? -1 : pixelIndex - 1;
  orientationContributers.z = (loc.y == imageSize.y - 1) ? -1 : (loc.y + 1)*imageSize.x + loc.x;
  orientationContributers.w = (loc.y == 0) ? -1 : (loc.y - 1)*imageSize.x + loc.x;
  return orientationContributers;
}
__device__ __forceinline__ int ssrlcv::floatToOrderedInt(float floatVal){
 int intVal = __float_as_int( floatVal );
 return (intVal >= 0 ) ? intVal : intVal ^ 0x7FFFFFFF;
}
__device__ __forceinline__ float ssrlcv::orderedIntToFloat(int intVal){
 return __int_as_float( (intVal >= 0) ? intVal : intVal ^ 0x7FFFFFFF);
}
__device__ __forceinline__ float ssrlcv::modulus(const float &x, const float &y){
    float z = x;
    int n;
    if(z < 0){
        n = (int)((-z)/y)+1;
        z += n*y;
    }
    n = (int)(z/y);
    z -= n*y;
    return z;
}
__device__ __forceinline__ float2 ssrlcv::rotateAboutPoint(const int2 &loc, const float &theta, const float2 &origin){
  float2 rotatedPoint = {(float) loc.x, (float) loc.y};
  rotatedPoint = rotatedPoint - origin;
  float2 temp = rotatedPoint;

  rotatedPoint.x = (temp.x*cosf(theta)) - (temp.y*sinf(theta)) + origin.x;
  rotatedPoint.y = (temp.x*sinf(theta)) + (temp.y*cosf(theta)) + origin.y;

  return rotatedPoint;
}

__device__ __forceinline__ float ssrlcv::atomicMinFloat (float * addr, float value){
  float old;
  old = (value >= 0) ? __int_as_float(atomicMin((int *)addr, __float_as_int(value))) :
    __uint_as_float(atomicMax((unsigned int *)addr, __float_as_uint(value)));
  return old;
}
__device__ __forceinline__ float ssrlcv::atomicMaxFloat (float * addr, float value){
  float old;
  old = (value >= 0) ? __int_as_float(atomicMax((int *)addr, __float_as_int(value))) :
    __uint_as_float(atomicMin((unsigned int *)addr, __float_as_uint(value)));
  return old;
}
__device__ __forceinline__ float ssrlcv::edgeness(const float (&hessian)[2][2]){
    float e = trace(hessian);
    return e*e/determinant(hessian);    
}

__global__ void ssrlcv::subtractImages(unsigned int numPixels, float* pixelsUpper, float* pixelsLower, float* pixelsOut){
    unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
    if(globalID < numPixels) pixelsOut[globalID] = pixelsUpper[globalID] - pixelsLower[globalID];
}

__global__ void ssrlcv::findExtrema(uint2 imageSize, float* pixelsUpper, float* pixelsMiddle, float* pixelsLower, int* extrema){
    int blockId = blockIdx.y* gridDim.x+ blockIdx.x;
    int x = blockId%imageSize.x;
    int y = blockId/imageSize.x;
    if(x > 0 && y > 0 && x < imageSize.x - 1 && y < imageSize.y - 1){
        float value = 0.0f;
        x += (((int)threadIdx.x) - 1);
        y += (((int)threadIdx.y) - 1);
        __shared__ float maximumValue;
        __shared__ float minimumValue;
        minimumValue = FLT_MAX;
        maximumValue = -FLT_MAX;
        __syncthreads();
        if(threadIdx.z == 0){
            value = pixelsLower[y*imageSize.x + x];
        }
        else if(threadIdx.z == 1){
            value = pixelsMiddle[y*imageSize.x + x];
        }
        else{
            value = pixelsUpper[y*imageSize.x + x];
        }
        atomicMaxFloat(&maximumValue,value);
        atomicMinFloat(&minimumValue,value);
        __syncthreads();
        if(threadIdx.x == 1 && threadIdx.y == 1 && threadIdx.z == 1){
            if(maximumValue == value || minimumValue == value){
                extrema[blockId] = blockId;
            }
            else{
                extrema[blockId] = -1;
            }
        }
        else return;
    }
}
__global__ void ssrlcv::fillExtrema(int numKeyPoints, uint2 imageSize, float pixelWidth, int2 ssLoc, int* extremaAddresses, float* pixels, FeatureFactory::ScaleSpace::SSKeyPoint* scaleSpaceKP){
    int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
    if(globalID < numKeyPoints){
        int index = extremaAddresses[globalID];
        float2 loc = {(float)(index%imageSize.x),(float)(index/imageSize.x)};
        scaleSpaceKP[globalID] = {ssLoc.x,ssLoc.y,loc,pixels[index],0.0f,-1.0f,false};
    }
}

__global__ void ssrlcv::refineLocation(unsigned int numKeyPoints, uint2 imageSize, float sigmaMin, float pixelWidthRatio, float pixelWidth, unsigned int numBlurs, float** pixels, FeatureFactory::ScaleSpace::SSKeyPoint* scaleSpaceKP){
    int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
    if(globalID < numKeyPoints){
        FeatureFactory::ScaleSpace::SSKeyPoint kp = scaleSpaceKP[globalID];
        int2 loc = {(int)roundf(kp.loc.x),(int)roundf(kp.loc.y)};
        float hessian[3][3] = {0.0f};
        float hessian_inv[3][3] = {0.0f};
        float gradient[3] = {0.0f};
        float temp[3] = {0.0f};
        float offset[3] = {0.0f};
        float* pixelsLower = pixels[kp.blur - 1];
        float* pixelsMiddle = pixels[kp.blur];
        float* pixelsUpper = pixels[kp.blur + 1];
 
        for(int attempt = 0; attempt < 5; ++attempt){
            gradient[0] =  pixelsMiddle[loc.y*imageSize.x + loc.x + 1] - pixelsMiddle[loc.y*imageSize.x + loc.x - 1];
            gradient[1] =  pixelsMiddle[(loc.y+1)*imageSize.x + loc.x] - pixelsMiddle[(loc.y-1)*imageSize.x + loc.x];
            gradient[2] =  pixelsUpper[loc.y*imageSize.x + loc.x] - pixelsLower[loc.y*imageSize.x + loc.x];
            hessian[0][0] = gradient[0] - 2*pixelsMiddle[loc.y*imageSize.x + loc.x];
            hessian[0][1] = (pixelsMiddle[(loc.y+1)*imageSize.x + loc.x + 1] - 
                pixelsMiddle[(loc.y-1)*imageSize.x + loc.x + 1] - 
                pixelsMiddle[(loc.y+1)*imageSize.x + loc.x - 1] + 
                pixelsMiddle[(loc.y-1)*imageSize.x + loc.x - 1])/4.0f;
            hessian[0][2] = (pixelsUpper[loc.y*imageSize.x + loc.x + 1] - 
                pixelsLower[loc.y*imageSize.x + loc.x + 1] - 
                pixelsUpper[loc.y*imageSize.x + loc.x - 1] + 
                pixelsLower[loc.y*imageSize.x + loc.x - 1])/4.0f;
            hessian[1][0] = hessian[0][1];
            hessian[1][1] = gradient[1] - 2*pixelsMiddle[loc.y*imageSize.x + loc.x];
            hessian[1][2] = (pixelsUpper[(loc.y+1)*imageSize.x + loc.x] - 
                pixelsLower[(loc.y+1)*imageSize.x + loc.x] - 
                pixelsUpper[(loc.y-1)*imageSize.x + loc.x] + 
                pixelsLower[(loc.y-1)*imageSize.x + loc.x])/4.0f;
            hessian[2][0] = hessian[0][2];
            hessian[2][1] = hessian[1][2];
            hessian[2][2] = gradient[2] - 2*pixelsMiddle[loc.y*imageSize.x + loc.x];
            for(int r = 0; r < 3; ++r){
                for(int c = 0; c < 3; ++c){
                    hessian[r][c] *= -1.0f;
                }
            }
            inverse(hessian,hessian_inv);
            multiply(hessian_inv,gradient,offset);
            multiply(gradient, hessian, temp);
            if(offset[0] <= 0.5f && offset[1] <= 0.5f && offset[2] <= 0.5f){ 
                kp.loc = {(float)loc.x + offset[0],(float)loc.y + offset[1]};
                loc = {(int)roundf(kp.loc.x),(int)roundf(kp.loc.y)};
                kp.discard = (loc.x <= 0 || loc.y <= 0 || loc.x >= imageSize.x - 1 || loc.y >= imageSize.y - 1);
                if(kp.discard) break;//to prevent more operations
                kp.intensity = pixelsMiddle[loc.y*imageSize.x + loc.x] - (0.5f*dotProduct(temp,gradient));
                kp.sigma = pixelWidthRatio*sigmaMin*powf(2,(offset[2]+kp.blur)/3);
                break;
            }
            else if(attempt == 4){
                kp.discard = true;
                break;
            }
            else{
                loc.x += (int)roundf(offset[0]);
                loc.y += (int)roundf(offset[1]);
                kp.loc = {(float)loc.x,(float)loc.y};
                kp.blur += (int)roundf(offset[2]);
                if(kp.blur >= numBlurs - 1||kp.blur <= 0||loc.x <= 0||
                    loc.y <= 0||loc.x >= imageSize.x - 1||loc.y >= imageSize.y - 1){//cannot traverse blurs anymore
                    kp.discard = true;
                    break;
                }
                pixelsLower = pixels[kp.blur - 1];
                pixelsMiddle = pixels[kp.blur];
                pixelsUpper = pixels[kp.blur + 1];
            }
        }
        scaleSpaceKP[globalID] = kp;
    }
}
__global__ void ssrlcv::flagNoise(unsigned int numKeyPoints, FeatureFactory::ScaleSpace::SSKeyPoint* scaleSpaceKP, float threshold){
    unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
    if(globalID < numKeyPoints){
        scaleSpaceKP[globalID].discard = abs(scaleSpaceKP[globalID].intensity) < threshold;
    }
}
__global__ void ssrlcv::flagEdges(unsigned int numKeyPoints, unsigned int startingIndex, uint2 imageSize, FeatureFactory::ScaleSpace::SSKeyPoint* scaleSpaceKP, float* pixels, float threshold){
    unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
    if(globalID < numKeyPoints){
        globalID += startingIndex;
        int2 loc = {(int)roundf(scaleSpaceKP[globalID].loc.x),(int)roundf(scaleSpaceKP[globalID].loc.y)};
        float hessian[2][2] = {0.0f};
        hessian[0][0] = -2.0f*pixels[loc.y*imageSize.x + loc.x];
        hessian[1][1] = hessian[0][0] + pixels[(loc.y + 1)*imageSize.x + loc.x] + pixels[(loc.y - 1)*imageSize.x + loc.x];
        hessian[0][0] += pixels[loc.y*imageSize.x + loc.x + 1] + pixels[loc.y*imageSize.x + loc.x - 1];
        hessian[0][1] = (
            pixels[(loc.y + 1)*imageSize.x + loc.x + 1] - pixels[(loc.y - 1)*imageSize.x + loc.x + 1] -
            pixels[(loc.y + 1)*imageSize.x + loc.x - 1] + pixels[(loc.y - 1)*imageSize.x + loc.x - 1]
        );
        hessian[1][0] = hessian[0][1];
        scaleSpaceKP[globalID].discard = edgeness(hessian) > threshold;
    }
}

__global__ void ssrlcv::flagBorder(unsigned int numKeyPoints, uint2 imageSize, FeatureFactory::ScaleSpace::SSKeyPoint* scaleSpaceKP, float2 border){
    unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
    if(globalID < numKeyPoints){
        FeatureFactory::ScaleSpace::SSKeyPoint kp = scaleSpaceKP[globalID];
        if(kp.loc.x < border.x || kp.loc.y < border.y || kp.loc.x >= (float)imageSize.x - border.x || kp.loc.y >= (float)imageSize.y - border.y){
            scaleSpaceKP[globalID].discard = true;
        }
    }

}


__global__ void ssrlcv::computeThetas(unsigned long numKeyPoints, unsigned int keyPointIndex, uint2 imageSize, float pixelWidth, 
float lambda, FeatureFactory::ScaleSpace::SSKeyPoint* keyPoints, float2* gradients, 
int* thetaNumbers, unsigned int maxOrientations, float orientationThreshold, float* thetas){
   unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
    if(globalID < numKeyPoints){
        FeatureFactory::ScaleSpace::SSKeyPoint kp = keyPoints[globalID+keyPointIndex];
        float2 keyPoint = kp.loc;
        float windowWidth = kp.sigma*3.0f*lambda;
        int regNumOrient = maxOrientations;

        float2 min = {(keyPoint.x - windowWidth)/pixelWidth,(keyPoint.y - windowWidth)/pixelWidth};
        if(min.x < 0.0f) min.x = 0.0f;
        if(min.y < 0.0f) min.y = 0.0f;
        float2 max = {(keyPoint.x + windowWidth)/pixelWidth,(keyPoint.x + windowWidth)/pixelWidth};
        if(max.x >= imageSize.x - 1) max.x = imageSize.x - 1;
        if(max.y >= imageSize.y - 1) max.y = imageSize.y - 1;

        float orientationHist[36] = {0.0f};
        float maxHist = 0.0f;
        float2 gradient = {0.0f,0.0f};
        float2 temp2 = {0.0f,0.0f};
        unsigned int imageWidth = imageSize.x;
        for(float y = min.y; y <= max.y; y+=1.0f){
            for(float x = min.x; x <= max.x; x+=1.0f){
                gradient = {
                    (float)gradients[llroundf(y)*imageWidth + llroundf(x)].x,
                    (float)gradients[llroundf(y)*imageWidth + llroundf(x)].y
                };
                temp2 = {x*pixelWidth - keyPoint.x,y*pixelWidth - keyPoint.y};
                orientationHist[llroundf(36.0f*getTheta(gradient)/(2.0f*pi))] += expf(-getMagnitude(temp2)/(2.0f*lambda*lambda*kp.sigma*kp.sigma))*getMagnitude(gradient);
            }
        }
        float3 convHelper = {orientationHist[35],orientationHist[0],orientationHist[1]};
        for(int i = 0; i < 6; ++i){
            temp2.x = orientationHist[0];//need to hold on to this for id = 35 conv
            for(int id = 1; id < 36; ++id){
                orientationHist[id] = (convHelper.x+convHelper.y+convHelper.z)/3.0f;
                convHelper.x = convHelper.y;
                convHelper.y = convHelper.z;
                convHelper.z = (id < 35) ? orientationHist[id+1] : temp2.x;
                if(i == 5){
                    if(orientationHist[id] > maxHist){
                        maxHist = orientationHist[id];
                    }
                }
            }
        }
        maxHist *= orientationThreshold;//% of max orientation value

        float2* bestMagWThetas = new float2[regNumOrient]();
        float2 tempMagWTheta = {0.0f,0.0f};
        for(int b = 0; b < 36; ++b){
            if(orientationHist[b] < maxHist ||
            (b > 0 && orientationHist[b] < orientationHist[b-1]) ||
            (b < 35 && orientationHist[b] < orientationHist[b+1]) ||
            (orientationHist[b] < bestMagWThetas[regNumOrient-1].x)) continue;

            tempMagWTheta.x = orientationHist[b];

            if(b == 0){
              tempMagWTheta.y = (orientationHist[35]-orientationHist[1])/(orientationHist[35]-(2.0f*orientationHist[0])+orientationHist[1]);
            }
            else if(b == 35){
              tempMagWTheta.y = (orientationHist[34]-orientationHist[0])/(orientationHist[34]-(2.0f*orientationHist[35])+orientationHist[0]);
            }
            else{
              tempMagWTheta.y = (orientationHist[b-1]-orientationHist[b+1])/(orientationHist[b-1]-(2.0f*orientationHist[b])+orientationHist[b+1]);
            }

            tempMagWTheta.y *= (pi/36.0f);
            tempMagWTheta.y += (float)b*(pi/18.0f);
            if(tempMagWTheta.y < 0.0f){
              tempMagWTheta.y += 2.0f*pi;
            }

            for(int i = 0; i < regNumOrient; ++i){
              if(tempMagWTheta.x > bestMagWThetas[i].x){
                for(int ii = i; ii < regNumOrient; ++ii){
                  temp2 = bestMagWThetas[ii];
                  bestMagWThetas[ii] = tempMagWTheta;
                  tempMagWTheta = temp2;
                }
              }
            }
        }
        for(int i = 0; i < regNumOrient; ++i){
            if(bestMagWThetas[i].x == 0.0f){
                thetaNumbers[globalID*regNumOrient + i] = -1;
                thetas[globalID*regNumOrient + i] = -1.0f;
            }
            else{
                thetaNumbers[globalID*regNumOrient + i] = globalID + keyPointIndex;
                thetas[globalID*regNumOrient + i] = bestMagWThetas[i].y;
            }
        }
        delete[] bestMagWThetas;
    } 
}

__global__ void ssrlcv::expandKeyPoints(unsigned int numKeyPoints, FeatureFactory::ScaleSpace::SSKeyPoint* keyPointsIn, FeatureFactory::ScaleSpace::SSKeyPoint* keyPointsOut, int* thetaAddresses, float* thetas){
    unsigned int globalID = (blockIdx.y* gridDim.x+ blockIdx.x)*blockDim.x + threadIdx.x;
    if(globalID < numKeyPoints){
        FeatureFactory::ScaleSpace::SSKeyPoint kp = keyPointsIn[thetaAddresses[globalID]];
        kp.theta = thetas[globalID];
        keyPointsOut[globalID] = kp;
    }

}