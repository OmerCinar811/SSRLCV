#include "hip/hip_runtime.h"
#include "common_includes.h"
#include "octree.cuh"
#include "poisson.cuh"
using namespace std;



int main(int argc, char *argv[]){
  try{
    if(argc == 2){
      string filePath = argv[1];
      clock_t totalTimer = clock();
      clock_t partialTimer = clock();

      //if we want further depth than 10 our nodeKeys will need to then be long or long long
      int depth = 10;
      Octree octree = Octree(filePath, depth);
      /*
      KEEP IN MIND THAT NORMALS ARE CURRENTLY READ FROM A PLY AND ARE INWARD FACING
      THIS MEANS THAT NORMALS INSTANTIATION WILL NEED TO BE REMOVED FROM Octree::parsePLY
      AND COLOR WILL BE READ IN ITS PLACE
      */

      octree.init_octree_gpu();
      octree.generateKeys();
      octree.sortByKey();
      octree.compactData();
      octree.fillUniqueNodesAtFinestLevel();
      octree.createFinalNodeArray();
      octree.freePrereqArrays();

      octree.fillLUTs();
      octree.fillNeighborhoods();

      octree.checkForGeneralNodeErrors();

      octree.computeVertexArray();
      octree.computeEdgeArray();
      octree.computeFaceArray();

      partialTimer = clock() - partialTimer;
      printf("\nOCTREE BUILD TOOK %f seconds.\n\n",((float) partialTimer)/CLOCKS_PER_SEC);
      partialTimer = clock();

      /*
      OCTREE HAS BEEN GENERATED NOW ONTO NORMAL COMPUTATION
      //TODO implement this as right now it is read in through the ply
      */

      //octree.computeNormals();

      /*
      RECONTRUCTION PREP HAS COMPLETED NOW ONTO POISSON RECONSTRUCTION
      */
      Poisson poisson = Poisson(octree);

      //poisson.computeLaplacianMatrix();
      //poisson.computeDivergenceVector();
      //poisson.computeImplicitFunction();
      //poisson.marchingCubes();
      //poisson.isosurfaceExtraction();

      hipDeviceReset();
      totalTimer = clock() - totalTimer;
      printf("POISSON RECONSTRUCTION TOOK %f seconds.\n\n",((float) totalTimer)/CLOCKS_PER_SEC);

      return 0;
    }
    else{
      cout<<"LACK OF PLY INPUT...goodbye"<<endl;
      exit(1);
    }
  }
  catch (const std::exception &e){
      std::cerr << "Caught exception: " << e.what() << '\n';
      std::exit(1);
  }
  catch (...){
      std::cerr << "Caught unknown exception\n";
      std::exit(1);
  }

}
