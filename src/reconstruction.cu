#include "hip/hip_runtime.h"
#include "common_includes.h"
#include "octree.cuh"
#include "poisson.cuh"
using namespace std;

//TODO across octree and poisson determine if you really need to instantiate all device array values

int main(int argc, char *argv[]){
  try{
    if(argc == 2){
      string filePath = argv[1];
      clock_t totalTimer = clock();
      clock_t partialTimer = clock();

      //if we want further depth than 10 our nodeKeys will need to then be long or long long
      int depth = 10;
      Octree octree = Octree(filePath, depth);
      /*
      KEEP IN MIND THAT NORMALS ARE CURRENTLY READ FROM A PLY AND ARE INWARD FACING
      THIS MEANS THAT NORMALS INSTANTIATION WILL NEED TO BE REMOVED FROM Octree::parsePLY
      AND COLOR WILL BE READ IN ITS PLACE
      */

      octree.init_octree_gpu();
      octree.generateKeys();
      octree.sortByKey();
      octree.compactData();
      octree.fillUniqueNodesAtFinestLevel();
      octree.createFinalNodeArray();
      octree.freePrereqArrays();

      octree.fillLUTs();
      //octree.printLUTs();
      octree.fillNeighborhoods();

      octree.checkForGeneralNodeErrors();

      octree.computeVertexArray();
      octree.computeEdgeArray();
      octree.computeFaceArray();

      partialTimer = clock() - partialTimer;
      printf("\nOCTREE BUILD TOOK %f seconds.\n\n",((float) partialTimer)/CLOCKS_PER_SEC);
      partialTimer = clock();
      /*
      OCTREE HAS BEEN GENERATED NOW ONTO NORMAL COMPUTATION
      //TODO implement this as right now it is read in through the ply
      */

      //octree.computeNormals();

      /*
      RECONTRUCTION PREP HAS COMPLETED NOW ONTO POISSON RECONSTRUCTION
      */
      //TODO figure out if you want to free octree device data during poisson or leave until full delete octree
      Poisson poisson = Poisson(&octree);

      poisson.computeLUTs();
      poisson.computeDivergenceVector();
      //poisson.computeImplicitFunction();
      //poisson.marchingCubes();
      //poisson.isosurfaceExtraction();

      //hipDeviceReset();
      partialTimer = clock() - partialTimer;
      printf("POISSON RECONSTRUCTION TOOK %f seconds.\n\n",((float) partialTimer)/CLOCKS_PER_SEC);

      totalTimer = clock() - totalTimer;
      printf("TOTAL TIME = %f seconds.\n\n",((float) totalTimer)/CLOCKS_PER_SEC);

      return 0;
    }
    else{
      cout<<"LACK OF PLY INPUT...goodbye"<<endl;
      exit(1);
    }
  }
  catch (const std::exception &e){
      std::cerr << "Caught exception: " << e.what() << '\n';
      std::exit(1);
  }
  catch (...){
      std::cerr << "Caught unknown exception\n";
      std::exit(1);
  }

}
