#include "hip/hip_runtime.h"
#include "Quadtree.cuh"

/*
CLASS AND STRUCT METHODS
*/


template<typename T>
__device__ __host__ ssrlcv::Quadtree<T>::Node::Node(){
  this->key = -1;
  this->dataIndex = -1;
  this->numElements = 0;
  this->center = {-1,-1};
  this->depth = -1;
  this->parent = -1;
  for(int i = 0; i < 4; ++i) this->children[i] = -1;
  for(int i = 0; i < 9; ++i) this->neighbors[i] = -1;
  for(int i = 0; i < 4; ++i) this->edges[i] = -1;
  for(int i = 0; i < 4; ++i) this->vertices[i] = -1;
}
template<typename T>
__device__ __host__ ssrlcv::Quadtree<T>::Vertex::Vertex(){
  this->loc = {-1,-1};
  for(int i = 0; i < 4; ++i) this->nodes[i] = -1;
  this->depth = -1;
}
template<typename T>
__device__ __host__ ssrlcv::Quadtree<T>::Edge::Edge(){
  this->vertices = {-1,-1};
  for(int i = 0; i < 2; ++i) this->nodes[i] = -1;
  this->depth = -1;
}

template<typename T>
ssrlcv::Quadtree<T>::Quadtree(){
  this->width = 0;
  this->imageSize = {0,0};
  this->nodes = nullptr;
  this->data = nullptr;
  this->edges = nullptr;
  this->vertices = nullptr;
  this->imageSize = {0,0};
}
template<typename T>
ssrlcv::Quadtree<T>::Quadtree(uint2 imageSize, ssrlcv::Unity<T>* data){
  this->nodes = nullptr;
  this->edges = nullptr;
  this->vertices = nullptr;
  this->data = data;
  this->imageSize = imageSize;
  this->width = (imageSize.x > imageSize.y) ? imageSize.x : imageSize.y;
  while(this->width % 4) ++this->width;
  this->depth = {0, log2(this->width)};
  this->generateLeafNodes();
  this->generateParentNodes();
}
template<typename T>
ssrlcv::Quadtree<T>::~Quadtree(){
  if(this->nodes != nullptr) delete this->nodes;
  if(this->vertices != nullptr) delete this->vertices;
  if(this->edges != nullptr) delete this->edges;
  if(this->data != nullptr) delete this->data;
  if(this->nodeDepthIndex != nullptr) delete this->nodeDepthIndex;
  if(this->vertexDepthIndex != nullptr) delete this->vertexDepthIndex;
  if(this->edgeDepthIndex != nullptr) delete this->edgeDepthIndex;
}


//TODO ensure numLeafNodes cant go over max int (conditional usage of gridDim.y)
template<typename T>
void ssrlcv::Quadtree<T>::generateLeafNodes(){
  int* leafNodeKeys_device = nullptr;
  float2* leafNodeCenters_device = nullptr;
  unsigned int* nodeDataIndex_device = nullptr;

  unsigned long numLeafNodes = 0;
  numLeafNodes = this->data->numElements;
  CudaSafeCall(hipMalloc((void**)&leafNodeKeys_device, numLeafNodes*sizeof(int)));
  CudaSafeCall(hipMalloc((void**)&leafNodeCenters_device, numLeafNodes*sizeof(float2)));
  dim3 grid = {(numLeafNodes/1024) + 1,1,1};
  dim3 block = {1024,1,1};
  getKeys<<<grid,block>>>(leafNodeKeys_device, leafNodeCenters_device, this->width, this->imageSize, this->depth.y);
  CudaCheckError();

  thrust::counting_iterator<unsigned int> iter(0);
  thrust::device_vector<unsigned int> indices(this->data->numElements);
  thrust::copy(iter, iter + this->data->numElements, indices.begin());

  if(this->depth.y == log2(this->width)){
    CudaSafeCall(hipMalloc((void**)&nodeDataIndex_device, numLeafNodes*sizeof(unsigned int)));
    CudaSafeCall(hipMemcpy(nodeDataIndex_device, thrust::raw_pointer_cast(indices.data()), numLeafNodes*sizeof(unsigned int),hipMemcpyDeviceToDevice));
  }
  else{
    //TO USE IN POINT BASED QUADTREE

    thrust::device_ptr<int> kys(leafNodeKeys_device);
    thrust::sort_by_key(kys, kys + this->data->numElements, indices.begin());

    if(this->data->fore != ssrlcv::gpu){
      this->data->transferMemoryTo(ssrlcv::gpu);
    }

    thrust::device_ptr<float2> cnts(leafNodeCenters_device);
    thrust::device_vector<float2> sortedCnts(this->data->numElements);
    thrust::gather(indices.begin(), indices.end(), cnts, sortedCnts.begin());
    CudaSafeCall(hipMemcpy(leafNodeCenters_device, thrust::raw_pointer_cast(sortedCnts.data()), this->data->numElements*sizeof(float2),hipMemcpyDeviceToDevice));

    thrust::device_ptr<T> dataSorter(this->data->device);
    thrust::device_vector<T> sortedData(this->data->numElements);
    thrust::gather(indices.begin(), indices.end(), dataSorter, sortedData.begin());
    //determine if this is necessary
    this->data->setData(thrust::raw_pointer_cast(sortedData.data()), this->data->numElements, ssrlcv::gpu);
    this->data->transferMemoryTo(ssrlcv::cpu);
    this->data->clearDevice();

    //there may be a faster way to do this
    thrust::pair<int*, thrust::device_vector<unsigned int>::iterator> new_end;//the last value of these node array
    new_end = thrust::unique_by_key(kys,kys + this->data->numElements, indices.begin());
    numLeafNodes = thrust::get<1>(new_end) - indices.begin();

    CudaSafeCall(hipMalloc((void**)&nodeDataIndex_device, numLeafNodes*sizeof(unsigned int)));
    CudaSafeCall(hipMemcpy(nodeDataIndex_device, thrust::raw_pointer_cast(indices.data()), numLeafNodes*sizeof(unsigned int),hipMemcpyDeviceToDevice));

  }

  Node leafNodes_device = nullptr;
  CudaSafeCall(hipMalloc((void**)&leafNodes_device, numLeafNodes*sizeof(Node)));

  grid = {(numLeafNodes/1024) + 1, 1,1};
  block = {1024,1,1};

  fillLeafNodes<<<grid,block>>>(numLeafNodes,leafNodes_device,leafNodeKeys_device,leafNodeCenters_device,nodeDataIndex_device);

  this->nodes = new Unity<Node>(leafNodes_device, numLeafNodes, ssrlcv::gpu);
}

template<typename T>
void ssrlcv::Quadtree<T>::generateParentNodes(){
  if(this->nodes == nullptr || this->nodes->state == ssrlcv::null){
    //TODO potentially develop support for bottom up growth
    throw ssrlcv::NullUnityException("Cannot generate parent nodes before children");
  }

}

/*
CUDA implementations
*/
__global__ void ssrlcv::getKeys(int* keys, float2* nodeCenters, unsigned int width, uint2 imageSize, int depth){
  int globalID = blockIdx.x *blockDim.x + threadIdx.x;
  if(globalID < imageSize.x*imageSize.y){
    int x = globalID%imageSize.x;
    int y = globalID/imageSize.x;
    if(imageSize.x > imageSize.y){
      x += width/2;
    }
    else{
      y += width/2;
    }
    int key = 0;
    unsigned int depth_reg = depth;
    int currentDepth = 1;
    int W = width/2;
    int2 center = {W,W};
    while(W > 1 && depth_reg != currentDepth){
      W /= 2;
      if(x < center.x){
        key <<= 1;
        center.x -= W;
      }
      else{
        key = (key << 1) + 1;
        center.x += W;
      }
      if(y < center.y){
        key <<= 1;
        center.y -= W;
      }
      else{
        key = (key << 1) + 1;
        center.y += W;
      }
      currentDepth++;
    }
    keys[globalID] = key;
    nodeCenters[globalID] = {center.x + 0.5f, center.y + 0.5f};
  }
}

template<typename T>
__global__ void ssrlcv::fillLeafNodes(unsigned long numLeafNodes, typename ssrlcv::Quadtree<T>::Node* leafNodes,
int* keys, float2* nodeCenters, unsigned int* nodeDataIndex){

  int globalID = blockIdx.x *blockDim.x + threadIdx.x;
  if(globalID < numLeafNodes){
    typename Quadtree<T>::Node node = typename Quadtree<T>::Node();
    node.key = keys[globalID];
    node.center = nodeCenters[globalID];
    node.dataIndex = nodeDataIndex[globalID];
    leafNodes[globalID] = node;
  }
}
