#include "hip/hip_runtime.h"
#include "poisson.cuh"

using namespace std;

// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )


inline void __cudaSafeCall(hipError_t err, const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  if (hipSuccess != err) {
      fprintf(stderr, "cudaSafeCall() failed at %s:%i : %s\n",
      file, line, hipGetErrorString(err));
      exit(-1);
  }
#endif

  return;
}
inline void __cudaCheckError(const char *file, const int line) {
#ifdef CUDA_ERROR_CHECK
  hipError_t err = hipGetLastError();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() failed at %s:%i : %s\n",
    file, line, hipGetErrorString(err));
    exit(-1);
  }

  // More careful checking. However, this will affect performance.
  // Comment away if needed.
  //err = hipDeviceSynchronize();
  if (hipSuccess != err) {
    fprintf(stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
    file, line, hipGetErrorString(err));
    exit(-1);
  }
#endif

  return;
}

__device__ __host__ float3 operator+(const float3 &a, const float3 &b) {
  return {a.x+b.x, a.y+b.y, a.z+b.z};
}
__device__ __host__ float3 operator-(const float3 &a, const float3 &b) {
  return {a.x-b.x, a.y-b.y, a.z-b.z};
}
__device__ __host__ float3 operator/(const float3 &a, const float3 &b) {
  return {a.x/b.x, a.y/b.y, a.z/b.z};
}
__device__ __host__ float3 operator*(const float3 &a, const float3 &b) {
  return {a.x*b.x, a.y*b.y, a.z*b.z};
}
__device__ __host__ float dotProduct(const float3 &a, const float3 &b){
  return (a.x*b.x) + (a.y*b.y) + (a.z*b.z);
}
__device__ __host__ float3 operator+(const float3 &a, const float &b){
  return {a.x+b, a.y+b, a.z+b};
}
__device__ __host__ float3 operator-(const float3 &a, const float &b){
  return {a.x-b, a.y-b, a.z-b};
}
__device__ __host__ float3 operator/(const float3 &a, const float &b){
  return {a.x/b, a.y/b, a.z/b};
}
__device__ __host__ float3 operator*(const float3 &a, const float &b){
  return {a.x*b, a.y*b, a.z*b};
}
__device__ __host__ float3 operator+(const float &a, const float3 &b) {
  return {a+b.x, a+b.y, a+b.z};
}
__device__ __host__ float3 operator-(const float &a, const float3 &b) {
  return {a-b.x, a-b.y, a-b.z};
}
__device__ __host__ float3 operator/(const float &a, const float3 &b) {
  return {a/b.x, a/b.y, a/b.z};
}
__device__ __host__ float3 operator*(const float &a, const float3 &b) {
  return {a*b.x, a*b.y, a*b.z};
}
__device__ __host__ bool operator==(const float3 &a, const float3 &b){
  return (a.x==b.x)&&(a.y==b.y)&&(a.z==b.z);
}

//TODO maybe get the third convolution to get closer to gausian filter
__device__ __host__ float3 blender(const float3 &a, const float3 &b, const float &bw){
  float t[3] = {(a.x-b.x)/bw,(a.y-b.y)/bw,(a.z-b.z)/bw};
  float result[3] = {0.0f};
  for(int i = 0; i < 3; ++i){
    if(t[i] > 0.5 && t[i] <= 1.5){
      result[i] = (t[i]-1.5)*(t[i]-1.5)/(bw*bw*bw);
    }
    else if(t[i] < -0.5 && t[i] >= -1.5){
      result[i] = (t[i]+1.5)*(t[i]+1.5)/(bw*bw*bw);
    }
    else if(t[i] <= 0.5 && t[i] >= -0.5){
      result[i] = (1.5-(t[i]*t[i]))/(2.0f*bw*bw*bw);
    }
    else return {0.0f,0.0f,0.0f};
  }
  return {result[0],result[1],result[2]};
}
__device__ __host__ float3 blenderPrime(const float3 &a, const float3 &b, const float &bw){
  float t[3] = {(a.x-b.x)/bw,(a.y-b.y)/bw,(a.z-b.z)/bw};
  float result[3] = {0.0f};
  for(int i = 0; i < 3; ++i){
    if(t[i] > 0.5 && t[i] <= 1.5){
      result[i] = (2.0f*t[i] + 3.0f)/(bw*bw*bw);
    }
    else if(t[i] < -0.5 && t[i] >= -1.5){
      result[i] = (2.0f*t[i] - 3.0f)/(bw*bw*bw);
    }
    else if(t[i] <= 0.5 && t[i] >= -0.5){
      result[i] = (-2.0f*t[i])/(2.0f*bw*bw*bw);
    }
    else return {0.0f,0.0f,0.0f};
  }
  return {result[0],result[1],result[2]};
}
__device__ __host__ float3 blenderPrimePrime(const float3 &a, const float3 &b, const float &bw){
  float t[3] = {(a.x-b.x)/bw,(a.y-b.y)/bw,(a.z-b.z)/bw};
  float result[3] = {0.0f};
  for(int i = 0; i < 3; ++i){
    if(t[i] > 0.5 && t[i] <= 1.5){
      result[i] = 2.0f/(bw*bw*bw);
    }
    else if(t[i] < -0.5 && t[i] >= -1.5){
      result[i] = 2.0f/(bw*bw*bw);
    }
    else if(t[i] <= 0.5 && t[i] >= -0.5){
      result[i] = -1.0f/(bw*bw*bw);
    }
    else return {0.0f,0.0f,0.0f};
  }
  return {result[0],result[1],result[2]};
}

__device__ __host__ int3 splitCrunchBits3(const unsigned int &size, const int &key){
  int3 xyz = {0,0,0};
  for(int i = size - 1;i >= 0;){
    xyz.x = (xyz.x << 1) + ((key >> i) & 1);
    --i;
    xyz.y = (xyz.y << 1) + ((key >> i) & 1);
    --i;
    xyz.z = (xyz.z << 1) + ((key >> i) & 1);
    --i;
  }
  return xyz;
}

__global__ void computeVectorFeild(Node* nodeArray, int numFinestNodes, float3* vectorField, float3* normals, float3* points){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numFinestNodes){
    __shared__ float3 vec;
    vec = {0.0f, 0.0f, 0.0f};
    __syncthreads();
    int neighborIndex = nodeArray[blockID].neighbors[threadIdx.x];
    if(neighborIndex != -1){
      int currentPoint = nodeArray[neighborIndex].pointIndex;
      int stopIndex = nodeArray[neighborIndex].numPoints + currentPoint;
      float3 blend = {0.0f,0.0f,0.0f};
      float width = nodeArray[blockID].width;
      float3 center = nodeArray[blockID].center;
      for(int i = currentPoint; i < stopIndex; ++i){
        //n = 2 Fo(q) make bounds {0.0f, 1.0f}
          //blend = 1.0f - blend;
        //n = 2 Fo(q) make bounds {-1.0f, 0.0f}
          //blend = blend + 1.0f;
        //n currently = 3
        blend = blender(points[i],center,width)*normals[i];
        if(blend.x == 0.0f && blend.y == 0.0f && blend.z == 0.0f) continue;
        atomicAdd(&vec.x, blend.x);
        atomicAdd(&vec.y, blend.y);
        atomicAdd(&vec.z, blend.z);
      }
    }
    __syncthreads();
    if(threadIdx.x != 0) return;
    else vectorField[blockID] = vec;
  }
}
__global__ void computeDivergenceFine(int depthOfOctree, Node* nodeArray, int numNodes, int depthIndex, float3* vectorField, float* divCoeff, float* fPrimeLUT){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numNodes){
    __shared__ float coeff;
    int neighborIndex = nodeArray[blockID + depthIndex].neighbors[threadIdx.x];
    if(neighborIndex != -1){
      int numFinestChildren = nodeArray[neighborIndex].numFinestChildren;
      int finestChildIndex = nodeArray[neighborIndex].finestChildIndex;
      int3 xyz1;
      int3 xyz2;
      xyz1 = splitCrunchBits3(depthOfOctree*3, nodeArray[blockID + depthIndex].key);
      int mult = pow(2,depthOfOctree + 1) - 1;
      for(int i = finestChildIndex; i < finestChildIndex + numFinestChildren; ++i){
        xyz2 = splitCrunchBits3(depthOfOctree*3, nodeArray[i].key);
        atomicAdd(&coeff, dotProduct(vectorField[i], {fPrimeLUT[xyz1.x*mult + xyz2.x],fPrimeLUT[xyz1.y*mult + xyz2.y],fPrimeLUT[xyz1.z*mult + xyz2.z]}));
      }
      __syncthreads();
      //may want only one thread doing this should not matter though
      divCoeff[blockID + depthIndex] = coeff;
    }
  }
}
__global__ void findRelatedChildren(Node* nodeArray, int numNodes, int depthIndex, int2* relativityIndicators){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numNodes){
    __shared__ int numRelativeChildren;
    __shared__ int firstRelativeChild;
    numRelativeChildren = 0;
    firstRelativeChild = 2147483647;//max int
    int neighborIndex = nodeArray[blockID + depthIndex].neighbors[threadIdx.x];
    if(neighborIndex != -1){
      //may not be helping anything by doing this but it prevents 2 accesses to global memory
      int registerChildChecker = nodeArray[neighborIndex].numFinestChildren;
      int registerChildIndex = nodeArray[neighborIndex].finestChildIndex;
      if(registerChildIndex != -1 && registerChildChecker != 0){
        atomicAdd(&numRelativeChildren, nodeArray[neighborIndex].numFinestChildren);
        atomicMin(&firstRelativeChild, nodeArray[neighborIndex].finestChildIndex);
      }
    }
    __syncthreads();
    //may want only one thread doing this should not matter though
    relativityIndicators[blockID].x = firstRelativeChild;
    relativityIndicators[blockID].y = numRelativeChildren;
  }
}
__global__ void computeDivergenceCoarse(int depthOfOctree, Node* nodeArray, int2* relativityIndicators, int currentNode, int depthIndex, float3* vectorField, float* divCoeff, float* fPrimeLUT){
  int globalID = blockIdx.x *blockDim.x + threadIdx.x;
  if(globalID < relativityIndicators[currentNode].y){
    globalID += relativityIndicators[currentNode].x;
    int3 xyz1;
    int3 xyz2;
    xyz1 = splitCrunchBits3(depthOfOctree*3, nodeArray[currentNode + depthIndex].key);
    xyz2 = splitCrunchBits3(depthOfOctree*3, nodeArray[globalID].key);
    int mult = pow(2,depthOfOctree + 1) - 1;
    //TODO try and find a way to optimize this so that it is not using atomics and global memory
    float fx,fy,fz;
    fx = fPrimeLUT[xyz1.x*mult + xyz2.x];
    fy = fPrimeLUT[xyz1.y*mult + xyz2.y];
    fz = fPrimeLUT[xyz1.z*mult + xyz2.z];
    float divergenceContributer = dotProduct(vectorField[globalID], {fx,fy,fz});
    atomicAdd(&divCoeff[currentNode + depthIndex], divergenceContributer);
  }
}

//TODO find out why most laplacians are 0
__global__ void computeLd(int depthOfOctree, Node* nodeArray, int numNodes, int depthIndex, float* offDiagonal, int* offDiagonalIndex, float* diagonals, int* numNonZeroEntries,float* fLUT, float* fPrimePrimeLUT){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numNodes){
    __shared__ int numNonZero;
    numNonZero = 0;
    __syncthreads();
    int neighborIndex = nodeArray[blockID + depthIndex].neighbors[threadIdx.x];
    if(neighborIndex != -1){
      int3 xyz1;
      int3 xyz2;
      xyz1 = splitCrunchBits3(depthOfOctree*3, nodeArray[blockID + depthIndex].key);
      xyz2 = splitCrunchBits3(depthOfOctree*3, nodeArray[neighborIndex].key);
      int mult = pow(2,depthOfOctree + 1) - 1;
      float laplacianValue = (fPrimePrimeLUT[xyz1.x*mult + xyz2.x]*fLUT[xyz1.y*mult + xyz2.y]*fLUT[xyz1.z*mult + xyz2.z])+
      (fLUT[xyz1.x*mult + xyz2.x]*fPrimePrimeLUT[xyz1.y*mult + xyz2.y]*fLUT[xyz1.z*mult + xyz2.z])+
      (fLUT[xyz1.x*mult + xyz2.x]*fLUT[xyz1.y*mult + xyz2.y]*fPrimePrimeLUT[xyz1.z*mult + xyz2.z]);
      if(laplacianValue != 0.0f){
        if(threadIdx.x == 13){
          diagonals[blockID] = laplacianValue;
        }
        else{
          atomicAdd(&numNonZero, 1);
          offDiagonal[blockID*26 + (threadIdx.x > 13 ? threadIdx.x - 1 : threadIdx.x)] = laplacianValue;
          offDiagonalIndex[blockID*26 + (threadIdx.x > 13 ? threadIdx.x - 1 : threadIdx.x)] = neighborIndex;
        }
        //printf("%d,%d -> laplacian = %.9f\n",blockID + depthIndex,neighborIndex, laplacianValue);
      }
      else{
        offDiagonalIndex[threadIdx.x] = -1;
      }
    }
    else{
      offDiagonalIndex[threadIdx.x] = -1;
    }
    __syncthreads();
    numNonZeroEntries[blockID] = numNonZero;
  }
}

__global__ void updateDivergence(int depthOfOctree, Node* nodeArray, int numNodes, int depthIndex, float* divCoeff, float* fLUT, float* fPrimePrimeLUT, float* nodeImplicit){
  int blockID = blockIdx.y * gridDim.x + blockIdx.x;
  if(blockID < numNodes){
    int parent = nodeArray[blockID + depthIndex].parent;
    int parentNeighbor = nodeArray[parent].neighbors[threadIdx.x];
    if(parentNeighbor != -1){
      float nodeImplicitValue = nodeImplicit[parentNeighbor];
      int3 xyz1;
      int3 xyz2;
      xyz1 = splitCrunchBits3(depthOfOctree*3, nodeArray[blockID + depthIndex].key);
      xyz2 = splitCrunchBits3(depthOfOctree*3, nodeArray[parentNeighbor].key);
      int mult = pow(2,depthOfOctree + 1) - 1;
      float laplacianValue = (fPrimePrimeLUT[xyz1.x*mult + xyz2.x]*fLUT[xyz1.y*mult + xyz2.y]*fLUT[xyz1.z*mult + xyz2.z])+(fLUT[xyz1.x*mult + xyz2.x]*fPrimePrimeLUT[xyz1.y*mult + xyz2.y]*fLUT[xyz1.z*mult + xyz2.z])+(fLUT[xyz1.x*mult + xyz2.x]*fLUT[xyz1.y*mult + xyz2.y]*fPrimePrimeLUT[xyz1.z*mult + xyz2.z]);
      atomicAdd(&divCoeff[blockID + depthIndex], -1.0f*laplacianValue*nodeImplicitValue);
    }
  }
}

Poisson::Poisson(Octree* octree){
  this->octree = octree;
  float* divergenceVector = new float[this->octree->totalNodes];
  for(int i = 0; i < this->octree->totalNodes; ++i){
    divergenceVector[i] = 0.0f;
  }
  CudaSafeCall(hipMalloc((void**)&this->divergenceVectorDevice, this->octree->totalNodes*sizeof(float)));
  CudaSafeCall(hipMemcpy(this->divergenceVectorDevice, divergenceVector, this->octree->totalNodes*sizeof(float), hipMemcpyHostToDevice));
  this->octree->copyPointsToDevice();
  this->octree->copyNormalsToDevice();
}

Poisson::~Poisson(){

}

//TODO OPTMIZE THIS YOU FUCK TARD
void Poisson::computeLUTs(){
  clock_t timer;
  timer = clock();

  float currentWidth = this->octree->width;
  float3 currentCenter = this->octree->center;
  float3 tempCenter = {0.0f,0.0f,0.0f};
  int pow2 = 1;
  vector<float3> centers;
  queue<float3> centersTemp;
  centersTemp.push(currentCenter);
  for(int d = 0; d <= this->octree->depth; ++d){
    for(int i = 0; i < pow2; ++i){
      tempCenter = centersTemp.front();
      centersTemp.pop();
      centers.push_back(tempCenter);
      centersTemp.push(tempCenter - (currentWidth/4));
      centersTemp.push(tempCenter + (currentWidth/4));
    }
    currentWidth /= 2;
    pow2 *= 2;
  }
  int numCenters = centers.size();
  printf("number of absolute unique centers = %d\n\n",numCenters);

  unsigned int size = (pow(2, this->octree->depth + 1) - 1);
  float** f = new float*[size];
  float** ff = new float*[size];
  float** fff = new float*[size];
  for(int i = 0; i < size; ++i){
    f[i] = new float[size];
    ff[i] = new float[size];
    fff[i] = new float[size];
  }

  float totalWidth = this->octree->width;
  int pow2i = 1;
  int offseti = 0;
  int pow2j = 1;
  int offsetj = 0;
  for(int i = 0; i <= this->octree->depth; ++i){
    offseti = pow2i - 1;
    pow2j = 1;
    for(int j = 0; j <= this->octree->depth; ++j){
      offsetj = pow2j - 1;
      for(int k = offseti; k < offseti + pow2i; ++k){
        for(int l = offsetj; l < offsetj + pow2j; ++l){
          f[k][l] = dotProduct(blender(centers[l],centers[k],totalWidth/pow2i),blender(centers[k],centers[l],totalWidth/pow2j));
          ff[k][l] = dotProduct(blender(centers[l],centers[k],totalWidth/pow2i),blenderPrime(centers[k],centers[l],totalWidth/pow2j));
          fff[k][l] = dotProduct(blender(centers[l],centers[k],totalWidth/pow2i),blenderPrimePrime(centers[k],centers[l],totalWidth/pow2j));
        }
      }
      pow2j *= 2;
    }
    pow2i *= 2;
  }
  this->fLUT = new float[size*size];
  this->fPrimeLUT = new float[size*size];
  this->fPrimePrimeLUT = new float[size*size];
  for(int i = 0; i < size; ++i){
    for(int j = 0; j < size; ++j){
      this->fLUT[i*size + j] = f[i][j];
      this->fPrimeLUT[i*size + j] = ff[i][j];
      this->fPrimePrimeLUT[i*size + j] = fff[i][j];
    }
  }
  timer = clock() - timer;
  printf("blending LUT generation took %f seconds fully on the CPU.\n\n",((float) timer)/CLOCKS_PER_SEC);
}

//TODO should optimize computeDivergenceCoarse
//TODO THERE ARE MEMORY ACCESS PROBLEMS ORIGINATING PROBABLY FROM LUT STUFF!!!!!!!!!!!!!! FIXXXXXXXXx
void Poisson::computeDivergenceVector(){
  clock_t cudatimer;
  cudatimer = clock();
  /*
  FIRST COMPUTE VECTOR FIELD
  */

  int numNodesAtDepth = 0;
  dim3 grid = {1,1,1};
  dim3 block = {1,1,1};
  numNodesAtDepth = this->octree->depthIndex[1];
  if(numNodesAtDepth < 65535) grid.x = (unsigned int) numNodesAtDepth;
  else{
    grid.x = 65535;
    while(grid.x*grid.y < numNodesAtDepth){
      ++grid.y;
    }
    while(grid.x*grid.y > numNodesAtDepth){
      --grid.x;

    }
    if(grid.x*grid.y < numNodesAtDepth){
      ++grid.x;
    }
  }
  block.x = 27;
  float3* vectorField = new float3[numNodesAtDepth];
  for(int i = 0; i < numNodesAtDepth; ++i){
    vectorField[i] = {0.0f,0.0f,0.0f};
  }
  float3* vectorFieldDevice;
  CudaSafeCall(hipMalloc((void**)&vectorFieldDevice, numNodesAtDepth*sizeof(float3)));
  CudaSafeCall(hipMemcpy(vectorFieldDevice, vectorField, numNodesAtDepth*sizeof(float3), hipMemcpyHostToDevice));
  computeVectorFeild<<<grid,block>>>(this->octree->finalNodeArrayDevice, numNodesAtDepth, vectorFieldDevice, this->octree->normalsDevice, this->octree->pointsDevice);
  hipDeviceSynchronize();//force this to finish as it is necessary for next kernels
  CudaCheckError();
  /*
  CudaSafeCall(hipMemcpy(vectorField, vectorFieldDevice, numNodesAtDepth*sizeof(float3), hipMemcpyDeviceToHost));
  for(int i = 0; i < numNodesAtDepth; ++i){
    if(vectorField[i].x != 0.0f && vectorField[i].y != 0.0f && vectorField[i].z != 0.0f){
      cout<<vectorField[i].x<<","<<vectorField[i].y<<","<<vectorField[i].z<<endl;
    }
  }
  */
  delete[] vectorField;
  cudatimer = clock() - cudatimer;
  printf("Vector field generation kernel took %f seconds.\n\n",((float) cudatimer)/CLOCKS_PER_SEC);
  cudatimer = clock();
  /*
  NOW COMPUTE DIVERGENCE VECTOR AFTER FINDING VECTOR FIELD
  */

  unsigned int size = (pow(2, this->octree->depth + 1) - 1);
  CudaSafeCall(hipMalloc((void**)&this->fPrimeLUTDevice, size*size*sizeof(float)));
  CudaSafeCall(hipMemcpy(this->fPrimeLUTDevice, this->fPrimeLUT, size*size*sizeof(float), hipMemcpyHostToDevice));

  int2* relativityIndicators;
  int2* relativityIndicatorsDevice;
  for(int d = 0; d <= this->octree->depth; ++d){
    block = {27,1,1};
    grid = {1,1,1};
    if(d != this->octree->depth){
      numNodesAtDepth = this->octree->depthIndex[d + 1] - this->octree->depthIndex[d];
    }
    else numNodesAtDepth = 1;

    if(numNodesAtDepth < 65535) grid.x = (unsigned int) numNodesAtDepth;
    else{
      grid.x = 65535;
      while(grid.x*grid.y < numNodesAtDepth){
        ++grid.y;
      }
      while(grid.x*grid.y > numNodesAtDepth){
        --grid.x;
        if(grid.x*grid.y < numNodesAtDepth){
          ++grid.x;//to ensure that numThreads > nodes
          break;
        }
      }
    }
    if(d <= 5){//evaluate divergence coefficients at finer depths
      computeDivergenceFine<<<grid, block>>>(this->octree->depth, this->octree->finalNodeArrayDevice, numNodesAtDepth, this->octree->depthIndex[d], vectorFieldDevice, this->divergenceVectorDevice, this->fPrimeLUTDevice);
      CudaCheckError();
    }
    else{//evaluate divergence coefficients at coarser depths
      relativityIndicators = new int2[numNodesAtDepth];
      for(int i = 0; i < numNodesAtDepth; ++i){
        relativityIndicators[i] = {0,0};
      }
      CudaSafeCall(hipMalloc((void**)&relativityIndicatorsDevice, numNodesAtDepth*sizeof(int2)));
      CudaSafeCall(hipMemcpy(relativityIndicatorsDevice, relativityIndicators, numNodesAtDepth*sizeof(int2), hipMemcpyHostToDevice));
      findRelatedChildren<<<grid, block>>>(this->octree->finalNodeArrayDevice, numNodesAtDepth, this->octree->depthIndex[d], relativityIndicatorsDevice);
      hipDeviceSynchronize();
      CudaCheckError();
      CudaSafeCall(hipMemcpy(relativityIndicators, relativityIndicatorsDevice, numNodesAtDepth*sizeof(int2), hipMemcpyDeviceToHost));
      for(int currentNode = 0; currentNode < numNodesAtDepth; ++currentNode){
        block.x = 1;
        grid.y = 1;
        if(relativityIndicators[currentNode].y == 0) continue;//TODO ensure this assumption is valid
        else if(relativityIndicators[currentNode].y < 65535) grid.x = (unsigned int) relativityIndicators[currentNode].y;
        else{
          grid.x = 65535;
          while(grid.x*block.x < relativityIndicators[currentNode].y){
            ++block.x;
          }
          while(grid.x*block.x > relativityIndicators[currentNode].y){
            --grid.x;
            if(grid.x*block.x < relativityIndicators[currentNode].y){
              ++grid.x;//to ensure that numThreads > nodes
              break;
            }
          }
        }
        computeDivergenceCoarse<<<grid, block>>>(this->octree->depth, this->octree->finalNodeArrayDevice, relativityIndicatorsDevice, currentNode, this->octree->depthIndex[d], vectorFieldDevice, this->divergenceVectorDevice, this->fPrimeLUTDevice);
        CudaCheckError();
      }
      CudaSafeCall(hipFree(relativityIndicatorsDevice));
      delete[] relativityIndicators;
    }
  }
  CudaSafeCall(hipFree(vectorFieldDevice));
  CudaSafeCall(hipFree(this->fPrimeLUTDevice));
  delete[] this->fPrimeLUT;

  cudatimer = clock() - cudatimer;
  printf("Divergence vector generation kernel took %f seconds.\n\n",((float) cudatimer)/CLOCKS_PER_SEC);
}

//TODO separate diagonals and non diagonals
void Poisson::computeImplicitFunction(){
  clock_t cudatimer;
  cudatimer = clock();

  unsigned int size = (pow(2, this->octree->depth + 1) - 1);
  float* nodeImplicit = new float[this->octree->totalNodes];
  for(int i = 0; i < this->octree->totalNodes; ++i){
    nodeImplicit[i] = 0.0f;
  }
  CudaSafeCall(hipMalloc((void**)&this->fLUTDevice, size*size*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&this->fPrimePrimeLUTDevice, size*size*sizeof(float)));
  CudaSafeCall(hipMalloc((void**)&this->nodeImplicitDevice, this->octree->totalNodes*sizeof(float)));
  CudaSafeCall(hipMemcpy(this->fLUTDevice, this->fLUT, size*size*sizeof(float), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(this->fPrimePrimeLUTDevice, this->fPrimePrimeLUT, size*size*sizeof(float), hipMemcpyHostToDevice));
  CudaSafeCall(hipMemcpy(this->nodeImplicitDevice, nodeImplicit, this->octree->totalNodes*sizeof(float), hipMemcpyHostToDevice));

  int numNodesAtDepth = 0;
  float* diagonalsDevice;
  float* offDiagonalsDevice;
  int* offDiagonalIndicesDevice;
  int* numOffDiagonalsDevice;
  float* temp;
  int* numNonZeroEntries;
  float* diag;

  dim3 grid;
  dim3 block;

  for(int d = this->octree->depth; d >= 0; --d){
    //update divergence coefficients based on solutions at coarser depths
    grid = {1,1,1};
    block = {27,1,1};
    if(d != this->octree->depth){
      numNodesAtDepth = this->octree->depthIndex[d + 1] - this->octree->depthIndex[d];
      if(numNodesAtDepth < 65535) grid.x = (unsigned int) numNodesAtDepth;
      else{
        grid.x = 65535;
        while(grid.x*grid.y < numNodesAtDepth){
          ++grid.y;
        }
        while(grid.x*grid.y > numNodesAtDepth){
          --grid.x;
        }
        if(grid.x*grid.y < numNodesAtDepth){
          ++grid.x;
        }
      }
      for(int dcoarse = d + 1; dcoarse <= this->octree->depth; ++dcoarse){
        updateDivergence<<<grid, block>>>(this->octree->depth, this->octree->finalNodeArrayDevice, numNodesAtDepth,
          this->octree->depthIndex[d], this->divergenceVectorDevice,
          this->fLUTDevice, this->fPrimePrimeLUTDevice, this->nodeImplicitDevice);
        CudaCheckError();
      }
    }

    /*
    MULTIGRID SOLVER
    */

    //setup

    temp = new float[numNodesAtDepth*26];
    diag = new float[numNodesAtDepth];
    numNonZeroEntries = new int[numNodesAtDepth];
    for(int i = 0; i < numNodesAtDepth*26; ++i){
      temp[i] = 0;
      if(i % 26 == 0){
        numNonZeroEntries[i/26] = -1;
        diag[i/26] = 0.0f;
      }
      temp[i] = 0.0f;
    }

    CudaSafeCall(hipMalloc((void**)&diagonalsDevice, numNodesAtDepth*sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&offDiagonalsDevice, numNodesAtDepth*26*sizeof(float)));
    CudaSafeCall(hipMalloc((void**)&offDiagonalIndicesDevice, numNodesAtDepth*26*sizeof(int)));
    CudaSafeCall(hipMalloc((void**)&numOffDiagonalsDevice, numNodesAtDepth*sizeof(int)));
    CudaSafeCall(hipMemcpy(diagonalsDevice, diag, numNodesAtDepth*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(offDiagonalsDevice, temp, numNodesAtDepth*26*sizeof(float), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(offDiagonalIndicesDevice, temp, numNodesAtDepth*26*sizeof(int), hipMemcpyHostToDevice));
    CudaSafeCall(hipMemcpy(numOffDiagonalsDevice, numNonZeroEntries, numNodesAtDepth*sizeof(int), hipMemcpyHostToDevice));
    computeLd<<<grid, block>>>(this->octree->depth, this->octree->finalNodeArrayDevice, numNodesAtDepth, this->octree->depthIndex[d],
      offDiagonalsDevice, offDiagonalIndicesDevice, diagonalsDevice, numOffDiagonalsDevice,
      this->fLUTDevice, this->fPrimePrimeLUTDevice);
    CudaCheckError();

    CudaSafeCall(hipMemcpy(diag, diagonalsDevice, numNodesAtDepth*sizeof(float), hipMemcpyDeviceToHost));
    // for(int i = 0; i < numNodesAtDepth; ++i){
    //   cout<<i<<"-"<<diag[i]<<endl;
    // }

    delete[] diag;
    delete[] numNonZeroEntries;
    delete[] temp;


    //multiplication???


    CudaSafeCall(hipFree(diagonalsDevice));
    CudaSafeCall(hipFree(offDiagonalsDevice));
    CudaSafeCall(hipFree(offDiagonalIndicesDevice));
    CudaSafeCall(hipFree(numOffDiagonalsDevice));

  }

  CudaSafeCall(hipFree(this->fLUTDevice));
  CudaSafeCall(hipFree(this->fPrimePrimeLUTDevice));
  CudaSafeCall(hipFree(this->divergenceVectorDevice));
  delete[] this->fLUT;
  delete[] this->fPrimePrimeLUT;
  delete[] nodeImplicit;
  cudatimer = clock() - cudatimer;
  printf("Node Implicit f(n) generation kernel took %f seconds.\n\n",((float) cudatimer)/CLOCKS_PER_SEC);
}

void Poisson::marchingCubes(){
  this->octree->copyPointsToDevice();
  this->octree->copyNormalsToDevice();

  CudaSafeCall(hipFree(this->nodeImplicitDevice));
}
