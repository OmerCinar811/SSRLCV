#include "hip/hip_runtime.h"
#include "common_includes.h"
#include "Image.cuh"
#include "io_util.h"
#include "FeatureFactory.cuh"
#include "MatchFactory.cuh"
#include "PointCloudFactory.cuh"
#include "MeshFactory.cuh"

int main(int argc, char *argv[]){
  try{
    //CUDA INITIALIZATION
    hipInit(0);
    clock_t totalTimer = clock();
    clock_t partialTimer = clock();

    //ARG PARSING
    if(argc < 2 || argc > 4){
      std::cout<<"USAGE ./bin/SFM </path/to/image/directory/>"<<std::endl;
      exit(-1);
    }
    std::string path = argv[1];
    std::vector<std::string> imagePaths = ssrlcv::findFiles(path);

    int numImages = (int) imagePaths.size();

    /*
    DENSE SIFT
    */

    ssrlcv::SIFT_FeatureFactory featureFactory = ssrlcv::SIFT_FeatureFactory();
    std::vector<ssrlcv::Image*> images;
    std::vector<ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>*> allFeatures;
    unsigned int convertColorDepthTo = 1;
    for(int i = 0; i < numImages; ++i){
      ssrlcv::Image* image = new ssrlcv::Image(imagePaths[i],convertColorDepthTo,i);
      //sift border is 24 due to 1xbin would normally be 12
      image->quadtree->setNodeFlags({24.0f+image->quadtree->border.x,24.0f+image->quadtree->border.y},true);
      image->quadtree->writePLY();
      ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>* features = featureFactory.generateFeaturesDensly(image,1);
      allFeatures.push_back(features);
      images.push_back(image);
    }

    ssrlcv::MatchFactory matchFactory = ssrlcv::MatchFactory();
    ssrlcv::Unity<ssrlcv::Match>* matches = matchFactory.generateMatchesBruteForce(images[0],allFeatures[0],images[1],allFeatures[1]);

    //






    return 0;
  }
  catch (const std::exception &e){
      std::cerr << "Caught exception: " << e.what() << '\n';
      std::exit(1);
  }
  catch (const ssrlcv::UnityException &e){
      std::cerr << "Caught exception: " << e.what() << '\n';
      std::exit(1);
  }
  catch (...){
      std::cerr << "Caught unknown exception\n";
      std::exit(1);
  }

}
