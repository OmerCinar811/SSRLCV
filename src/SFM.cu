#include "hip/hip_runtime.h"
#include "common_includes.h"
#include "Image.cuh"
#include "io_util.h"
#include "SIFT_FeatureFactory.cuh"
#include "MatchFactory.cuh"
#include "PointCloudFactory.cuh"
#include "MeshFactory.cuh"

//TODO fix gaussian operators - currently creating very low values


int main(int argc, char *argv[]){
  try{

    //CUDA INITIALIZATION
    hipInit(0);
    clock_t totalTimer = clock();
    clock_t partialTimer = clock();

    //ARG PARSING
    if(argc < 2 || argc > 4){
      std::cout<<"USAGE ./bin/SFM </path/to/image/directory/> </path/to/optional/seedimage.png>"<<std::endl;
      exit(-1);
    }
    std::string path = argv[1];
    std::vector<std::string> imagePaths = ssrlcv::findFiles(path);

    int numImages = (int) imagePaths.size();

    ssrlcv::SIFT_FeatureFactory featureFactory = ssrlcv::SIFT_FeatureFactory(1.5f,6.0f);
    ssrlcv::MatchFactory<ssrlcv::SIFT_Descriptor> matchFactory = ssrlcv::MatchFactory<ssrlcv::SIFT_Descriptor>(0.6f,250.0f*250.0f);

    /*
    FEATURE EXTRACTION
    */
    //seed features extraction

    ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>* seedFeatures = nullptr;
    if(argc == 3){
      std::string seedPath = argv[2];
      ssrlcv::Image* seed = new ssrlcv::Image(seedPath,-1);
      seedFeatures = featureFactory.generateFeatures(seed,false,2,0.8);
      matchFactory.setSeedFeatures(seedFeatures);
      delete seed;
    }

    std::vector<ssrlcv::Image*> images;
    std::vector<ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>*> allFeatures;
    for(int i = 0; i < numImages; ++i){
      ssrlcv::Image* image = new ssrlcv::Image(imagePaths[i],i);
      ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>* features = featureFactory.generateFeatures(image,false,2,0.8);
      features->transferMemoryTo(ssrlcv::cpu);
      images.push_back(image);
      allFeatures.push_back(features);
    }


    /*
    MATCHING
    */
    //seeding with false photo

    std::cout << "Starting matching..." << std::endl;
    ssrlcv::Unity<float>* seedDistances = (argc == 3) ? matchFactory.getSeedDistances(allFeatures[0]) : nullptr;
    ssrlcv::Unity<ssrlcv::DMatch>* distanceMatches = matchFactory.generateDistanceMatches(images[0],allFeatures[0],images[1],allFeatures[1],seedDistances);
    if(seedDistances != nullptr) delete seedDistances;

    distanceMatches->transferMemoryTo(ssrlcv::cpu);
    float maxDist = 0.0f;
    for(int i = 0; i < distanceMatches->numElements; ++i){
      if(maxDist < distanceMatches->host[i].distance) maxDist = distanceMatches->host[i].distance;
    }
    printf("max euclidean distance between features = %f\n",maxDist);
    if(distanceMatches->state != ssrlcv::gpu) distanceMatches->setMemoryState(ssrlcv::gpu);
    ssrlcv::Unity<ssrlcv::Match>* matches = matchFactory.getRawMatches(distanceMatches);
    delete distanceMatches;
    std::string delimiter = "/";
    std::string matchFile = imagePaths[0].substr(0,imagePaths[0].rfind(delimiter)) + "/matches.txt";
    ssrlcv::writeMatchFile(matches, matchFile);

    /*
    STEREODISPARITY
    */
    ssrlcv::PointCloudFactory demPoints = ssrlcv::PointCloudFactory();
    ssrlcv::Unity<float3>* points = demPoints.stereo_disparity(matches,8.0);

    delete matches;
    ssrlcv::writePLY("out/test.ply",points);
    delete points;

    for(int i = 0; i < imagePaths.size(); ++i){
      delete images[i];
      delete allFeatures[i];
    }

    return 0;
  }
  catch (const std::exception &e){
      std::cerr << "Caught exception: " << e.what() << '\n';
      std::exit(1);
  }
  catch (...){
      std::cerr << "Caught unknown exception\n";
      std::exit(1);
  }

}
