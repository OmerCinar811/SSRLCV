#include "hip/hip_runtime.h"

// _______________________________________________________________________________________________________________
//  _____/\\\\\\\\\\\_______/\\\\\\\\\\\______/\\\\\\\\\______/\\\____________________/\\\\\\\\\__/\\\________/\\\_
//   ___/\\\/////////\\\___/\\\/////////\\\__/\\\///////\\\___\/\\\_________________/\\\////////__\/\\\_______\/\\\_
//    __\//\\\______\///___\//\\\______\///__\/\\\_____\/\\\___\/\\\_______________/\\\/___________\//\\\______/\\\__
//     ___\////\\\___________\////\\\_________\/\\\\\\\\\\\/____\/\\\______________/\\\______________\//\\\____/\\\___
//      ______\////\\\___________\////\\\______\/\\\//////\\\____\/\\\_____________\/\\\_______________\//\\\__/\\\____
//       _________\////\\\___________\////\\\___\/\\\____\//\\\___\/\\\_____________\//\\\_______________\//\\\/\\\_____
//        __/\\\______\//\\\___/\\\______\//\\\__\/\\\_____\//\\\__\/\\\______________\///\\\______________\//\\\\\______
//         _\///\\\\\\\\\\\/___\///\\\\\\\\\\\/___\/\\\______\//\\\_\/\\\\\\\\\\\\\\\____\////\\\\\\\\\______\//\\\_______
//          ___\///////////_______\///////////_____\///________\///__\///////////////________\/////////________\///________
//           _______________________________________________________________________________________________________________



#include "common_includes.h"
#include "Image.cuh"
#include "io_util.h"
#include "SIFT_FeatureFactory.cuh"
#include "MatchFactory.cuh"
#include "PointCloudFactory.cuh"
#include "MeshFactory.cuh"

//TODO fix gaussian operators - currently creating very low values


int main(int argc, char *argv[]){
  try{

    //CUDA INITIALIZATION
    hipInit(0);
    clock_t totalTimer = clock();
    clock_t partialTimer = clock();

    //ARG PARSING

    std::map<std::string,ssrlcv::arg*> args = ssrlcv::parseArgs(argc,argv);
    if(args.find("dir") == args.end()){
      std::cerr<<"ERROR: SFM executable requires a directory of images"<<std::endl;
      exit(-1);
    }
    ssrlcv::SIFT_FeatureFactory featureFactory = ssrlcv::SIFT_FeatureFactory(1.5f,6.0f);
    ssrlcv::MatchFactory<ssrlcv::SIFT_Descriptor> matchFactory = ssrlcv::MatchFactory<ssrlcv::SIFT_Descriptor>(0.6f,200.0f*200.0f);
    bool seedProvided = false;
    ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>* seedFeatures = nullptr;
    if(args.find("seed") != args.end()){
      seedProvided = true;
      std::string seedPath = ((ssrlcv::img_arg*)args["seed"])->path;
      ssrlcv::Image* seed = new ssrlcv::Image(seedPath,-1);
      seedFeatures = featureFactory.generateFeatures(seed,false,2,0.8);
      matchFactory.setSeedFeatures(seedFeatures);
      delete seed;
    }
    std::vector<std::string> imagePaths = ((ssrlcv::img_dir_arg*)args["dir"])->paths;
    int numImages = (int) imagePaths.size();
    std::cout<<"found "<<numImages<<" in directory given"<<std::endl;

    std::vector<ssrlcv::Image*> images;
    std::vector<ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>*> allFeatures;
    for(int i = 0; i < numImages; ++i){
      ssrlcv::Image* image = new ssrlcv::Image(imagePaths[i],i);
      ssrlcv::Unity<ssrlcv::Feature<ssrlcv::SIFT_Descriptor>>* features = featureFactory.generateFeatures(image,false,2,0.8);
      features->transferMemoryTo(ssrlcv::cpu);
      images.push_back(image);
      allFeatures.push_back(features);
    }

    //
    // MATCHING
    //

    std::cout << "Starting matching..." << std::endl;

    ssrlcv::Unity<float>* seedDistances = (seedProvided) ? matchFactory.getSeedDistances(allFeatures[0]) : nullptr;
    ssrlcv::Unity<ssrlcv::DMatch>* distanceMatches = matchFactory.generateDistanceMatches(images[0],allFeatures[0],images[1],allFeatures[1],seedDistances);
    if(seedDistances != nullptr) delete seedDistances;

    distanceMatches->transferMemoryTo(ssrlcv::cpu);
    float maxDist = 0.0f;
    for(int i = 0; i < distanceMatches->size(); ++i){
      if(maxDist < distanceMatches->host[i].distance) maxDist = distanceMatches->host[i].distance;
    }
    printf("max euclidean distance between features = %f\n",maxDist);
    if(distanceMatches->getMemoryState() != ssrlcv::gpu) distanceMatches->setMemoryState(ssrlcv::gpu);
    ssrlcv::Unity<ssrlcv::Match>* matches = matchFactory.getRawMatches(distanceMatches);
    delete distanceMatches;

    std::string delimiter = "/";
    std::string matchFile = imagePaths[0].substr(0,imagePaths[0].rfind(delimiter)) + "/matches.txt";
    // ssrlcv::writeMatchFile(matches, matchFile);

    // Need to fill into to MatchSet boi
    std::cout << "Generating MatchSet ..." << std::endl;
    ssrlcv::MatchSet matchSet;

    if (images.size() == 2){
      //
      // 2 View Case
      //
      matchSet.keyPoints = new ssrlcv::Unity<ssrlcv::KeyPoint>(nullptr,matches->size()*2,ssrlcv::cpu);
      matchSet.matches = new ssrlcv::Unity<ssrlcv::MultiMatch>(nullptr,matches->size(),ssrlcv::cpu);
      matches->setMemoryState(ssrlcv::cpu);
      matchSet.matches->setMemoryState(ssrlcv::cpu);
      matchSet.keyPoints->setMemoryState(ssrlcv::cpu);
      for(int i = 0; i < matchSet.matches->size(); i++){
        matchSet.keyPoints->host[i*2] = matches->host[i].keyPoints[0];
        matchSet.keyPoints->host[i*2 + 1] = matches->host[i].keyPoints[1];
        matchSet.matches->host[i] = {2,i*2};
      }
      std::cout << "Generated MatchSet ..." << std::endl << "Total Matches: " << matches->size() << std::endl << std::endl;
    } else {
      //
      // N View Case
      //
      matchSet = matchFactory.generateMatchesExaustive(images,allFeatures);
      matches->setMemoryState(ssrlcv::cpu);
      matchSet.matches->setMemoryState(ssrlcv::cpu);
      matchSet.keyPoints->setMemoryState(ssrlcv::cpu);

      // optional to save output
      // matchSet.keyPoints->checkpoint(0,"out/kp");
      // matchSet.matches->checkpoint(0,"out/m");
    }

    // the bois
    ssrlcv::PointCloudFactory demPoints = ssrlcv::PointCloudFactory();
    ssrlcv::MeshFactory meshBoi = ssrlcv::MeshFactory();
    ssrlcv::Unity<float3>* points;
    ssrlcv::Unity<float>* errors;
    ssrlcv::BundleSet bundleSet;

    if (images.size() == 2){
      //
      // 2 View Case
      //
      std::cout << "Attempting 2-view Triangulation" << std::endl;

      float* linearError = (float*)malloc(sizeof(float));
      bundleSet = demPoints.generateBundles(&matchSet,images);
      points = demPoints.twoViewTriangulate(bundleSet, linearError);
      ssrlcv::writePLY("out/unfiltered.ply",points);
      demPoints.saveDebugLinearErrorCloud(&matchSet,images, "linearErrorsColored");
      // it's good to do a cutoff filter first how this is chosen is mostly based on ur gut
      // if a poor estimate is chosen then you will have to statistical filter multiple times
      // option 1: pick a fixed value
        demPoints.linearCutoffFilter(&matchSet,images,1000); // <--- removes linear errors over 1000
      // option 2: tie the initial cutoff to some fraction of the initial linear error
        // demPoints.linearCutoffFilter(&matchSet,images,*linearError / (bundleSet.bundles->size() * 3));
      // option 3: don't use the linear cutoff at all and just use multiple statistical filters (it is safer)
      bundleSet = demPoints.generateBundles(&matchSet,images);
      points = demPoints.twoViewTriangulate(bundleSet, linearError);
      ssrlcv::writePLY("out/linearCutoff.ply",points);
      // here you can filter points in a number of ways before bundle adjustment or triangulation
      demPoints.deterministicStatisticalFilter(&matchSet,images, 1.0, 0.1); // <---- samples 10% of points and removes anything past 3.0 sigma
      bundleSet = demPoints.generateBundles(&matchSet,images);

      /*
      // OPTIONAL
      // a second filter can re-filter the new error histogram
      // this is usually a good idea, as there will be new relative extrema to remove
      // doing this too many times will simply over filter the point cloud
      demPoints.deterministicStatisticalFilter(&matchSet,images, 2.0, 0.1); // <---- samples 10% of points and removes anything past 2.0 sigma
      bundleSet = demPoints.generateBundles(&matchSet,images);
      */

      // Planar filtering is very good at removing noise that is not close to the estimated model.
      demPoints.planarCutoffFilter(&matchSet, images, 10.0f); // <---- this will remove any points more than +/- 10 km from the  estimated plane

      // OPTIONAL
      // to visualize the estimated plane which the structure lies within you can use
      // the demPoints.visualizePlaneEstimation() method like so:
      demPoints.visualizePlaneEstimation(points, images, "planeEstimation");

      // the version that will be used normally
      points = demPoints.twoViewTriangulate(bundleSet, linearError);
      std::cout << "Total Linear Error: " << std::fixed << std::setprecision(12) << *linearError << std::endl;

      /*
      // OPTIONAL
      // a sensitivity analysis allows one to view the functions and camera parameter derivates pre bundles adjustment
      // this should not be used in produciton and is really only useful for debugging optimizers used in bundle adjustment
      std::string temp_filename = "sensitivity";
      demPoints.generateSensitivityFunctions(&matchSet,images,temp_filename);
      */

      /*
      // OPTIONAL
      // Varoius scaling examples:
        demPoints.scalePointCloud(10.0,points);
        ssrlcv::writePLY("out/scaledx10.ply",points);
        demPoints.scalePointCloud(10.0,points);
        ssrlcv::writePLY("out/scaledx100.ply",points);
        demPoints.scalePointCloud(10.0,points);
        ssrlcv::writePLY("out/scaledx1000.ply",points);
      */

      // OPTIONAL
      // to compare a points cloud with a ground truth model the first need to be scaled
      // the distance values here are in km but most truth models are in meters
      demPoints.scalePointCloud(1000.0,points); // scales from km into meters
      // rotate pi around the y axis
      float3 rotation = {0.0f, PI, 0.0f};
      demPoints.rotatePointCloud(rotation, points);
      // load the example mesh to do the comparison, here I assume we are using the everst PLY
      meshBoi.loadMesh("data/truth/Everest_ground_truth.ply");
        // to save a mesh as a PLY simply:
        // meshBoi.saveMesh("testMesh");
      // to calculate the "missmatch" between the point cloud and the ground truth you can use this method:
      float error = meshBoi.calculateAverageDifference(points, {0.0f , 0.0f, 1.0f}); // (0,0,1) is the Normal to the X-Y plane, which the point cloud and mesh are on
      std::cout << "Average error to ground truth is: " << error << " km, " << (error * 1000) << " meters" << std::endl;
      // this methods saves the error on each point
      ssrlcv::Unity<float>* truthErrors = meshBoi.calculatePerPointDifference(points, {0.0f , 0.0f, 1.0f});
      // then you can save these errors in a CSV
      ssrlcv::writeCSV(truthErrors, "resolutionErrors");
      // you can also save them as color coded
      ssrlcv::writePLY("resolutionErrors",points, truthErrors, 300); // NOTE it has already been scaled to meters, set error the cutoff to 300 meters

      /*
      // OPTIONAL
      // Tests can be done with bundle adjustment to check bounds on how
      // well it performs
      ssrlcv::Unity<float>* noise = new ssrlcv::Unity<float>(nullptr,6,ssrlcv::cpu);
      noise->host[0] = 0.0; // X
      noise->host[1] = 0.2; // Y
      noise->host[2] = 0.0; // Z
      noise->host[3] = 0.0; // X^
      noise->host[4] = 0.0; // Y^
      noise->host[5] = 0.0; // Z^
      demPoints.testBundleAdjustmentTwoView(&matchSet,images, 10, noise);
      */

      // starting bundle adjustment here
      // std::cout << "Starting Bundle Adjustment Loop ..." << std::endl;
      // points = demPoints.BundleAdjustTwoView(&matchSet,images, 10);


    } else {
      //
      // N View Case
      //
      std::cout << "Attempting N-view Triangulation" << std::endl;

      // if we are checkout errors
      errors = new ssrlcv::Unity<float>(nullptr,matchSet.matches->size(),ssrlcv::cpu);

      float* angularError = (float*)malloc(sizeof(float));
      bundleSet = demPoints.generateBundles(&matchSet,images);
      points = demPoints.nViewTriangulate(bundleSet, errors, angularError);
      ssrlcv::writePLY("out/unfiltered.ply",points);

      demPoints.saveDebugLinearErrorCloud(&matchSet,images, "linearErrorsColored");
      demPoints.saveViewNumberCloud(&matchSet,images, "ViewNumbers");

      std::cout << "Initial Angular Error: " << *angularError << std::endl;
      //ssrlcv::writeCSV(errors->host, (int) errors->size(), "individualAngularErrors1");

      demPoints.linearCutoffFilter(&matchSet,images,300);
      bundleSet = demPoints.generateBundles(&matchSet,images);

      // Planar filtering is very good at removing noise that is not close to the estimated model.
      demPoints.planarCutoffFilter(&matchSet, images, 10.0f); // <---- this will remove any points more than +/- 10 km from the  estimated plane

      // OPTIONAL
      // to visualize the estimated plane which the structure lies within you can use
      // the demPoints.visualizePlaneEstimation() method like so:
      demPoints.visualizePlaneEstimation(points, images, "planeEstimation");

      // multiple filters are needed, because outlier points are discovered in stages
      // decreasing sigma over time is best because the real "mean" error becomes more
      // accurate as truely noisey points are removed
      for (int i = 0; i < 3; i++){
        demPoints.deterministicStatisticalFilter(&matchSet,images, 3.0, 0.1); // <---- samples 10% of points and removes anything past 3.0 sigma
        bundleSet = demPoints.generateBundles(&matchSet,images);
      }
      for (int i = 0; i < 6; i++){
        demPoints.deterministicStatisticalFilter(&matchSet,images, 1.0, 0.1); // <---- samples 10% of points and removes anything past 1.0 sigma
        bundleSet = demPoints.generateBundles(&matchSet,images);
      }
      // then, if the cloud is large enough still, one last filter
      demPoints.deterministicStatisticalFilter(&matchSet,images, 0.2, 0.1); // <---- samples 10% of points and removes anything past 0.2 sigma
      bundleSet = demPoints.generateBundles(&matchSet,images);

      // now redo triangulation with the newlyfiltered boi
      points = demPoints.nViewTriangulate(bundleSet, errors, angularError);

      // OPTIONAL
      // to compare a points cloud with a ground truth model the first need to be scaled
      // the distance values here are in km but most truth models are in meters
      demPoints.scalePointCloud(1000.0,points); // scales from km into meters
      // rotate pi around the y axis
      float3 rotation = {0.0f, PI, 0.0f};
      demPoints.rotatePointCloud(rotation, points);
      // you can compare to a "ground truth" mesh
      // load the example mesh to do the comparison, here I assume we are using the everst PLY
      meshBoi.loadMesh("data/truth/Everest_ground_truth.ply");
      float error = meshBoi.calculateAverageDifference(points, {0.0f , 0.0f, 1.0f}); // (0,0,1) is the Normal to the X-Y plane, which the point cloud and mesh are on
      std::cout << "Average error to ground truth is: " << error << " km, " << (error * 1000) << " meters" << std::endl;
      ssrlcv::Unity<float>* truthErrors = meshBoi.calculatePerPointDifference(points, {0.0f , 0.0f, 1.0f});
      // then you can save these errors in a CSV
      ssrlcv::writeCSV(truthErrors, "resolutionErrors");
      // you can also save them as color coded
      ssrlcv::writePLY("resolutionErrors",points, truthErrors, 300); // NOTE it has already been scaled to meters, set error the cutoff to 300 meters


      //ssrlcv::writeCSV(errors->host, (int) errors->size(), "individualAngularErrors2");
      demPoints.saveDebugCloud(points, bundleSet, images);

    }

    std::cout << "writing final PLY ..." << std::endl;
    ssrlcv::writePLY("final",points);

    // cleanup
    delete points;
    delete matches;
    delete matchSet.matches;
    delete matchSet.keyPoints;
    delete bundleSet.bundles;
    delete bundleSet.lines;
    for(int i = 0; i < imagePaths.size(); ++i){
      delete images[i];
      delete allFeatures[i];
    }

    return 0;
  }
  catch (const std::exception &e){
      std::cerr << "Caught exception: " << e.what() << '\n';
      std::exit(1);
  }
  catch (...){
      std::cerr << "Caught unknown exception\n";
      std::exit(1);
  }

}

























































// yeet
