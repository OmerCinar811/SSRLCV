#include "hip/hip_runtime.h"
#include "matrix_util.cuh"

__device__ __host__ float ssrlcv::sum(const float3 &a){
  return a.x + a.y + a.z;
}
__device__ __host__ void ssrlcv::multiply(const float (&A)[9], const float (&B)[3][3], float (&C)[3][3]){
  for(int r = 0; r < 3; ++r){
    for(int c = 0; c < 3; ++c){
      float entry = 0;
      for(int z = 0; z < 3; ++z){
        entry += A[r*3 + z]*B[z][c];
      }
      C[r][c] = entry;
    }
  }
}
__device__ __host__ void ssrlcv::multiply(const float3 (&A)[3], const float3 (&B)[3], float3 (&C)[3]){
  for(int r = 0; r < 3; ++r){
    C[r].x = (A[r].x*B[0].x) + (A[r].y*B[1].x) + (A[r].z*B[2].x);
    C[r].y = (A[r].x*B[0].y) + (A[r].y*B[1].y) + (A[r].z*B[2].y);
    C[r].z = (A[r].x*B[0].z) + (A[r].y*B[1].z) + (A[r].z*B[2].z);
  }
}
__device__ __host__ void ssrlcv::multiply(const float (&A)[3][3], const float (&B)[3][3], float (&C)[3][3]){
  for(int r = 0; r < 3; ++r){
    for(int c = 0; c < 3; ++c){
      float entry = 0;
      for(int z = 0; z < 3; ++z){
        entry += A[r][z]*B[z][c];
      }
      C[r][c] = entry;
    }
  }
}
__device__ __host__ void ssrlcv::multiply(const float (&A)[9], const float (&B)[3], float (&C)[3]){
   for (int r = 0; r < 3; ++r){
    float val = 0;
    for (int c = 0; c < 3; ++c){
      val += A[r*3 + c] * B[c];
    }
    C[r] = val;
  }
}
__device__ __host__ void ssrlcv::multiply(const float3 (&A)[3], const float3 &B, float3 &C){
  C.x = (A[0].x * B.x) + (A[0].y * B.y) + (A[0].z * B.z);
  C.y = (A[1].x * B.x) + (A[1].y * B.y) + (A[1].z * B.z);
  C.z = (A[2].x * B.x) + (A[2].y * B.y) + (A[2].z * B.z);
}

__device__ __host__ void ssrlcv::multiply(const float (&A)[3][3], const float (&B)[3], float (&C)[3]){
  for (int r = 0; r < 3; ++r){
    float val = 0;
    for (int c = 0; c < 3; ++c){
      val += A[r][c] * B[c];
    }
    C[r] = val;
  }
}
__device__ __host__ void ssrlcv::multiply(const float (&A)[3], const float (&B)[3][3], float (&C)[3]){
  for (int c = 0; c < 3; ++c){
    float val = 0;
    for (int r = 0; r < 3; ++r){
      val += B[r][c] * A[r];
    }
    C[c] = val;
  }
}
__device__ __host__ void ssrlcv::multiply(const float (&A)[2][2], const float (&B)[2][2], float (&C)[2][2]){
   for(int r = 0; r < 2; ++r){
    for(int c = 0; c < 2; ++c){
      float entry = 0;
      for(int z = 0; z < 3; ++z){
        entry += A[r][z]*B[z][c];
      }
      C[r][c] = entry;
    }
  }
}

__device__ __host__ float ssrlcv::dotProduct(const float (&A)[3], const float (&B)[3]){
  return (A[0]*B[0]) + (A[1]*B[1]) + (A[2]*B[2]);
}

__device__ __host__ bool ssrlcv::inverse(const float (&M)[3][3], float (&M_out)[3][3]){
  float d1 = M[1][1] * M[2][2] - M[2][1] * M[1][2];
  float d2 = M[1][0] * M[2][2] - M[1][2] * M[2][0];
  float d3 = M[1][0] * M[2][1] - M[1][1] * M[2][0];
  float det = M[0][0]*d1 - M[0][1]*d2 + M[0][2]*d3;
  if(det == 0){
    return false;
  }
  float invdet = 1/det;
  M_out[0][0] = d1*invdet;
  M_out[0][1] = (M[0][2]*M[2][1] - M[0][1]*M[2][2]) * invdet;
  M_out[0][2] = (M[0][1]*M[1][2] - M[0][2]*M[1][1]) * invdet;
  M_out[1][0] = -1 * d2 * invdet;
  M_out[1][1] = (M[0][0]*M[2][2] - M[0][2]*M[2][0]) * invdet;
  M_out[1][2] = (M[1][0]*M[0][2] - M[0][0]*M[1][2]) * invdet;
  M_out[2][0] = d3 * invdet;
  M_out[2][1] = (M[2][0]*M[0][1] - M[0][0]*M[2][1]) * invdet;
  M_out[2][2] = (M[0][0]*M[1][1] - M[1][0]*M[0][1]) * invdet;
  return true;
}
__device__ __host__ bool ssrlcv::inverse(const float3 (&M)[3], float3 (&M_out)[3]){
  float d1 = M[1].y * M[2].z - M[2].y * M[1].z;
  float d2 = M[1].x * M[2].z - M[1].z * M[2].x;
  float d3 = M[1].x * M[2].y - M[1].y * M[2].x;
  float det = M[0].x*d1 - M[0].y*d2 + M[0].z*d3;
  if(det == 0){
    return false;
  }
  float invdet = 1/det;
  M_out[0].x = d1*invdet;
  M_out[0].y = (M[0].z*M[2].y - M[0].y*M[2].z) * invdet;
  M_out[0].z = (M[0].y*M[1].z - M[0].z*M[1].y) * invdet;
  M_out[1].x = -1 * d2 * invdet;
  M_out[1].y = (M[0].x*M[2].z - M[0].z*M[2].x) * invdet;
  M_out[1].z = (M[1].x*M[0].z - M[0].x*M[1].z) * invdet;
  M_out[2].x = d3 * invdet;
  M_out[2].y = (M[2].x*M[0].y - M[0].x*M[2].y) * invdet;
  M_out[2].z = (M[0].x*M[1].y - M[1].x*M[0].y) * invdet;
  return true;
}
__device__ __host__ void ssrlcv::transpose(const float (&M)[3][3], float (&M_out)[3][3]){
  for(int r = 0; r < 3; ++r){
    for(int c = 0; c < 3; ++c){
      M_out[r][c] = M[c][r];
    }
  }
}
__device__ __host__ void ssrlcv::transpose(const float3 (&M)[3], float3 (&M_out)[3]){
  M_out[0].x = M[0].x;
  M_out[0].y = M[1].x;
  M_out[0].z = M[2].x;
  M_out[1].x = M[0].y;
  M_out[1].y = M[1].y;
  M_out[1].z = M[2].y;
  M_out[2].x = M[0].z;
  M_out[2].y = M[1].z;
  M_out[2].z = M[2].z;
}

__device__ __host__ void ssrlcv::transpose(const float (&M)[2][2], float (&M_out)[2][2]){
  for(int r = 0; r < 2; ++r){
    for(int c = 0; c < 2; ++c){
      M_out[c][r] = M[r][c];
    }
  }
}
__device__ __host__ float ssrlcv::determinant(const float (&M)[2][2]){
  return (M[0][0]*M[1][1]) - (M[0][1]*M[1][0]);
}
__device__ __host__ float ssrlcv::trace(const float(&M)[2][2]){
  return M[0][0] + M[1][1];
}
__device__ __host__ float ssrlcv::trace(const float(&M)[3][3]){
  return M[0][0] + M[1][1] + M[2][2];
}


__device__ __host__ void ssrlcv::normalize(float (&v)[3]){
  float mag = magnitude(v);
  if(mag > 0){
    v[0] = v[0]/mag;
    v[1] = v[1]/mag;
    v[2] = v[2]/mag;
  }
}
__device__ __host__ void ssrlcv::normalize(float3 &v){
  float mag = magnitude(v);
  if(mag > 0){
    v.x = v.x/mag;
    v.y = v.y/mag;
    v.z = v.z/mag;
  }
}
__device__ __host__ float ssrlcv::magnitude(const float (&v)[3]){
  return sqrtf(dotProduct({v[0],v[1],v[2]}, {v[0],v[1],v[2]}));
}
__device__ __host__ float ssrlcv::magnitude(const float3 &v){
  return sqrtf(dotProduct(v, v));
}

__device__ float3 ssrlcv::matrixMulVector(float3 x, float A[3][3]){
  float temp[3] = {x.x, x.y, x.z};
  float b[3];
  for (int r = 0; r < 3; ++r)
  {
    float val = 0;
    for (int c = 0; c < 3; ++c)
    {
      val += A[r][c] * temp[c];
    }
    b[r] = val;
  }
  return {b[0], b[1], b[2]};
}

__device__ float3 ssrlcv::getVectorAngles(float3 v){
  float3 angles;
  float3 x_n = {1.0f, 0.0f, 0.0f};
  float3 y_n = {0.0f, 1.0f, 0.0f};
  float3 z_n = {0.0f, 0.0f, 1.0f};
  // x angle
  float a = dotProduct(v,x_n);
  float b = dotProduct(v,v);
  float c = (a)/(sqrtf(b));
  angles.x = acosf(c);
  // y angle
  a = dotProduct(v,y_n);
  b = dotProduct(v,v);
  c = (a)/(sqrtf(b));
  angles.y = acosf(c);
  // z angle
  a = dotProduct(v,z_n);
  b = dotProduct(v,v);
  c = (a)/(sqrtf(b));
  angles.z = acosf(c);
  return angles;
}

__device__ float3 ssrlcv::rotatePoint(float3 point, float3 angle) {
  // this is just a 3D rotation matrix
  // contains all R3 rotations multiplied together
  float rotationMatrix[3][3];
  rotationMatrix[0][0] = cosf(angle.y) * cosf(angle.z);
  rotationMatrix[0][1] = -1 * cosf(angle.y) * sinf(angle.z);
  rotationMatrix[0][2] = sinf(angle.y);
  rotationMatrix[1][0] = sinf(angle.x) * sinf(angle.y) * cosf(angle.z) + cosf(angle.x) * sinf(angle.z);
  rotationMatrix[1][1] = cosf(angle.x) * cosf(angle.z) - sinf(angle.x) * sinf(angle.y) * sinf(angle.z);
  rotationMatrix[1][2] = -1 * sinf(angle.x) * cosf(angle.y);
  rotationMatrix[2][0] = sinf(angle.x) * sinf(angle.z) - cosf(angle.x) * sinf(angle.y) * cosf(angle.z);
  rotationMatrix[2][1] = cosf(angle.x) * sinf(angle.y) * sinf(angle.z) + sinf(angle.x) * cosf(angle.z);
  rotationMatrix[2][2] = cosf(angle.x) * cosf(angle.y);
  point = matrixMulVector(point, rotationMatrix);
  return point;
}

// this assumes we are ALWAYS starting with a rotation
// this should only be used for reprojection
// TODO this could be combined into one matrix, and thus one matrix multiplication
// we should move to that. I (caleb) only did it this way at first because it was much much easier to test
__device__ float3 ssrlcv::rotatePointKP(float3 point, float3 goal, float axangle) {

    // old
  // // we rotate in the z axis around <0 0 1> first, so we make a z rotation matrix
  // float zRotationMatrix[3][3];
  // zRotationMatrix[0][0] = cosf(axangle);
  // zRotationMatrix[0][1] = -1 * sinf(axangle);
  // zRotationMatrix[0][2] = 0;
  // zRotationMatrix[1][0] = sinf(axangle);
  // zRotationMatrix[1][1] = cosf(axangle);
  // zRotationMatrix[1][2] = 0;
  // zRotationMatrix[2][0] = 0;
  // zRotationMatrix[2][1] = 0;
  // zRotationMatrix[2][2] = 1;
  // point = matrixMulVector(point, zRotationMatrix);



  // now calculate the needed rotation in the x axis
  // this isn't explicitly given, so we have to calculate it
  // from our current heading <0 0 1>
  float3 tempgoal = {0.0, goal.y, goal.z};
  normalize(tempgoal);
  float xrot = acosf( dotProduct({1.0, 0.0, 0.0}, tempgoal));
  printf("xRotation Estimate: %f \n", xrot);
  float xRotationMatrix[3][3];
  xRotationMatrix[0][0] = 1;
  xRotationMatrix[0][1] = 0;
  xRotationMatrix[0][2] = 0;
  xRotationMatrix[1][0] = 0;
  xRotationMatrix[1][1] = cosf(xrot);
  xRotationMatrix[1][2] = -1 * sinf(xrot);
  xRotationMatrix[2][0] = 0;
  xRotationMatrix[2][1] = sinf(xrot);
  xRotationMatrix[2][2] = cosf(xrot);
  point = matrixMulVector(point, xRotationMatrix);
  // tempgoal =
  // Last step is to calculate the rotation in the y axis
  // using the same method and assumtion as above.
  float3 tempgoal2 = {goal.x, 0.0, goal.z};
  normalize(tempgoal2);
  float yrot = acosf( dotProduct({0.0, 1.0, 0.0}, tempgoal2));
  printf("yRotation Estimate: %f \n", yrot);
  float yRotationMatrix[3][3];
  yRotationMatrix[0][0] = cosf(yrot);
  yRotationMatrix[0][1] = 0;
  yRotationMatrix[0][2] = sinf(yrot);
  yRotationMatrix[1][0] = 0;
  yRotationMatrix[1][1] = 1;
  yRotationMatrix[1][2] = 0;
  yRotationMatrix[2][0] = -1 * sinf(yrot);
  yRotationMatrix[2][1] = 0;
  yRotationMatrix[2][2] = cosf(yrot);
  point = matrixMulVector(point, yRotationMatrix);
  // now calc the change in the z axis that is needed
  float3 tempgoal3 = {goal.x, goal.y, 0.0};
  normalize(tempgoal3);
  float zrot = acosf( dotProduct({0.0, 0.0, 1.0}, tempgoal3));
  printf("zRotation Estimate: %f \n", zrot);
  float zRotationMatrix[3][3];
  zRotationMatrix[0][0] = cosf(zrot);
  zRotationMatrix[0][1] = -1 * sinf(zrot);
  zRotationMatrix[0][2] = 0;
  zRotationMatrix[1][0] = sinf(zrot);
  zRotationMatrix[1][1] = cosf(zrot);
  zRotationMatrix[1][2] = 0;
  zRotationMatrix[2][0] = 0;
  zRotationMatrix[2][1] = 0;
  zRotationMatrix[2][2] = 1;
  point = matrixMulVector(point, zRotationMatrix);
  // then we're done!
  // then we're done!
  return point;
}


















































// yeet
